#include "hip/hip_runtime.h"
#include "RapidCloth.cuh"

namespace zeno{

template <int codim>
typename RapidClothSystem::T RapidClothSystem::infNorm(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString& tag, std::size_t maxInd, zs::wrapv<codim>) {
    using namespace zs;
    using T = typename RapidClothSystem::T;
    constexpr auto space = execspace_e::cuda;
    auto nwarps = count_warps(maxInd);
    temp.resize(nwarps);
    cudaPol(range(nwarps * 32), [data = view<space>({}, vtemp), res = view<space>(temp), n = maxInd,
                             offset = vtemp.getPropertyOffset(tag)] __device__(int pi) mutable {
        T val = 0; 
        if (pi < n)
        {
            auto v = data.pack(dim_c<codim>, offset, pi);
            val = v.abs().max();            
        }

#if __CUDA_ARCH__ >= 800
        auto tile = zs::cg::tiled_partition<32>(zs::cg::this_thread_block());
        auto ret = zs::cg::reduce(tile, val, zs::cg::greater<T>());
        if (tile.thread_rank() == 0)
            res[pi / 32] = ret;
#else
        auto [mask, numValid] = warp_mask(pi, n);
        auto locid = threadIdx.x & 31;
        for (int stride = 1; stride < 32; stride <<= 1) {
            auto tmp = __shfl_down_sync(mask, val, stride);
            if (locid + stride < numValid)
                val = zs::max(val, tmp);
        }
        if (locid == 0)
            res[pi / 32] = val;
#endif
    });
    return reduce(cudaPol, temp, thrust::maximum<T>{});
}
template typename RapidClothSystem::T RapidClothSystem::infNorm<3>(
    zs::CudaExecutionPolicy&, const zs::SmallString&, std::size_t, zs::wrapv<3>); 
template typename RapidClothSystem::T RapidClothSystem::infNorm<1>(
    zs::CudaExecutionPolicy&, const zs::SmallString&, std::size_t, zs::wrapv<1>); 

typename RapidClothSystem::T RapidClothSystem::l2Norm(zs::CudaExecutionPolicy &pol, const zs::SmallString &tag, std::size_t maxInd) {
    return zs::sqrt(dot(pol, tag, tag, maxInd));
}

typename RapidClothSystem::T RapidClothSystem::dot(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString &tag0,
                                                 const zs::SmallString &tag1, std::size_t maxInd) {
    using namespace zs;
    using T = typename RapidClothSystem::T;
    constexpr auto space = execspace_e::cuda;
    auto nwarps = count_warps(maxInd);
    temp.resize(nwarps);
    temp.reset(0);
    cudaPol(range(maxInd), [data = view<space>({}, vtemp), res = view<space>(temp), n = maxInd,
                             offset0 = vtemp.getPropertyOffset(tag0),
                             offset1 = vtemp.getPropertyOffset(tag1)] __device__(int pi) mutable {
        auto v0 = data.pack(dim_c<3>, offset0, pi);
        auto v1 = data.pack(dim_c<3>, offset1, pi);
        reduce_to(pi, n, v0.dot(v1), res[pi / 32]);
    });
    return reduce(cudaPol, temp, thrust::plus<T>{});
}
}