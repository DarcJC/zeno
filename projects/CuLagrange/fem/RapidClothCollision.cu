#include "hip/hip_runtime.h"
#include "RapidCloth.cuh"
#include "RapidClothUtils.hpp"
#include "Structures.hpp"
#include "zensim/geometry/Friction.hpp"
#include "zensim/geometry/SpatialQuery.hpp"
#include "zensim/geometry/Distance.hpp"
#include "RapidClothGradHess.inl"

namespace zeno {
void RapidClothSystem::findConstraintsImpl(zs::CudaExecutionPolicy &pol, 
    typename RapidClothSystem::T radius, bool withBoundary, const zs::SmallString &tag)
{
    using namespace zs; 
    constexpr auto space = execspace_e::cuda; 
    
    // p -> t
    const auto &stbvh = withBoundary ? bouStBvh : stBvh;
    auto &stfront = withBoundary ? boundaryStFront : selfStFront;
    opt = ne; 
    pol(Collapse{stfront.size()},
        [spInds = proxy<space>({}, spInds), svOffset = svOffset, coOffset = coOffset, 
         eles = proxy<space>({}, withBoundary ? *coEles : stInds),
         vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(stbvh), 
         front = proxy<space>(stfront), tempPT = proxy<space>({}, tempPT),
         vCons = proxy<space>({}, vCons), 
         nPT = proxy<space>(nPT), radius, voffset = withBoundary ? coOffset : 0,
         frontManageRequired = frontManageRequired, tag] __device__(int i) mutable {
            auto vi = front.prim(i);
            vi = spInds("inds", vi, int_c); 
            const auto dHat2 = zs::sqr(radius);
            auto p = vtemp.pack(dim_c<3>, tag, vi);
            auto bv = bv_t{get_bounding_box(p - radius, p + radius)};
            auto f = [&](int stI) {
                auto tri = eles.pack(dim_c<3>, "inds", stI, int_c) + voffset;
                if (vi == tri[0] || vi == tri[1] || vi == tri[2])
                    return;
                // ccd
                auto t0 = vtemp.pack(dim_c<3>, tag, tri[0]);
                auto t1 = vtemp.pack(dim_c<3>, tag, tri[1]);
                auto t2 = vtemp.pack(dim_c<3>, tag, tri[2]);

                if (auto d2 = dist2_pt(p, t0, t1, t2); d2 < dHat2) {
                    auto no = atomic_add(exec_cuda, &nPT[0], 1); 
                    auto inds = pair4_t{vi, tri[0], tri[1], tri[2]}; 
                    tempPT.tuple(dim_c<4>, "inds", no, int_c) = inds; 
                    // auto consInd = no + opt; 
                    // for (int k = 0; k < 4; k++)
                    // {
                    //     auto no = atomic_add(exec_cuda, &vCons("n", inds[k]), 1); 
                    //     // vCons("cons", no, inds[k]) = consInd; 
                    // }
                }
            }; 

            if (frontManageRequired)
                bvh.iter_neighbors(bv, i, front, f);
            else
                bvh.iter_neighbors(bv, front.node(i), f);
        });
    if (frontManageRequired)
        stfront.reorder(pol);   
    // npt = nPT.getVal(); 
    // oee = opt + npt; 

    // e -> e
    const auto &sebvh = withBoundary ? bouSeBvh : seBvh;
    auto &seefront = withBoundary ? boundarySeeFront : selfSeeFront;
    pol(Collapse{seefront.size()},
        [seInds = proxy<space>({}, seInds), sedges = proxy<space>({}, withBoundary ? *coEdges : seInds),
            vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(sebvh), front = proxy<space>(seefront),
            vCons = proxy<space>({}, vCons), 
            tempEE = proxy<space>({}, tempEE), nEE = proxy<space>(nEE), dHat2 = zs::sqr(radius),
            radius, voffset = withBoundary ? coOffset : 0,
            frontManageRequired = frontManageRequired, tag] __device__(int i) mutable {
            auto sei = front.prim(i);
            auto eiInds = seInds.pack(dim_c<2>, "inds", sei, int_c);
            auto v0 = vtemp.pack(dim_c<3>, tag, eiInds[0]);
            auto v1 = vtemp.pack(dim_c<3>, tag, eiInds[1]);
            auto [mi, ma] = get_bounding_box(v0, v1);
            auto bv = bv_t{mi - radius, ma + radius};
            auto f = [&](int sej) {
                if (voffset == 0 && sei <= sej)
                    return;
                auto ejInds = sedges.pack(dim_c<2>, "inds", sej, int_c) + voffset;
                if (eiInds[0] == ejInds[0] || eiInds[0] == ejInds[1] || eiInds[1] == ejInds[0] ||
                    eiInds[1] == ejInds[1])
                    return;
                auto v2 = vtemp.pack(dim_c<3>, tag, ejInds[0]);
                auto v3 = vtemp.pack(dim_c<3>, tag, ejInds[1]);

                if (auto d2 = dist2_ee(v0, v1, v2, v3); d2 < dHat2) {
                    auto no = atomic_add(exec_cuda, &nEE[0], 1); 
                    auto inds = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                    tempEE.tuple(dim_c<4>, "inds", no, int_c) = inds; 
                    // auto consInd = no + oee; 
                    // for (int k = 0; k < 4; k++)
                    // {
                    //     auto no = atomic_add(exec_cuda, &vCons("n", inds[k]), 1); 
                    //     // vCons("cons", no, inds[k]) = consInd; 
                    // }
                }
            };
            if (frontManageRequired)
                bvh.iter_neighbors(bv, i, front, f);
            else
                bvh.iter_neighbors(bv, front.node(i), f);
        });
    if (frontManageRequired)
        seefront.reorder(pol);
    // nee = nEE.getVal(); 
    // ope = oee + nee; 

    // e -> p 
    auto &sevfront = withBoundary ? boundarySevFront : selfSevFront;
    pol(Collapse{sevfront.size()},
        [spInds = proxy<space>({}, spInds), svOffset = svOffset, coOffset = coOffset, 
            sedges = proxy<space>({}, withBoundary ? *coEdges : seInds),
            vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(sebvh), front = proxy<space>(sevfront),
            tempPE = proxy<space>({}, tempPE), nPE = proxy<space>(nPE), dHat2 = zs::sqr(radius),
            radius, voffset = withBoundary ? coOffset : 0,
            frontManageRequired = frontManageRequired, tag] __device__(int i) mutable {
            auto vi = front.prim(i);
            vi = spInds("inds", vi, int_c); 
            const auto dHat2 = zs::sqr(radius);
            auto p = vtemp.pack(dim_c<3>, tag, vi);
            auto bv = bv_t{get_bounding_box(p - radius, p + radius)};
            auto f = [&](int sej) {
                auto ejInds = sedges.pack(dim_c<2>, "inds", sej, int_c) + voffset;
                if (vi == ejInds[0] || vi == ejInds[1])
                    return; 
                auto v2 = vtemp.pack(dim_c<3>, tag, ejInds[0]);
                auto v3 = vtemp.pack(dim_c<3>, tag, ejInds[1]);

                if (auto d2 = dist2_pe(p, v2, v3); d2 < dHat2) {
                    auto no = atomic_add(exec_cuda, &nPE[0], 1); 
                    auto inds = pair3_t{vi, ejInds[0], ejInds[1]};
                    tempPE.tuple(dim_c<3>, "inds", no, int_c) = inds; 
                    // auto consInd = no + ope; 
                    // for (int k = 0; k < 3; k++)
                    // {
                    //     auto no = atomic_add(exec_cuda, &vCons("n", inds[k]), 1); 
                    //     // vCons("cons", no, inds[k]) = consInd; 
                    // }
                }
            };
            if (frontManageRequired)
                bvh.iter_neighbors(bv, i, front, f);
            else
                bvh.iter_neighbors(bv, front.node(i), f);
        });
    if (frontManageRequired)
        sevfront.reorder(pol);
    // npe = nPE.getVal(); 
    // opp = npe + ope; 

    // v-> v
    if (!withBoundary)
    {
        const auto &svbvh = svBvh;
        auto &svfront = selfSvFront;
        pol(Collapse{svfront.size()},
            [spInds = proxy<space>({}, spInds), svOffset = svOffset, coOffset = coOffset, 
            bvh = proxy<space>(svbvh), front = proxy<space>(svfront), tempPP = proxy<space>({}, tempPP),
            eles = proxy<space>({}, svInds), 
            vCons = proxy<space>({}, vCons), 
            vtemp = proxy<space>({}, vtemp), 
            nPP = proxy<space>(nPP), radius, voffset = withBoundary ? coOffset : 0,
            frontManageRequired = frontManageRequired, tag] __device__(int i) mutable {
                auto svI = front.prim(i);
                auto vi = spInds("inds", svI, int_c); 
                const auto dHat2 = zs::sqr(radius);
                auto pi = vtemp.pack(dim_c<3>, tag, vi);
                auto bv = bv_t{get_bounding_box(pi - radius, pi + radius)};
                auto f = [&](int svJ) {
                    if (voffset == 0 && svI <= svJ)
                        return; 
                    auto vj = eles("inds", svJ, int_c) + voffset; 
                    auto pj = vtemp.pack(dim_c<3>, tag, vj); 
                    if (auto d2 = dist2_pp(pi, pj); d2 < dHat2) {
                        auto no = atomic_add(exec_cuda, &nPP[0], 1); 
                        auto inds = pair_t{vi, vj};
                        tempPP.tuple(dim_c<2>, "inds", no, int_c) = inds; 
                    }
                }; 

                if (frontManageRequired)
                    bvh.iter_neighbors(bv, i, front, f);
                else
                    bvh.iter_neighbors(bv, front.node(i), f);
            });     
        if (frontManageRequired)
            svfront.reorder(pol);   
        // npp = nPP.getVal(); 
    }
}

void RapidClothSystem::findConstraints(zs::CudaExecutionPolicy &pol, T dist, const zs::SmallString &tag)
{
    // TODO: compute oE in initialize
    using namespace zs; 
    constexpr auto space = execspace_e::cuda; 

    nPP.setVal(0);
    nPE.setVal(0);  
    nPT.setVal(0); 
    nEE.setVal(0); 
    pol(range(vCons.size()), [vCons = proxy<space>({}, vCons)] __device__ (int i) mutable {
        vCons("n", i) = vCons("nE", i); 
    }); 
    
    // nE.setVal(0); TODO: put into findEdgeConstraints(bool init = false) and calls it in every iteration 

    // collect PP, PE, PT, EE, E constraints from bvh 
    bvs.resize(svInds.size()); 
    retrieve_bounding_volumes(pol, vtemp, tag, svInds, zs::wrapv<1>{}, 0, bvs);
    svBvh.refit(pol, bvs); 
    bvs.resize(stInds.size());
    retrieve_bounding_volumes(pol, vtemp, tag, stInds, zs::wrapv<3>{}, 0, bvs);
    stBvh.refit(pol, bvs);
    bvs.resize(seInds.size());
    retrieve_bounding_volumes(pol, vtemp, tag, seInds, zs::wrapv<2>{}, 0, bvs);
    seBvh.refit(pol, bvs);

    findConstraintsImpl(pol, dist, false, tag); 

    if (hasBoundary()) {
        bvs.resize(coEles->size());
        retrieve_bounding_volumes(pol, vtemp, tag, *coEles, zs::wrapv<3>{}, coOffset, bvs); 
        bouStBvh.refit(pol, bvs); 
        bvs.resize(coEdges->size()); 
        retrieve_bounding_volumes(pol, vtemp, tag, *coEdges, zs::wrapv<2>{}, coOffset, bvs);
        bouSeBvh.refit(pol, bvs);

        findConstraintsImpl(pol, dist, true, tag); 
    }

    updateConstraintCnt(); 
    D = D_max; 
    // TODO: coloring for multi-color PGS 
    consColoring(pol); 
}


static void constructVertexConsList(zs::CudaExecutionPolicy &pol, 
    typename RapidClothSystem::tiles_t& tempPair, 
    typename RapidClothSystem::itiles_t& vCons, 
    int pairNum, 
    int pairSize, 
    std::size_t offset)
{
    using namespace zs; 
    constexpr auto space = execspace_e::cuda; 

    pol(range(pairNum), 
        [tempPair = proxy<space>({}, tempPair), 
         vCons = proxy<space>({}, vCons), 
         offset, pairSize] __device__ (int i) mutable {
            for (int k = 0; k < pairSize; k++)
            {
                auto vi = tempPair("inds", k, i, int_c); 
                auto n = atomic_add(exec_cuda, &vCons("n", vi), 1); 
                auto nE = vCons("nE", vi); 
                vCons("cons", n + nE, vi) = i + offset; 
                vCons("cons", n + nE, vi) = k; 
            }
        }); 
}

static void constructEEVertexConsList(zs::CudaExecutionPolicy &pol, 
    typename RapidClothSystem::tiles_t& tempE, 
    typename RapidClothSystem::itiles_t& vCons, 
    int pairNum)
{
    using namespace zs; 
    constexpr auto space = execspace_e::cuda; 

    pol(range(pairNum), 
        [tempE = proxy<space>({}, tempE), 
         vCons = proxy<space>({}, vCons)] __device__ (int i) mutable {
            for (int k = 0; k < 2; k++)
            {
                auto vi = tempE("inds", k, i, int_c); 
                auto nE = atomic_add(exec_cuda, &vCons("nE", vi), 1); 
                vCons("cons", nE, vi) = i; 
                vCons("ind", nE, vi) = k; 
            }
        }); 
}

void RapidClothSystem::initPalettes(zs::CudaExecutionPolicy &pol, 
    typename RapidClothSystem::tiles_t &tempPair, 
    typename RapidClothSystem::itiles_t &vCons, 
    typename RapidClothSystem::itiles_t &tempCons, 
    int pairNum, 
    int pairSize, 
    std::size_t offset, 
    typename RapidClothSystem::T shrinking)
{
    using namespace zs; 
    constexpr auto space = execspace_e::cuda; 
    pol(range(pairNum), 
        [tempPair = proxy<space>({}, tempPair), 
         vCons = proxy<space>({}, vCons), 
         tempCons = proxy<space>({}, tempCons), 
         lcpMatIs = proxy<space>(lcpMatIs), 
         lcpMatJs = proxy<space>(lcpMatJs), 
         lcpMatSize = proxy<space>(lcpMatSize), 
         pairSize, offset, shrinking] __device__ (int i) mutable {
            int degree = 0; 
            for (int k = 0; k < pairSize; k++)
            {
                auto vi = tempPair("inds", k, i, int_c); 
                auto nE = vCons("nE", vi); 
                auto n = vCons("n", vi); 
                degree += nE + n; 
                tempCons("vi", k, i + offset) = vi; 
                for (int j = 0; j < nE + n; j++)
                {
                    int aj = vCons("cons", j, vi); 
                    auto no = atomic_add(exec_cuda, &lcpMatSize[0], 1); 
                    lcpMatIs[no] = i; 
                    lcpMatJs[no] = aj; 
                }
            }
            int max_color = (int)zs::ceil(((T)degree) / shrinking); 
            if (max_color < 2)
                max_color = 2;
            tempCons("fixed", i + offset) = 0; 
            tempCons("max_color", i + offset) = max_color; 
            tempCons("num_color", i + offset) = max_color; 
            constexpr int len = sizeof(int) * 8; 
            tempCons("colors", i + offset) = (1 << (len - 2)) - 1 + (1 << (len - 2)); 
            tempCons("vN", i + offset) = pairSize; 
         }); 
    auto lcpSize = lcpMatSize.getVal();
    lcpMatIs.resize(lcpSize); 
    lcpMatJs.resize(lcpSize); 
}

static constexpr int simple_hash(int a)
{
    // https://burtleburtle.net/bob/hash/integer.html
    a = (a ^ 61) ^ (a >> 16);
    a = a + (a << 3);
    a = a ^ (a >> 4);
    a = a * 0x27d4eb2d;
    a = a ^ (a >> 15);
    return a; 
}

bool RapidClothSystem::checkConsColoring(zs::CudaExecutionPolicy &pol)
{
    using namespace zs; 
    constexpr auto space = execspace_e::cuda; 

    zs::Vector<int> correct; 
    correct.setVal(1); 
    pol(range(nCons), 
        [tempCons = proxy<space>({}, tempCons), 
         vCons = proxy<space>({}, vCons), 
         lcpMat = proxy<space>(lcpMat), 
         correct = proxy<space>(correct)] __device__ (int i) mutable {
            int color = tempCons("color", i); 
            auto &ap = lcpMat._ptrs; 
            auto &aj = lcpMat._inds; 
            for (int k = ap[i]; k < ap[i + 1]; k++)
            {
                if (tempCons("color", aj[k]) == color)
                {
                    correct[k] = 0;  
                    return; 
                }
            }
        }); 
    return correct.getVal() == 1; 
}

void RapidClothSystem::consColoring(zs::CudaExecutionPolicy &pol, T shrinking)
{
    // TOOD: use SparseMatrix 
    using namespace zs; 
    constexpr auto space = execspace_e::cuda; 
    // clear vertex -> cons list size 
    pol(range(vCons.size()), 
        [vCons = proxy<space>({}, vCons)] __device__ (int i) mutable {
            vCons("n", i) = 0; 
            vCons("nE", i) = 0; 
        }); 
    // construct vertex -> cons list 
    constructEEVertexConsList(pol, tempE, vCons, ne); 
    constructVertexConsList(pol, tempPP, vCons, npp, 2, opp); 
    constructVertexConsList(pol, tempPE, vCons, npe, 3, ope); 
    constructVertexConsList(pol, tempPT, vCons, npt, 4, opt); 
    constructVertexConsList(pol, tempEE, vCons, nee, 4, oee); 
    // construct cons adj list 
    lcpMatSize.setVal(0); 
    initPalettes(pol, tempE, vCons, tempCons, ne, 2, 0, shrinking); 
    initPalettes(pol, tempPP, vCons, tempCons, npp, 2, opp, shrinking); 
    initPalettes(pol, tempPE, vCons, tempCons, npe, 3, ope, shrinking); 
    initPalettes(pol, tempPT, vCons, tempCons, npt, 4, opt, shrinking); 
    initPalettes(pol, tempEE, vCons, tempCons, nee, 4, oee, shrinking); 
    lcpMat.build(pol, nCons, nCons, lcpMatIs, lcpMatJs); 
    lcpMat.localOrdering(pol, 128); 
    // TODO: construct cons adj list by constructing a sparse matrix 
    // cons graph coloring 
    zs::Vector<int> finished; 
    finished.setVal(1); 
    int seed = 0; 
    while (!finished.getVal())
    { 
        // pick random color for unfixed constraints
        pol(range(nCons), 
            [tempCons = proxy<space>({}, tempCons), seed = seed++] __device__ (int i) mutable {
                tempCons("tmp", i) = 0; 
                if (tempCons("fixed", i))
                    return; 
                int ind = simple_hash(simple_hash(seed) + simple_hash(i)) % tempCons("num_color", i);
                int colors = tempCons("colors", i); 
                int maxColor = tempCons("max_color", i); 
                int curInd = -1; 
                int pos = -1;  
                while(++pos < maxColor && colors)
                {
                    int digit = colors % 2; 
                    if (digit && (++curInd == ind))
                        break; 
                    colors >>= 1; 
                }
                if (curInd < ind)
                {
                    printf("err in coloring: palette exhausted in the random-picking phase!\n"); 
                    return; 
                }
                tempCons("color", i) = pos; 
            }); 

        // conflict resolution: fix the colors of 'good' constraints, remove them from their neighbors' palettes
        pol(range(nCons), 
            [tempCons = proxy<space>({}, tempCons), 
             lcpMat = proxy<space>(lcpMat)] __device__ (int i) mutable {
                if (tempCons("fixed", i))
                    return; 
                int color = tempCons("color", i); 
                bool flagConflict = false; 
                bool flagHigherInd = true; 
                auto &ap = lcpMat._ptrs; 
                auto &aj = lcpMat._inds; 
                for (int k = ap[i]; k < ap[i + 1]; k++)
                {
                    int neCons = aj[k]; // neighbor constraint 
                    if (neCons == i)
                        continue; 
                    if (neCons > i)
                        flagHigherInd = false; 
                    int neColor = tempCons("color", neCons); 
                    if (neCons == color)
                        flagConflict = true; 
                    if (flagConflict && !flagHigherInd)
                        break; 
                }
                if (!flagConflict || flagHigherInd)
                {
                    tempCons("fixed", i) = 1; 
                    tempCons("tmp", i) = 1; // 1 means need to remove current color from neighbors' palettes
                }
             }); 

        pol(range(nCons), 
            [tempCons = proxy<space>({}, tempCons), 
             lcpMat = proxy<space>(lcpMat), 
             vCons = proxy<space>({}, vCons)] __device__ (int i) mutable {
                if (tempCons("fixed", i))
                    return; 
                int maxColor = tempCons("max_color", i); 
                int numColor = tempCons("num_color", i); 
                int colors = tempCons("colors", i); 
                auto &ap = lcpMat._ptrs; 
                auto &aj = lcpMat._inds; 
                for (int k = ap[i]; k < ap[i + 1]; k++)
                {
                    int neCons = aj[k]; // neighbor constraint 
                    if (neCons == i)
                        continue; 
                    if (tempCons("tmp", neCons))
                    {
                        int neColor = tempCons("color", neCons); 
                        if (neColor >= maxColor)
                            continue; 
                        if ((colors >> neColor) % 2)
                        {
                            numColor--; 
                            colors -= (1 << neColor); 
                        }
                    }
                }
                tempCons("colors", i) = colors; 
                tempCons("num_color", i) = numColor; 
             }); 

        // feed the hungry & check if finished 
        finished.setVal(1); 
        pol(range(nCons), 
            [tempCons = proxy<space>({}, tempCons), 
            finished = proxy<space>(finished)] __device__ (int i) mutable {
                if (tempCons("fixed", i))
                    return; 
                finished[0] = 1; 
                if (tempCons("num_color", i) == 0)
                    tempCons("max_color", i) += 1; 
            }); 
    }

    consColorBits.reset(0); 
    pol(range(nCons), 
        [tempCons = proxy<space>({}, tempCons), 
         consColorBits = proxy<space>(consColorBits)] __device__ (int i) mutable {
            consColorBits[tempCons("color", i)] = 1; 
         }); 
    nConsColor = 0; 
    for (int i = consColorBits.size() - 1; i >= 0; i--)
        if (consColorBits[i] == 1)
        {
            nConsColor = i; 
            break; 
        }
    fmt::print("\t\t[graph coloring] Ended with {} colors\n", nConsColor + 1); 

    if (checkConsColoring(pol))
        fmt::print("\t\t[graph coloring] The result is correct.\n");
    else 
        fmt::print("\t\t[graph coloring] Wrong results!"); 
}


// xl, cons -> c(xl), J(xl)   
void RapidClothSystem::computeConstraints(zs::CudaExecutionPolicy &pol, const zs::SmallString &tag)
{
    // TODO: use SparseMatrix to store J * M^{-1} * J.T
    using namespace zs; 
    constexpr auto space = execspace_e::cuda; 
    pol(range(ne), [vtemp = proxy<space>({}, vtemp), 
                    tempE = proxy<space>({}, tempE), 
                    tempCons = proxy<space>({}, tempCons), 
                    oe = oe, sigma = sigma, tag] __device__ (int i) mutable {
        // calculate grad 
        int consInd = i + oe; 
        auto inds = tempE.pack(dim_c<2>, "inds", i, int_c); 
        auto xi = vtemp.pack(dim_c<3>, tag, inds[0]); 
        auto xj = vtemp.pack(dim_c<3>, tag, inds[1]);
        auto yi = vtemp.pack(dim_c<3>, "y[k+1]", inds[0]); 
        auto yj = vtemp.pack(dim_c<3>, "y[k+1]", inds[1]);
        auto xij_norm = (xi - xj).norm() + limits<T>::epsilon(); 
        auto yij_norm_inv = 1.0f / ((yi - yj).norm() + limits<T>::epsilon()); 
        auto grad = - (xi - xj) / xij_norm * yij_norm_inv; 
        auto val = sigma - xij_norm * yij_norm_inv; 
        for (int d = 0; d < 3; d++)
            tempCons("grad", d, consInd, T_c) = grad(d); 
        for (int d = 0; d < 3; d++)
            tempCons("grad", d + 3, consInd, T_c) = -grad(d); 
        tempCons("val", consInd, T_c) = val; 
    }); 

    pol(range(npp), [vtemp = proxy<space>({}, vtemp), 
                    tempPP = proxy<space>({}, tempPP), 
                    tempCons = proxy<space>({}, tempCons), 
                    opp = opp, delta = delta, tag] __device__ (int i) mutable {
        // calculate grad 
        int consInd = i + opp; 
        auto inds = tempPP.pack(dim_c<2>, "inds", i, int_c); 
        auto xi = vtemp.pack(dim_c<3>, tag, inds[0]); 
        auto xj = vtemp.pack(dim_c<3>, tag, inds[1]);
        auto xij_norm = (xi - xj).norm() + limits<T>::epsilon(); 
        auto delta_inv = 1.0f / delta; 
        auto grad = (xi - xj) / xij_norm * delta_inv; 
        auto val = xij_norm * delta_inv - 1.0f; 
        for (int d = 0; d < 3; d++)
            tempCons("grad", d, consInd, T_c) = grad(d); 
        for (int d = 0; d < 3; d++)
            tempCons("grad", d + 3, consInd, T_c) = -grad(d); 
        tempCons("val", consInd) = val; 
    }); 

    pol(range(npe), [vtemp = proxy<space>({}, vtemp), 
                    tempPE = proxy<space>({}, tempPE), 
                    tempCons = proxy<space>({}, tempCons), 
                    ope = ope, delta = delta, tag] __device__ (int i) mutable {
        // calculate grad 
        int consInd = i + ope; 
        auto inds = tempPE.pack(dim_c<3>, "inds", i, int_c); 
        auto p = vtemp.pack(dim_c<3>, tag, inds[0]); 
        auto e0 = vtemp.pack(dim_c<3>, tag, inds[1]); 
        auto e1 = vtemp.pack(dim_c<3>, tag, inds[2]); 
        zs::vec<T, 9> grad; 
        PE_area2_grad(p.data(), e0.data(), e1.data(), grad.data()); 
        auto area = (e0 - p).cross(e1 - p).norm(); 
        T coef = (e1 - e0).norm() * delta; 
        grad /= (2.0f * area * coef + limits<T>::epsilon()); 
        tempCons.tuple(dim_c<9>, "grad", consInd, T_c) = grad; 
        tempCons("val", consInd, T_c) = area / coef - 1.0f; 
    }); 

    pol(range(npt), [vtemp = proxy<space>({}, vtemp), 
                    tempPT = proxy<space>({}, tempPT), 
                    tempCons = proxy<space>({}, tempCons), 
                    opt = opt, delta = delta, tag] __device__ (int i) mutable {
        // calculate grad 
        int consInd = i + opt; 
        auto inds = tempPT.pack(dim_c<4>, "inds", i, int_c); 
        auto p = vtemp.pack(dim_c<3>, tag, inds[0]); 
        auto t0 = vtemp.pack(dim_c<3>, tag, inds[1]); 
        auto t1 = vtemp.pack(dim_c<3>, tag, inds[2]); 
        auto t2 = vtemp.pack(dim_c<3>, tag, inds[3]); 
        zs::vec<T, 3, 3> mat;
        for (int d = 0; d < 3; d++)
        {
            mat(d, 0) = t0(d) - p(d); 
            mat(d, 1) = t1(d) - p(d); 
            mat(d, 2) = t2(d) - p(d); 
        }
        auto vol = determinant(mat); 
        auto sgn = vol > 0 ? 1.0f : -1.0f; 
        auto coef = sgn * (t1 - t0).cross(t2 - t0).norm() * delta + limits<T>::epsilon(); 
        mat = adjoint(mat).transpose();

        zs::vec<T, 3, 4> grad; 
        for (int d = 0; d < 3; d++)
            grad(d, 3) = 0; 
        for (int k = 1; k < 4; k++)
            for (int d = 0; d < 3; d++)
            {
                grad(d, k) = mat(d, k); 
                grad(d, 0) -= mat(d, k); 
            }
        grad /= coef; 
        tempCons.tuple(dim_c<12>, "grad", consInd, T_c) = grad; 
        tempCons("val", consInd, T_c) = vol / coef - 1.0f; 
    }); 

    pol(range(nee), [vtemp = proxy<space>({}, vtemp), 
                    tempEE = proxy<space>({}, tempEE), 
                    tempCons = proxy<space>({}, tempCons), 
                    opt = opt, delta = delta, tag] __device__ (int i) mutable {
        // calculate grad 
        int consInd = i + opt; 
        auto inds = tempEE.pack(dim_c<4>, "inds", i, int_c); 
        auto ei0 = vtemp.pack(dim_c<3>, tag, inds[0]); 
        auto ei1 = vtemp.pack(dim_c<3>, tag, inds[1]); 
        auto ej0 = vtemp.pack(dim_c<3>, tag, inds[2]); 
        auto ej1 = vtemp.pack(dim_c<3>, tag, inds[3]); 
        zs::vec<T, 3, 3> mat, rMat;
        for (int d = 0; d < 3; d++)
        {
            mat(d, 0) = ei1(d) - ei0(d); 
            mat(d, 1) = ei1(d) - ei0(d); 
            mat(d, 2) = ej1(d) - ei0(d); 
        }
        auto vol = determinant(mat); 
        auto gammas = edge_edge_closest_point(ei0, ei1, ej0, ej1);
        auto pi =  gammas[0] * (ei1 - ei0) + ei0; 
        auto pj = gammas[1] * (ej1 - ej0) + ej0; 
        auto dij = pj - pi; 
        auto dist = dij.norm(); 
        auto ri0 = ei0 + (dist - delta) * 0.5f * dij; 
        auto ri1 = ei1 + (dist - delta) * 0.5f * dij; 
        auto rj0 = ej0 - (dist - delta) * 0.5f * dij;
        auto rj1 = ej1 - (dist - delta) * 0.5f * dij; 
        for (int d = 0; d < 3; d++)
        {
            rMat(d, 0) = ri1(d) - ri0(d); 
            rMat(d, 1) = ri1(d) - ri0(d); 
            rMat(d, 2) = rj1(d) - ri0(d); 
        }
        auto coef = determinant(rMat);  
        mat = adjoint(mat).transpose();

        zs::vec<T, 3, 4> grad; 
        for (int d = 0; d < 3; d++)
            grad(d, 3) = 0; 
        for (int k = 1; k < 4; k++)
            for (int d = 0; d < 3; d++)
            {
                grad(d, k) = mat(d, k); 
                grad(d, 0) -= mat(d, k); 
            }
        grad /= coef; 
        tempCons.tuple(dim_c<12>, "grad", consInd, T_c) = grad; 
        tempCons("val", consInd, T_c) = vol / coef - 1.0f; 
    }); 

    // compute lcpMat = J * M^{-1} * J.T
    pol(range(lcpMat.nnz()), 
        [lcpMat = proxy<space>(lcpMat)] __device__ (int i) mutable {
            auto &ax = lcpMat._vals; 
            ax[i] = 0.f;
        });

    pol(range(nCons), 
        [tempCons = proxy<space>({}, tempCons), 
        vCons = proxy<space>({}, vCons), 
        vtemp = proxy<space>({}, vtemp), 
        lcpMat = proxy<space>(lcpMat)] __device__ (int i) mutable {
            auto &ap = lcpMat._ptrs;
            auto &aj = lcpMat._inds; 
            auto &ax = lcpMat._vals; 
            auto vN = tempCons("vN", i); 
            for (int j = 0; j < vN; j++)                        // this V
            {
                int vi = tempCons("vi", j, i); 
                int n = vCons("n", vi) + vCons("ne", vi); 
                for (int k = 0; k < n; k++)
                {
                    int neCons = vCons("cons", k, vi); 
                    int neV = vCons("ind", k, vi); 
                    auto mInv = 1.0f / (limits<T>::epsilon() + vtemp("ws", k, vi));                // TODO: BC vert mass?  'ws' -> 'm'
                    // cons.grad(j) * m_inv * neCons.grad(neV)
                    int val = 0; 
                    for (int d = 0; d < 3; d++)
                        val += tempCons("grad", j * 3 + d, i) * mInv * 
                            tempCons("grad", neV * 3 + d, neCons); 
                    auto spInd = lcpMat.locate(i, neCons); 
                    ax[spInd] += val; 
                }
            }
        }); 
}

// yl, y[k], (c, J), xl -> lambda_{l+1}, y_{l+1} 
void RapidClothSystem::solveLCP(zs::CudaExecutionPolicy &pol)
{
    // PGS solver 
    using namespace zs; 
    constexpr auto space = execspace_e::cuda; 

    // b = c(x(l)) - J(x(l)) * (y[k+1] - x(l))
    pol(range(nCons), 
        [tempCons = proxy<space>({}, tempCons), 
         vtemp = proxy<space>({}, vtemp)] __device__ (int ci) mutable {   
            int val = tempCons("val", ci, T_c); 
            for (int i = 0; i < tempCons("vN", ci); i++)
            {
                for (int d = 0; d < 3; d++)
                    val -= vtemp("grad", i * 3 + d) * 
                        (vtemp("y[k+1]", d, i) - vtemp("x(l)", d, i)); 
            }
            tempCons("b", ci, T_c) = val; 
         }); 
    
    for (int iter = 0; iter < lcpCap; iter++)
    {
        lcpConverged.setVal(1); 
        for (int color = 0; color < nConsColor; color++)
        {
            pol(range(nCons), 
                [tempCons = proxy<space>({}, tempCons), 
                lcpMat = proxy<space>(lcpMat), 
                lcpConverged = proxy<space>(lcpConverged), 
                lcpTol = lcpTol] __device__ (int i) mutable {
                    auto &ap = lcpMat._ptrs; 
                    auto &aj = lcpMat._inds; 
                    auto &ax = lcpMat._vals;
                    auto oldLam = tempCons("lambda", i, T_c); 
                    T maj = 0.f; 
                    T rhs = tempCons("b", i, T_c); 
                    for (int k = ap[i]; k < ap[i + 1]; k++)
                    {
                        auto j = aj[k]; 
                        if (j == i)
                        {
                            maj += ax[k]; 
                            continue; 
                        }
                        rhs -= ax[k] * tempCons("lambda", j, T_c); 
                    } 
                    auto newLam = rhs / maj; 
                    tempCons("lambda", i, T_c) = newLam;
                    if (zs::abs(newLam - oldLam) > lcpTol)
                        lcpConverged[0] = 0; 
                }); 
        }
    if (lcpConverged.getVal())
        break;         
    }
}      

// call cons + solveLCP 
void RapidClothSystem::backwardStep(zs::CudaExecutionPolicy &pol)
{

}   

// async stepping  
void RapidClothSystem::forwardStep(zs::CudaExecutionPolicy &pol)
{

}
}