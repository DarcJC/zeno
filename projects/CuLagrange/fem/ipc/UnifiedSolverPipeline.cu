#include "hip/hip_runtime.h"
#include "../Ccds.hpp"
#include "UnifiedSolver.cuh"
#include "Utils.hpp"
#include "zensim/geometry/Distance.hpp"
#include "zensim/geometry/Friction.hpp"
#include "zensim/geometry/SpatialQuery.hpp"
#include "zensim/math/DihedralAngle.hpp"
#include "zensim/profile/CppTimers.hpp"
#include "zensim/types/SmallVector.hpp"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <zeno/utils/log.h>

#define PROFILE_IPC 0

namespace zeno {

inline typename UnifiedIPCSystem::T infNorm(zs::CudaExecutionPolicy &cudaPol,
                                            typename UnifiedIPCSystem::dtiles_t &vertData,
                                            const zs::SmallString tag = "dir") {
    using namespace zs;
    using T = typename UnifiedIPCSystem::T;
    constexpr auto space = execspace_e::cuda;
    Vector<T> res{vertData.get_allocator(), count_warps(vertData.size())};
    zs::memset(zs::mem_device, res.data(), 0, sizeof(T) * count_warps(vertData.size()));
    cudaPol(range(vertData.size()), [data = proxy<space>({}, vertData), res = proxy<space>(res), n = vertData.size(),
                                     offset = vertData.getPropertyOffset(tag)] __device__(int pi) mutable {
        auto v = data.pack<3>(offset, pi);
        auto val = v.abs().max();

#if __CUDA_ARCH__ >= 800
        auto tile = zs::cg::tiled_partition<32>(zs::cg::this_thread_block());
        auto ret = zs::cg::reduce(tile, val, zs::cg::greater<T>());
        if (tile.thread_rank() == 0)
            res[pi / 32] = ret;
#else
        auto [mask, numValid] = warp_mask(pi, n);
        auto locid = threadIdx.x & 31;
        for (int stride = 1; stride < 32; stride <<= 1) {
            auto tmp = __shfl_down_sync(mask, val, stride);
            if (locid + stride < numValid)
                val = zs::max(val, tmp);
        }
        if (locid == 0)
            res[pi / 32] = val;
#endif
    });
    return reduce(cudaPol, res, thrust::maximum<T>{});
}

void UnifiedIPCSystem::computeConstraints(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    pol(Collapse{numDofs}, [vtemp = proxy<space>({}, vtemp)] __device__(int vi) mutable {
        auto BCtarget = vtemp.pack(dim_c<3>, "BCtarget", vi);
        int BCorder = vtemp("BCorder", vi);
        // auto x = BCbasis.transpose() * vtemp.pack<3>("xn", vi);
        auto x = vtemp.pack(dim_c<3>, "xn", vi);
        int d = 0;
        for (; d != BCorder; ++d)
            vtemp("cons", d, vi) = x[d] - BCtarget[d];
        for (; d != 3; ++d)
            vtemp("cons", d, vi) = 0;
    });
}
bool UnifiedIPCSystem::areConstraintsSatisfied(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    computeConstraints(pol);
    auto res = constraintResidual(pol);
    return res < s_constraint_residual;
}
typename UnifiedIPCSystem::T UnifiedIPCSystem::constraintResidual(zs::CudaExecutionPolicy &pol, bool maintainFixed) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    if (projectDBC)
        return 0;
    Vector<T> num{vtemp.get_allocator(), numDofs}, den{vtemp.get_allocator(), numDofs};
    pol(Collapse{numDofs}, [vtemp = proxy<space>({}, vtemp), den = proxy<space>(den), num = proxy<space>(num),
                            maintainFixed] __device__(int vi) mutable {
        auto BCtarget = vtemp.pack(dim_c<3>, "BCtarget", vi);
        int BCorder = vtemp("BCorder", vi);
        auto cons = vtemp.pack(dim_c<3>, "cons", vi);
        auto xt = vtemp.pack(dim_c<3>, "xhat", vi);
        T n = 0, d_ = 0;
        // https://ipc-sim.github.io/file/IPC-supplement-A-technical.pdf Eq5
        for (int d = 0; d != BCorder; ++d) {
            n += zs::sqr(cons[d]);
            d_ += zs::sqr(xt[d] - BCtarget[d]);
        }
        num[vi] = n;
        den[vi] = d_;
        if (maintainFixed && BCorder > 0) {
            if (d_ != 0) {
                if (zs::sqrt(n / d_) < 1e-6)
                    vtemp("BCfixed", vi) = 1;
            } else {
                if (zs::sqrt(n) < 1e-6)
                    vtemp("BCfixed", vi) = 1;
            }
        }
    });
    auto nsqr = reduce(pol, num);
    auto dsqr = reduce(pol, den);
    T ret = 0;
    if (dsqr == 0)
        ret = std::sqrt(nsqr);
    else
        ret = std::sqrt(nsqr / dsqr);
    return ret < 1e-3 ? 0 : ret;
}

// https://developer.nvidia.com/blog/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/
namespace cg = cooperative_groups;
__forceinline__ __device__ int atomicAggInc(int *ctr) noexcept {
    auto g = cg::coalesced_threads();
    int warp_res;
    if (g.thread_rank() == 0)
        warp_res = atomicAdd(ctr, g.size());
    return g.shfl(warp_res, 0) + g.thread_rank();
}
#define USE_COALESCED 1

void UnifiedIPCSystem::markSelfIntersectionPrimitives(zs::CudaExecutionPolicy &pol, std::true_type) {
    // exclSes, exclSts, stInds, seInds, seBvh
    using namespace zs;
    exclSes.reset(0);
    exclSts.reset(0);
    exclBouSes.reset(0);
    exclBouSts.reset(0);

    csPT.reset();
    csEE.reset();
    // exclSes, exclSts, exclBouSes, exclBouSts

    if (enableContactSelf) {
        bvs.resize(stInds.size());
        retrieve_bounding_volumes(pol, vtemp, "xn", stInds, zs::wrapv<3>{}, 0, bvs);
        stBvh.refit(pol, bvs);
        bvs.resize(seInds.size());
        retrieve_bounding_volumes(pol, vtemp, "xn", seInds, zs::wrapv<2>{}, 0, bvs);
        seBvh.refit(pol, bvs);
        findProximityPairs(pol, dHat, xi, false);
    }

    if (hasBoundary()) {
        bvs.resize(coEles->size());
        retrieve_bounding_volumes(pol, vtemp, "xn", *coEles, zs::wrapv<3>{}, coOffset, bvs);
        bouStBvh.refit(pol, bvs);
        bvs.resize(coEdges->size());
        retrieve_bounding_volumes(pol, vtemp, "xn", *coEdges, zs::wrapv<2>{}, coOffset, bvs);
        bouSeBvh.refit(pol, bvs);
        findProximityPairs(pol, dHat, xi, true);
    }
    return;
}
void UnifiedIPCSystem::markSelfIntersectionPrimitives(zs::CudaExecutionPolicy &pol) {
    //exclSes, exclSts, stInds, seInds, seBvh
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    exclSes.reset(0);
    exclSts.reset(0);
    exclBouSes.reset(0);
    exclBouSts.reset(0);

    Vector<int> cnt{vtemp.get_allocator(), 1};
    cnt.setVal(0);

    bvs.resize(seInds.size());
    retrieve_bounding_volumes(pol, vtemp, "xn", seInds, wrapv<2>{}, 0, bvs);
    seBvh.refit(pol, bvs);
    pol(range(stInds.size()), [vtemp = proxy<space>({}, vtemp), stInds = proxy<space>({}, stInds),
                               seInds = proxy<space>({}, seInds), exclSes = proxy<space>(exclSes),
                               exclSts = proxy<space>(exclSts), bvh = proxy<space>(seBvh), cnt = proxy<space>(cnt),
                               dHat = dHat] __device__(int sti) mutable {
        auto tri = stInds.pack(dim_c<3>, "inds", sti, int_c);
        auto t0 = vtemp.pack(dim_c<3>, "xn", tri[0]);
        auto t1 = vtemp.pack(dim_c<3>, "xn", tri[1]);
        auto t2 = vtemp.pack(dim_c<3>, "xn", tri[2]);
        auto bv = bv_t{get_bounding_box(t0, t1)};
        merge(bv, t2);
        bool allFixed = vtemp("BCorder", tri[0]) == 3 && vtemp("BCorder", tri[1]) == 3 && vtemp("BCorder", tri[2]) == 3;
        bool triIntersected = false;
        bvh.iter_neighbors(bv, [&](int sei) {
            auto line = seInds.pack(dim_c<2>, "inds", sei, int_c);
            if (tri[0] == line[0] || tri[0] == line[1] || tri[1] == line[0] || tri[1] == line[1] || tri[2] == line[0] ||
                tri[2] == line[1])
                return;
            if (allFixed && vtemp("BCorder", line[0]) == 3 && vtemp("BCorder", line[1]) == 3)
                return;
            if (et_intersected(vtemp.pack(dim_c<3>, "xn", line[0]), vtemp.pack(dim_c<3>, "xn", line[1]), t0, t1, t2)) {
                triIntersected = true;
                exclSes[sei] = 1;

                atomic_add(exec_cuda, &cnt[0], 1);
            }
        });
        if (triIntersected)
            exclSts[sti] = 1;
    });
    zeno::log_info("{} self et intersections\n", cnt.getVal());

    if (hasBoundary()) {
        cnt.setVal(0);
        bvs.resize(coEdges->size());
        retrieve_bounding_volumes(pol, vtemp, "xn", *coEdges, zs::wrapv<2>{}, coOffset, bvs);
        bouSeBvh.refit(pol, bvs);
        pol(range(stInds.size()),
            [vtemp = proxy<space>({}, vtemp), stInds = proxy<space>({}, stInds), seInds = proxy<space>({}, *coEdges),
             exclBouSes = proxy<space>(exclBouSes), exclSts = proxy<space>(exclSts), bvh = proxy<space>(bouSeBvh),
             cnt = proxy<space>(cnt), dHat = dHat, voffset = coOffset] __device__(int sti) mutable {
                auto tri = stInds.pack(dim_c<3>, "inds", sti, int_c);
                auto t0 = vtemp.pack(dim_c<3>, "xn", tri[0]);
                auto t1 = vtemp.pack(dim_c<3>, "xn", tri[1]);
                auto t2 = vtemp.pack(dim_c<3>, "xn", tri[2]);
                auto bv = bv_t{get_bounding_box(t0, t1)};
                merge(bv, t2);
                bool allFixed =
                    vtemp("BCorder", tri[0]) == 3 && vtemp("BCorder", tri[1]) == 3 && vtemp("BCorder", tri[2]) == 3;
                bool triIntersected = false;
                bvh.iter_neighbors(bv, [&](int sei) {
                    auto line = seInds.pack(dim_c<2>, "inds", sei, int_c) + voffset;
                    // no need to check common vertices here
                    if (allFixed && vtemp("BCorder", line[0]) == 3 && vtemp("BCorder", line[1]) == 3)
                        return;
                    if (et_intersected(vtemp.pack(dim_c<3>, "xn", line[0]), vtemp.pack(dim_c<3>, "xn", line[1]), t0, t1,
                                       t2)) {
                        triIntersected = true;
                        exclBouSes[sei] = 1;

                        atomic_add(exec_cuda, &cnt[0], 1);
                    }
                });
                if (triIntersected)
                    exclSts[sti] = 1;
            });

        bvs.resize(coEles->size());
        retrieve_bounding_volumes(pol, vtemp, "xn", *coEles, zs::wrapv<3>{}, coOffset, bvs);
        bouStBvh.refit(pol, bvs);
        pol(range(seInds.size()),
            [vtemp = proxy<space>({}, vtemp), seInds = proxy<space>({}, seInds), coTris = proxy<space>({}, *coEles),
             exclBouSts = proxy<space>(exclBouSts), exclSes = proxy<space>(exclSes), bvh = proxy<space>(bouStBvh),
             cnt = proxy<space>(cnt), dHat = dHat, voffset = coOffset] __device__(int sei) mutable {
                auto line = seInds.pack(dim_c<2>, "inds", sei, int_c);
                auto e0 = vtemp.pack(dim_c<3>, "xn", line[0]);
                auto e1 = vtemp.pack(dim_c<3>, "xn", line[1]);
                auto bv = bv_t{get_bounding_box(e0, e1)};
                bool allFixed = vtemp("BCorder", line[0]) == 3 && vtemp("BCorder", line[1]) == 3;
                bool edgeIntersected = false;
                bvh.iter_neighbors(bv, [&](int sti) {
                    auto tri = coTris.pack(dim_c<3>, "inds", sti, int_c) + voffset;
                    // no need to check common vertices here
                    if (allFixed && vtemp("BCorder", tri[0]) == 3 && vtemp("BCorder", tri[1]) == 3 &&
                        vtemp("BCorder", tri[2]) == 3)
                        return;
                    if (et_intersected(e0, e1, vtemp.pack(dim_c<3>, "xn", tri[0]), vtemp.pack(dim_c<3>, "xn", tri[1]),
                                       vtemp.pack(dim_c<3>, "xn", tri[2]))) {
                        edgeIntersected = true;
                        exclBouSts[sti] = 1;

                        atomic_add(exec_cuda, &cnt[0], 1);
                    }
                });
                if (edgeIntersected)
                    exclSes[sei] = 1;
            });
        zeno::log_info("{} boundary et intersections\n", cnt.getVal());
    }
    return;
}

void UnifiedIPCSystem::findCollisionConstraints(zs::CudaExecutionPolicy &pol, T dHat, T xi) {
    PP.reset();
    PE.reset();
    PT.reset();
    if (enableContactEE) {
        EE.reset();
        if (enableMollification) {
            PPM.reset();
            PEM.reset();
            EEM.reset();
        }
    }
    csPT.reset();
    csEE.reset();

#if PROFILE_IPC
    zs::CppTimer timer;
    timer.tick();
#endif
    if (enableContactSelf) {
        bvs.resize(stInds.size());
        retrieve_bounding_volumes(pol, vtemp, "xn", stInds, zs::wrapv<3>{}, 0, bvs);
        stBvh.refit(pol, bvs);
        bvs.resize(seInds.size());
        retrieve_bounding_volumes(pol, vtemp, "xn", seInds, zs::wrapv<2>{}, 0, bvs);
        seBvh.refit(pol, bvs);
        findCollisionConstraintsImpl(pol, dHat, xi, false);
    }

    if (hasBoundary()) {
        bvs.resize(coEles->size());
        retrieve_bounding_volumes(pol, vtemp, "xn", *coEles, zs::wrapv<3>{}, coOffset, bvs);
        bouStBvh.refit(pol, bvs);
        bvs.resize(coEdges->size());
        retrieve_bounding_volumes(pol, vtemp, "xn", *coEdges, zs::wrapv<2>{}, coOffset, bvs);
        bouSeBvh.refit(pol, bvs);
        findCollisionConstraintsImpl(pol, dHat, xi, true);

        /// @note assume stBvh is already updated
        if (!enableContactSelf) {
            bvs.resize(stInds.size());
            retrieve_bounding_volumes(pol, vtemp, "xn", stInds, zs::wrapv<3>{}, 0, bvs);
            stBvh.refit(pol, bvs);
        }
        findBoundaryCollisionConstraintsImpl(pol, dHat, xi);
    }
    auto [npt, nee] = getCollisionCnts();
#if PROFILE_IPC
    timer.tock(fmt::format("dcd broad phase [pt, ee]({}, {})", npt, nee));
#else
    fmt::print(fg(fmt::color::light_golden_rod_yellow), "dcd broad phase [pt, ee]({}, {})\n", npt, nee);
#endif
}
void UnifiedIPCSystem::findBoundaryCollisionConstraintsImpl(zs::CudaExecutionPolicy &pol, T dHat, T xi) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    pol.profile(PROFILE_IPC);
    /// pt
    snapshot(PP, PE, PT, csPT);
    do {
        pol(Collapse{numBouDofs},
            [eles = proxy<space>({}, stInds), vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(stBvh),
             PP = PP.port(), PE = PE.port(), PT = PT.port(), csPT = csPT.port(), dHat2 = zs::sqr(dHat + xi),
             thickness = xi + dHat, coOffset = coOffset] __device__(int i) mutable {
                auto vi = coOffset + i;
                auto p = vtemp.pack(dim_c<3>, "xn", vi);
                auto bv = bv_t{get_bounding_box(p - thickness, p + thickness)};
                auto f = [&](int stI) {
                    auto tri = eles.pack(dim_c<3>, "inds", stI, int_c);
                    // all affected by sticky boundary conditions
                    if (vtemp("BCorder", tri[0]) == 3 && vtemp("BCorder", tri[1]) == 3 && vtemp("BCorder", tri[2]) == 3)
                        return;
                    // ccd
                    auto t0 = vtemp.pack(dim_c<3>, "xn", tri[0]);
                    auto t1 = vtemp.pack(dim_c<3>, "xn", tri[1]);
                    auto t2 = vtemp.pack(dim_c<3>, "xn", tri[2]);

                    switch (pt_distance_type(p, t0, t1, t2)) {
                    case 0: {
                        if (auto d2 = dist2_pp(p, t0); d2 < dHat2) {
                            PP.try_push(pair_t{vi, tri[0]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 1: {
                        if (auto d2 = dist2_pp(p, t1); d2 < dHat2) {
                            PP.try_push(pair_t{vi, tri[1]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 2: {
                        if (auto d2 = dist2_pp(p, t2); d2 < dHat2) {
                            PP.try_push(pair_t{vi, tri[2]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 3: {
                        if (auto d2 = dist2_pe(p, t0, t1); d2 < dHat2) {
                            PE.try_push(pair3_t{vi, tri[0], tri[1]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 4: {
                        if (auto d2 = dist2_pe(p, t1, t2); d2 < dHat2) {
                            PE.try_push(pair3_t{vi, tri[1], tri[2]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 5: {
                        if (auto d2 = dist2_pe(p, t2, t0); d2 < dHat2) {
                            PE.try_push(pair3_t{vi, tri[2], tri[0]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 6: {
                        if (auto d2 = dist2_pt(p, t0, t1, t2); d2 < dHat2) {
                            PT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    default: break;
                    }
                };
                bvh.iter_neighbors(bv, f);
            });
        if (allFit(PP, PE, PT, csPT))
            break;
        resizeAndRewind(PP, PE, PT, csPT);
    } while (true);
    pol.profile(false);
}
void UnifiedIPCSystem::findCollisionConstraintsImpl(zs::CudaExecutionPolicy &pol, T dHat, T xi, bool withBoundary) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    pol.profile(PROFILE_IPC);
    /// pt
    const auto &stbvh = withBoundary ? bouStBvh : stBvh;
    snapshot(PP, PE, PT, csPT);
    do {
        pol(range(svInds, "inds", dim_c<1>, int_c),
            [eles = proxy<space>({}, withBoundary ? *coEles : stInds),
             exclTris = withBoundary ? proxy<space>(exclBouSts) : proxy<space>(exclSts),
             vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(stbvh), PP = PP.port(), PE = PE.port(), PT = PT.port(),
             csPT = csPT.port(), dHat, xi, thickness = xi + dHat,
             voffset = withBoundary ? coOffset : 0] __device__(int vi) mutable {
                // auto vi = front.prim(i);
                // vi = svInds("inds", vi, int_c);
                const auto dHat2 = zs::sqr(dHat + xi);
                int BCorder0 = vtemp("BCorder", vi);
                auto p = vtemp.pack(dim_c<3>, "xn", vi);
                auto bv = bv_t{get_bounding_box(p - thickness, p + thickness)};
                auto f = [&](int stI) {
                    if (exclTris[stI])
                        return;
                    auto tri = eles.pack(dim_c<3>, "inds", stI, int_c) + voffset;
                    if (vi == tri[0] || vi == tri[1] || vi == tri[2])
                        return;
                    // all affected by sticky boundary conditions
                    if (BCorder0 == 3 && vtemp("BCorder", tri[0]) == 3 && vtemp("BCorder", tri[1]) == 3 &&
                        vtemp("BCorder", tri[2]) == 3)
                        return;
                    // ccd
                    auto t0 = vtemp.pack(dim_c<3>, "xn", tri[0]);
                    auto t1 = vtemp.pack(dim_c<3>, "xn", tri[1]);
                    auto t2 = vtemp.pack(dim_c<3>, "xn", tri[2]);

                    switch (pt_distance_type(p, t0, t1, t2)) {
                    case 0: {
                        if (auto d2 = dist2_pp(p, t0); d2 < dHat2) {
                            PP.try_push(pair_t{vi, tri[0]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 1: {
                        if (auto d2 = dist2_pp(p, t1); d2 < dHat2) {
                            PP.try_push(pair_t{vi, tri[1]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 2: {
                        if (auto d2 = dist2_pp(p, t2); d2 < dHat2) {
                            PP.try_push(pair_t{vi, tri[2]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 3: {
                        if (auto d2 = dist2_pe(p, t0, t1); d2 < dHat2) {
                            PE.try_push(pair3_t{vi, tri[0], tri[1]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 4: {
                        if (auto d2 = dist2_pe(p, t1, t2); d2 < dHat2) {
                            PE.try_push(pair3_t{vi, tri[1], tri[2]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 5: {
                        if (auto d2 = dist2_pe(p, t2, t0); d2 < dHat2) {
                            PE.try_push(pair3_t{vi, tri[2], tri[0]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    case 6: {
                        if (auto d2 = dist2_pt(p, t0, t1, t2); d2 < dHat2) {
                            PT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                            csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        }
                        break;
                    }
                    default: break;
                    }
                };
                bvh.iter_neighbors(bv, f);
            });
        if (allFit(PP, PE, PT, csPT))
            break;
        resizeAndRewind(PP, PE, PT, csPT);
    } while (true);
    /// ee
    if (enableContactEE) {
        const auto &sebvh = withBoundary ? bouSeBvh : seBvh;
        snapshot(PP, PE, EE, PPM, PEM, EEM, csEE);
        do {
            pol(Collapse{seInds.size()},
                [seInds = proxy<space>({}, seInds), sedges = proxy<space>({}, withBoundary ? *coEdges : seInds),
                 exclSes = proxy<space>(exclSes), vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(sebvh),
                 PP = PP.port(), PE = PE.port(), EE = EE.port(),
                 // mollifier
                 PPM = PPM.port(), PEM = PEM.port(), EEM = EEM.port(), enableMollification = enableMollification,
                 //
                 csEE = csEE.port(), dHat2 = zs::sqr(dHat + xi), xi, thickness = xi + dHat,
                 voffset = withBoundary ? coOffset : 0] __device__(int sei) mutable {
                    if (exclSes[sei])
                        return;
                    auto eiInds = seInds.pack(dim_c<2>, "inds", sei, int_c);

                    bool selfFixed = vtemp("BCorder", eiInds[0]) == 3 && vtemp("BCorder", eiInds[1]) == 3;
                    auto v0 = vtemp.pack(dim_c<3>, "xn", eiInds[0]);
                    auto v1 = vtemp.pack(dim_c<3>, "xn", eiInds[1]);
                    auto rv0 = vtemp.pack(dim_c<3>, "x0", eiInds[0]);
                    auto rv1 = vtemp.pack(dim_c<3>, "x0", eiInds[1]);
                    auto [mi, ma] = get_bounding_box(v0, v1);
                    auto bv = bv_t{mi - thickness, ma + thickness};
                    auto f = [&](int sej) {
                        if (voffset == 0 && sei < sej)
                            return;
                        auto ejInds = sedges.pack(dim_c<2>, "inds", sej, int_c) + voffset;
                        if (eiInds[0] == ejInds[0] || eiInds[0] == ejInds[1] || eiInds[1] == ejInds[0] ||
                            eiInds[1] == ejInds[1])
                            return;
                        // all affected by sticky boundary conditions
                        if (selfFixed && vtemp("BCorder", ejInds[0]) == 3 && vtemp("BCorder", ejInds[1]) == 3)
                            return;
                        auto v2 = vtemp.pack(dim_c<3>, "xn", ejInds[0]);
                        auto v3 = vtemp.pack(dim_c<3>, "xn", ejInds[1]);
                        auto rv2 = vtemp.pack(dim_c<3>, "x0", ejInds[0]);
                        auto rv3 = vtemp.pack(dim_c<3>, "x0", ejInds[1]);

                        bool mollify = false;
                        if (enableMollification) {
                            // IPC (24)
                            T c = cn2_ee(v0, v1, v2, v3);
                            T epsX = mollifier_threshold_ee(rv0, rv1, rv2, rv3);
                            mollify = c < epsX;
                        }

                        switch (ee_distance_type(v0, v1, v2, v3)) {
                        case 0: {
                            if (auto d2 = dist2_pp(v0, v2); d2 < dHat2) {
                                csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                if (mollify) {
                                    PPM.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                    break;
                                }
                                PP.try_push(pair_t{eiInds[0], ejInds[0]});
                            }
                            break;
                        }
                        case 1: {
                            if (auto d2 = dist2_pp(v0, v3); d2 < dHat2) {
                                csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                if (mollify) {
                                    PPM.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[1], ejInds[0]});
                                    break;
                                }
                                PP.try_push(pair_t{eiInds[0], ejInds[1]});
                            }
                            break;
                        }
                        case 2: {
                            if (auto d2 = dist2_pe(v0, v2, v3); d2 < dHat2) {
                                csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                if (mollify) {
                                    PEM.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                    break;
                                }
                                PE.try_push(pair3_t{eiInds[0], ejInds[0], ejInds[1]});
                            }
                            break;
                        }
                        case 3: {
                            if (auto d2 = dist2_pp(v1, v2); d2 < dHat2) {
                                csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                if (mollify) {
                                    PPM.try_push(pair4_t{eiInds[1], eiInds[0], ejInds[0], ejInds[1]});
                                    break;
                                }
                                PP.try_push(pair_t{eiInds[1], ejInds[0]});
                            }
                            break;
                        }
                        case 4: {
                            if (auto d2 = dist2_pp(v1, v3); d2 < dHat2) {
                                csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                if (mollify) {
                                    PPM.try_push(pair4_t{eiInds[1], eiInds[0], ejInds[1], ejInds[0]});
                                    break;
                                }
                                PP.try_push(pair_t{eiInds[1], ejInds[1]});
                            }
                            break;
                        }
                        case 5: {
                            if (auto d2 = dist2_pe(v1, v2, v3); d2 < dHat2) {
                                csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                if (mollify) {
                                    PEM.try_push(pair4_t{eiInds[1], eiInds[0], ejInds[0], ejInds[1]});
                                    break;
                                }
                                PE.try_push(pair3_t{eiInds[1], ejInds[0], ejInds[1]});
                            }
                            break;
                        }
                        case 6: {
                            if (auto d2 = dist2_pe(v2, v0, v1); d2 < dHat2) {
                                csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                if (mollify) {
                                    PEM.try_push(pair4_t{ejInds[0], ejInds[1], eiInds[0], eiInds[1]});
                                    break;
                                }
                                PE.try_push(pair3_t{ejInds[0], eiInds[0], eiInds[1]});
                            }
                            break;
                        }
                        case 7: {
                            if (auto d2 = dist2_pe(v3, v0, v1); d2 < dHat2) {
                                csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                if (mollify) {
                                    PEM.try_push(pair4_t{ejInds[1], ejInds[0], eiInds[0], eiInds[1]});
                                    break;
                                }
                                PE.try_push(pair3_t{ejInds[1], eiInds[0], eiInds[1]});
                            }
                            break;
                        }
                        case 8: {
                            if (auto d2 = dist2_ee(v0, v1, v2, v3); d2 < dHat2) {
                                csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                if (mollify) {
                                    EEM.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                                    break;
                                }
                                EE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                            }
                            break;
                        }
                        default: break;
                        }
                    };
                    bvh.iter_neighbors(bv, f);
                });
            if (allFit(PP, PE, EE, PPM, PEM, EEM, csEE))
                break;
            resizeAndRewind(PP, PE, EE, PPM, PEM, EEM, csEE);
        } while (true);
    }
    pol.profile(false);
}
void UnifiedIPCSystem::findProximityPairs(zs::CudaExecutionPolicy &pol, T dHat, T xi, bool withBoundary) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    /// pt
    const auto &stbvh = withBoundary ? bouStBvh : stBvh;
    snapshot(csPT);
    do {
        pol(range(svInds, "inds", dim_c<1>, int_c),
            [eles = proxy<space>({}, withBoundary ? *coEles : stInds),
             exclTris = withBoundary ? proxy<space>(exclBouSts) : proxy<space>(exclSts),
             vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(stbvh), csPT = csPT.port(), dHat, xi,
             thickness = xi + dHat, voffset = withBoundary ? coOffset : 0] __device__(int vi) mutable {
                const auto dHat2 = zs::sqr(dHat + xi);
                int BCorder0 = vtemp("BCorder", vi);
                auto p = vtemp.pack(dim_c<3>, "xn", vi);
                auto bv = bv_t{get_bounding_box(p - thickness, p + thickness)};
                auto f = [&](int stI) {
                    auto tri = eles.pack(dim_c<3>, "inds", stI, int_c) + voffset;
                    if (vi == tri[0] || vi == tri[1] || vi == tri[2])
                        return;
                    // all affected by sticky boundary conditions
                    if (BCorder0 == 3 && vtemp("BCorder", tri[0]) == 3 && vtemp("BCorder", tri[1]) == 3 &&
                        vtemp("BCorder", tri[2]) == 3)
                        return;
                    // ccd
                    auto t0 = vtemp.pack(dim_c<3>, "xn", tri[0]);
                    auto t1 = vtemp.pack(dim_c<3>, "xn", tri[1]);
                    auto t2 = vtemp.pack(dim_c<3>, "xn", tri[2]);

                    if (auto d2 = dist_pt_sqr(p, t0, t1, t2); d2 < dHat2) {
                        csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                        exclTris[stI] = 1;
                    }
                };
                bvh.iter_neighbors(bv, f);
            });
        if (allFit(csPT))
            break;
        resizeAndRewind(csPT);
    } while (true);
    /// ee
    if (enableContactEE) {
        const auto &sebvh = withBoundary ? bouSeBvh : seBvh;
        snapshot(csEE);
        do {
            pol(Collapse{seInds.size()},
                [seInds = proxy<space>({}, seInds), sedges = proxy<space>({}, withBoundary ? *coEdges : seInds),
                 exclSes = proxy<space>(exclSes),
                 oExclSes = withBoundary ? proxy<space>(exclBouSes) : proxy<space>(exclSes),
                 vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(sebvh),
                 //
                 csEE = csEE.port(), dHat2 = zs::sqr(dHat + xi), xi, thickness = xi + dHat,
                 voffset = withBoundary ? coOffset : 0] __device__(int sei) mutable {
                    auto eiInds = seInds.pack(dim_c<2>, "inds", sei, int_c);
                    bool selfFixed = vtemp("BCorder", eiInds[0]) == 3 && vtemp("BCorder", eiInds[1]) == 3;
                    auto v0 = vtemp.pack(dim_c<3>, "xn", eiInds[0]);
                    auto v1 = vtemp.pack(dim_c<3>, "xn", eiInds[1]);
                    auto [mi, ma] = get_bounding_box(v0, v1);
                    auto bv = bv_t{mi - thickness, ma + thickness};
                    auto f = [&](int sej) {
                        if (voffset == 0 && sei < sej)
                            return;
                        auto ejInds = sedges.pack(dim_c<2>, "inds", sej, int_c) + voffset;
                        if (eiInds[0] == ejInds[0] || eiInds[0] == ejInds[1] || eiInds[1] == ejInds[0] ||
                            eiInds[1] == ejInds[1])
                            return;
                        // all affected by sticky boundary conditions
                        if (selfFixed && vtemp("BCorder", ejInds[0]) == 3 && vtemp("BCorder", ejInds[1]) == 3)
                            return;
                        auto v2 = vtemp.pack(dim_c<3>, "xn", ejInds[0]);
                        auto v3 = vtemp.pack(dim_c<3>, "xn", ejInds[1]);

                        if (auto d2 = dist_ee_sqr(v0, v1, v2, v3); d2 < dHat2) {
                            csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                            exclSes[sei] = 1;
                            oExclSes[sej] = 1;
                        }
                    };
                    bvh.iter_neighbors(bv, f);
                });
            if (allFit(csEE))
                break;
            resizeAndRewind(csEE);
        } while (true);
    }
}
void UnifiedIPCSystem::findCCDConstraints(zs::CudaExecutionPolicy &pol, T alpha, T xi) {
    csPT.reset();
    csEE.reset();

    if (enableContactSelf) {
        bvs.resize(stInds.size());
        retrieve_bounding_volumes(pol, vtemp, "xn", stInds, zs::wrapv<3>{}, vtemp, "dir", alpha, 0, bvs);
        stBvh.refit(pol, bvs);
        bvs.resize(seInds.size());
        retrieve_bounding_volumes(pol, vtemp, "xn", seInds, zs::wrapv<2>{}, vtemp, "dir", alpha, 0, bvs);
        seBvh.refit(pol, bvs);

        findCCDConstraintsImpl(pol, alpha, xi, false);
    }

#if PROFILE_IPC
    zs::CppTimer timer;
    timer.tick();
#endif

    if (hasBoundary()) {
        bvs.resize(coEles->size());
        retrieve_bounding_volumes(pol, vtemp, "xn", *coEles, zs::wrapv<3>{}, vtemp, "dir", alpha, coOffset, bvs);
        bouStBvh.refit(pol, bvs);

        bvs.resize(coEdges->size());
        retrieve_bounding_volumes(pol, vtemp, "xn", *coEdges, zs::wrapv<2>{}, vtemp, "dir", alpha, coOffset, bvs);
        bouSeBvh.refit(pol, bvs);

        findCCDConstraintsImpl(pol, alpha, xi, true);

        /// @note assume stBvh is already updated
        if (!enableContactSelf) {
            bvs.resize(stInds.size());
            retrieve_bounding_volumes(pol, vtemp, "xn", stInds, zs::wrapv<3>{}, vtemp, "dir", alpha, 0, bvs);
            stBvh.refit(pol, bvs);
        }
        findBoundaryCCDConstraintsImpl(pol, alpha, xi);
    }
    auto [npt, nee] = getCollisionCnts();
#if PROFILE_IPC
    timer.tock(fmt::format("ccd broad phase [pt, ee]({}, {})", npt, nee));
#else
    fmt::print(fg(fmt::color::light_golden_rod_yellow), "ccd broad phase [pt, ee]({}, {})\n", npt, nee);
#endif
}
void UnifiedIPCSystem::findBoundaryCCDConstraintsImpl(zs::CudaExecutionPolicy &pol, T alpha, T xi) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    const auto dHat2 = dHat * dHat;

    pol.profile(PROFILE_IPC);
    /// pt
    snapshot(csPT);
    do {
        pol(Collapse{numBouDofs},
            [eles = proxy<space>({}, stInds), vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(stBvh),
             csPT = csPT.port(), xi, alpha, coOffset = coOffset] __device__(int i) mutable {
                auto vi = coOffset + i;
                auto p = vtemp.pack(dim_c<3>, "xn", vi);
                auto dir = vtemp.pack(dim_c<3>, "dir", vi);
                auto bv = bv_t{get_bounding_box(p, p + alpha * dir)};
                bv._min -= xi;
                bv._max += xi;
                bvh.iter_neighbors(bv, [&](int stI) {
                    auto tri = eles.pack(dim_c<3>, "inds", stI, int_c);
                    if (vtemp("BCorder", tri[0]) == 3 && vtemp("BCorder", tri[1]) == 3 && vtemp("BCorder", tri[2]) == 3)
                        return;
                    csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                });
            });
        if (allFit(csPT))
            break;
        resizeAndRewind(csPT);
    } while (true);
    pol.profile(false);
}
void UnifiedIPCSystem::findCCDConstraintsImpl(zs::CudaExecutionPolicy &pol, T alpha, T xi, bool withBoundary) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    const auto dHat2 = dHat * dHat;

    pol.profile(PROFILE_IPC);
    /// pt
    const auto &stbvh = withBoundary ? bouStBvh : stBvh;
    snapshot(csPT);
    do {
        pol(range(svInds, "inds", dim_c<1>, int_c),
            [svInds = proxy<space>({}, svInds), eles = proxy<space>({}, withBoundary ? *coEles : stInds),
             vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(stbvh), csPT = csPT.port(), xi, alpha,
             voffset = withBoundary ? coOffset : 0] __device__(int vi) mutable {
                auto p = vtemp.pack(dim_c<3>, "xn", vi);
                auto dir = vtemp.pack(dim_c<3>, "dir", vi);
                auto bv = bv_t{get_bounding_box(p, p + alpha * dir)};
                bv._min -= xi;
                bv._max += xi;
                bvh.iter_neighbors(bv, [&](int stI) {
                    auto tri = eles.pack(dim_c<3>, "inds", stI, int_c) + voffset;
                    if (vi == tri[0] || vi == tri[1] || vi == tri[2])
                        return;
                    // all affected by sticky boundary conditions
                    if (vtemp("BCorder", vi) == 3 && vtemp("BCorder", tri[0]) == 3 && vtemp("BCorder", tri[1]) == 3 &&
                        vtemp("BCorder", tri[2]) == 3)
                        return;
                    csPT.try_push(pair4_t{vi, tri[0], tri[1], tri[2]});
                });
            });
        if (allFit(csPT))
            break;
        resizeAndRewind(csPT);
    } while (true);
    /// ee
    if (enableContactEE) {
        const auto &sebvh = withBoundary ? bouSeBvh : seBvh;
        snapshot(csEE);
        do {
            pol(Collapse{seInds.size()},
                [seInds = proxy<space>({}, seInds), sedges = proxy<space>({}, withBoundary ? *coEdges : seInds),
                 vtemp = proxy<space>({}, vtemp), bvh = proxy<space>(sebvh), csEE = csEE.port(), xi, alpha,
                 voffset = withBoundary ? coOffset : 0] __device__(int sei) mutable {
                    auto eiInds = seInds.pack(dim_c<2>, "inds", sei, int_c);
                    bool selfFixed = vtemp("BCorder", eiInds[0]) == 3 && vtemp("BCorder", eiInds[1]) == 3;
                    auto v0 = vtemp.pack(dim_c<3>, "xn", eiInds[0]);
                    auto v1 = vtemp.pack(dim_c<3>, "xn", eiInds[1]);
                    auto dir0 = vtemp.pack(dim_c<3>, "dir", eiInds[0]);
                    auto dir1 = vtemp.pack(dim_c<3>, "dir", eiInds[1]);
                    auto bv = bv_t{get_bounding_box(v0, v0 + alpha * dir0)};
                    merge(bv, v1);
                    merge(bv, v1 + alpha * dir1);
                    bv._min -= xi;
                    bv._max += xi;
                    bvh.iter_neighbors(bv, [&](int sej) {
                        if (voffset == 0 && sei < sej)
                            return;
                        auto ejInds = sedges.pack(dim_c<2>, "inds", sej, int_c) + voffset;
                        if (eiInds[0] == ejInds[0] || eiInds[0] == ejInds[1] || eiInds[1] == ejInds[0] ||
                            eiInds[1] == ejInds[1])
                            return;
                        // all affected by sticky boundary conditions
                        if (selfFixed && vtemp("BCorder", ejInds[0]) == 3 && vtemp("BCorder", ejInds[1]) == 3)
                            return;
                        csEE.try_push(pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]});
                    });
                });
            if (allFit(csEE))
                break;
            resizeAndRewind(csEE);
        } while (true);
    }
    pol.profile(false);
}
void UnifiedIPCSystem::precomputeFrictions(zs::CudaExecutionPolicy &pol, T dHat, T xi) {
    using namespace zs;

    if (!needFricPrecompute)
        return;
    needFricPrecompute = false;

    constexpr auto space = execspace_e::cuda;
    T activeGap2 = dHat * dHat + (T)2.0 * xi * dHat;
    FPP.reset();
    FPE.reset();
    FPT.reset();
    FEE.reset();
    if (enableContact) {
        if (s_enableSelfFriction) {
            FPP.assignCounterFrom(PP);
            FPE.assignCounterFrom(PE);
            FPT.assignCounterFrom(PT);
            FEE.assignCounterFrom(EE);

            auto numFPP = FPP.getCount();
            fricPP.resize(numFPP);
            pol(range(numFPP),
                [vtemp = proxy<space>({}, vtemp), fricPP = proxy<space>({}, fricPP), PP = PP.port(), FPP = FPP.port(),
                 xi2 = xi * xi, activeGap2, kappa = kappa] __device__(int fppi) mutable {
                    auto fpp = PP[fppi];
                    FPP[fppi] = fpp;
                    auto x0 = vtemp.pack<3>("xn", fpp[0]);
                    auto x1 = vtemp.pack<3>("xn", fpp[1]);
                    auto dist2 = dist2_pp(x0, x1);
                    auto bGrad = barrier_gradient(dist2 - xi2, activeGap2, kappa);
                    fricPP("fn", fppi) = -bGrad * 2 * zs::sqrt(dist2);
                    fricPP.tuple<6>("basis", fppi) = point_point_tangent_basis(x0, x1);
                });
            auto numFPE = FPE.getCount();
            fricPE.resize(numFPE);
            pol(range(numFPE),
                [vtemp = proxy<space>({}, vtemp), fricPE = proxy<space>({}, fricPE), PE = PE.port(), FPE = FPE.port(),
                 xi2 = xi * xi, activeGap2, kappa = kappa] __device__(int fpei) mutable {
                    auto fpe = PE[fpei];
                    FPE[fpei] = fpe;
                    auto p = vtemp.pack<3>("xn", fpe[0]);
                    auto e0 = vtemp.pack<3>("xn", fpe[1]);
                    auto e1 = vtemp.pack<3>("xn", fpe[2]);
                    auto dist2 = dist2_pe(p, e0, e1);
                    auto bGrad = barrier_gradient(dist2 - xi2, activeGap2, kappa);
                    fricPE("fn", fpei) = -bGrad * 2 * zs::sqrt(dist2);
                    fricPE("yita", fpei) = point_edge_closest_point(p, e0, e1);
                    fricPE.tuple<6>("basis", fpei) = point_edge_tangent_basis(p, e0, e1);
                });
            auto numFPT = FPT.getCount();
            fricPT.resize(numFPT);
            pol(range(numFPT),
                [vtemp = proxy<space>({}, vtemp), fricPT = proxy<space>({}, fricPT), PT = PT.port(), FPT = FPT.port(),
                 xi2 = xi * xi, activeGap2, kappa = kappa] __device__(int fpti) mutable {
                    auto fpt = PT[fpti];
                    FPT[fpti] = fpt;
                    auto p = vtemp.pack<3>("xn", fpt[0]);
                    auto t0 = vtemp.pack<3>("xn", fpt[1]);
                    auto t1 = vtemp.pack<3>("xn", fpt[2]);
                    auto t2 = vtemp.pack<3>("xn", fpt[3]);
                    auto dist2 = dist2_pt(p, t0, t1, t2);
                    auto bGrad = barrier_gradient(dist2 - xi2, activeGap2, kappa);
                    fricPT("fn", fpti) = -bGrad * 2 * zs::sqrt(dist2);
                    fricPT.tuple<2>("beta", fpti) = point_triangle_closest_point(p, t0, t1, t2);
                    fricPT.tuple<6>("basis", fpti) = point_triangle_tangent_basis(p, t0, t1, t2);
                });
            auto numFEE = FEE.getCount();
            fricEE.resize(numFEE);
            pol(range(numFEE),
                [vtemp = proxy<space>({}, vtemp), fricEE = proxy<space>({}, fricEE), EE = EE.port(), FEE = FEE.port(),
                 xi2 = xi * xi, activeGap2, kappa = kappa] __device__(int feei) mutable {
                    auto fee = EE[feei];
                    FEE[feei] = fee;
                    auto ea0 = vtemp.pack<3>("xn", fee[0]);
                    auto ea1 = vtemp.pack<3>("xn", fee[1]);
                    auto eb0 = vtemp.pack<3>("xn", fee[2]);
                    auto eb1 = vtemp.pack<3>("xn", fee[3]);
                    auto dist2 = dist2_ee(ea0, ea1, eb0, eb1);
                    auto bGrad = barrier_gradient(dist2 - xi2, activeGap2, kappa);
                    fricEE("fn", feei) = -bGrad * 2 * zs::sqrt(dist2);
                    fricEE.tuple<2>("gamma", feei) = edge_edge_closest_point(ea0, ea1, eb0, eb1);
                    fricEE.tuple<6>("basis", feei) = edge_edge_tangent_basis(ea0, ea1, eb0, eb1);
                });
        }
    }
    if (enableGround) {
        for (auto &primHandle : prims) {
            if (primHandle.isBoundary()) // skip soft boundary
                continue;
            const auto &svs = primHandle.getSurfVerts();
            pol(range(svs.size()),
                [vtemp = proxy<space>({}, vtemp), svs = proxy<space>({}, svs),
                 svtemp = proxy<space>({}, primHandle.svtemp), kappa = kappa, xi2 = xi * xi, activeGap2,
                 gn = s_groundNormal, svOffset = primHandle.svOffset] ZS_LAMBDA(int svi) mutable {
                    const auto vi = svs("inds", svi, int_c) + svOffset;
                    auto x = vtemp.pack<3>("xn", vi);
                    auto dist = gn.dot(x);
                    auto dist2 = dist * dist;
                    if (dist2 < activeGap2) {
                        auto bGrad = barrier_gradient(dist2 - xi2, activeGap2, kappa);
                        svtemp("fn", svi) = -bGrad * 2 * dist;
                    } else
                        svtemp("fn", svi) = 0;
                });
        }
    }
}

void UnifiedIPCSystem::project(zs::CudaExecutionPolicy &pol, const zs::SmallString tag) {
    using namespace zs;
    constexpr execspace_e space = execspace_e::cuda;
    // projection
    if (projectDBC)
        pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), tagOffset = vtemp.getPropertyOffset(tag),
                                 orderOffset = vtemp.getPropertyOffset("BCorder")] ZS_LAMBDA(int vi) mutable {
            int BCorder = vtemp(orderOffset, vi);
            for (int d = 0; d != BCorder; ++d)
                vtemp(tagOffset + d, vi) = 0;
        });
    else
        pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), tagOffset = vtemp.getPropertyOffset(tag),
                                 fixedOffset = vtemp.getPropertyOffset("BCfixed"),
                                 orderOffset = vtemp.getPropertyOffset("BCorder")] ZS_LAMBDA(int vi) mutable {
            int BCfixed = vtemp(fixedOffset, vi);
            if (BCfixed) {
                int BCorder = vtemp(orderOffset, vi);
                for (int d = 0; d != BCorder; ++d)
                    vtemp(tagOffset + d, vi) = 0;
            }
        });
}

void UnifiedIPCSystem::precondition(zs::CudaExecutionPolicy &pol, const zs::SmallString srcTag,
                                    const zs::SmallString dstTag) {
    using namespace zs;
    constexpr execspace_e space = execspace_e::cuda;
// precondition
#if 0
    pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), srcTag, dstTag] ZS_LAMBDA(int vi) mutable {
        vtemp.tuple(dim_c<3>, dstTag, vi) = vtemp.pack(dim_c<3, 3>, "P", vi) * vtemp.pack(dim_c<3>, srcTag, vi);
    });
#endif
    pol(zs::range(numDofs * 3), [vtemp = proxy<space>({}, vtemp), srcOffset = vtemp.getPropertyOffset(srcTag),
                                 dstOffset = vtemp.getPropertyOffset(dstTag),
                                 POffset = vtemp.getPropertyOffset("P")] ZS_LAMBDA(int vi) mutable {
        int d = vi % 3;
        vi /= 3;
        float sum = 0;
        POffset += d * 3;
        for (int j = 0; j != 3; ++j)
            sum += vtemp(POffset + j, vi) * vtemp(srcOffset + j, vi);
        vtemp(dstOffset + d, vi) = sum;
    });
}

void UnifiedIPCSystem::systemMultiply(zs::CudaExecutionPolicy &pol, const zs::SmallString dxTag,
                                      const zs::SmallString bTag) {
    using namespace zs;
    constexpr execspace_e space = execspace_e::cuda;
    constexpr auto execTag = wrapv<space>{};
    using T = typename RM_CVREF_T(linsys)::T;
    using vec3 = zs::vec<T, 3>;
    // dx -> b
    pol(range(numDofs), [execTag, vtemp = proxy<space>({}, vtemp), bTag] ZS_LAMBDA(int vi) mutable {
        vtemp.tuple(dim_c<3>, bTag, vi) = vec3::zeros();
    });
    // CppTimer timer;
    // timer.tick();
    {
        const auto &spmat = linsys.spmat;
        /// upper part (with diagonal)
        pol(range(spmat.outerSize() * 32),
            [vtemp = view<space>(vtemp), dxOffset = vtemp.getPropertyOffset(dxTag),
             bOffset = vtemp.getPropertyOffset(bTag), spmat = proxy<space>(spmat)] ZS_LAMBDA(int tid) mutable {
                auto tile = zs::cg::tiled_partition<32>(zs::cg::this_thread_block());
                auto row = tid / tile.num_threads();
                auto bg = spmat._ptrs[row];
                auto ed = spmat._ptrs[row + 1];
                auto sum = vec3::zeros();
                for (auto i = bg + tile.thread_rank(); i < ed; i += tile.num_threads())
                    sum += spmat._vals[i] * vtemp.pack(dim_c<3>, dxOffset, spmat._inds[i]);
                T sumx = zs::cg::reduce(tile, sum[0], zs::cg::plus<T>());
                T sumy = zs::cg::reduce(tile, sum[1], zs::cg::plus<T>());
                T sumz = zs::cg::reduce(tile, sum[2], zs::cg::plus<T>());
                if (tile.thread_rank() == 0)
                    vtemp.tuple(dim_c<3>, bOffset, row) = vtemp.pack(dim_c<3>, bOffset, row) + vec3f{sumx, sumy, sumz};
            });

        /// lower part (without diagonal)
        pol(range(spmat.outerSize() * 32),
            [vtemp = view<space>(vtemp), dxOffset = vtemp.getPropertyOffset(dxTag),
             bOffset = vtemp.getPropertyOffset(bTag), spmat = proxy<space>(spmat)] ZS_LAMBDA(int tid) mutable {
                auto tile = zs::cg::tiled_partition<32>(zs::cg::this_thread_block());
                auto col = tid / tile.num_threads();
                auto bg = spmat._ptrs[col] + 1; // skip the diagonal part
                auto ed = spmat._ptrs[col + 1];

                auto dx = vtemp.pack(dim_c<3>, dxOffset, col);
                for (auto k = bg + tile.thread_rank(); k < ed; k += tile.num_threads()) {
                    auto row = spmat._inds[k];
                    auto inc = spmat._vals[k].transpose() * dx;
                    for (int d = 0; d != 3; ++d)
                        atomic_add(exec_cuda, &vtemp(bOffset + d, row), inc(d));
                }
            });
    }
#if 0
    // hess1
    pol(zs::range(numDofs), [execTag, hess1 = proxy<space>(hess1), cgtemp = proxy<space>({}, cgtemp),
                             dxOffset = cgtemp.getPropertyOffset(dxTag),
                             bOffset = cgtemp.getPropertyOffset(bTag)] __device__(int i) mutable {
        auto H = hess1.hess[i];
        zs::vec<float, 3> dx{cgtemp(dxOffset, i), cgtemp(dxOffset + 1, i), cgtemp(dxOffset + 2, i)};
        // auto dx = cgtemp.pack(dim_c<3>, dxTag, i);
        dx = H * dx;
        for (int d = 0; d != 3; ++d)
            atomic_add(execTag, &cgtemp(bOffset + d, i), dx(d));
    });
#endif

    // hess2
    const auto &hess2 = linsys.hess2;
    const auto &hess3 = linsys.hess3;
    const auto &hess4 = linsys.hess4;
    pol(Collapse{hess2.count(), 32},
        [execTag, hess2 = proxy<space>(hess2), vtemp = proxy<space>(vtemp), dxOffset = vtemp.getPropertyOffset(dxTag),
         bOffset = vtemp.getPropertyOffset(bTag)] ZS_LAMBDA(int ei, int tid) mutable {
            int rowid = tid / 5;
            int colid = tid % 5;
            auto inds = hess2.inds[ei];
            auto H = hess2.hess[ei];
            T entryH = 0, entryDx = 0, entryG = 0;
            if (tid < 30) {
                entryH = H.val(rowid * 6 + colid);
                entryDx = vtemp(dxOffset + colid % 3, inds[colid / 3]);
                entryG = entryH * entryDx;
                if (colid == 0) {
                    entryG += H.val(rowid * 6 + 5) * vtemp(dxOffset + 2, inds[1]);
                }
            }
            for (int iter = 1; iter <= 4; iter <<= 1) {
                T tmp = __shfl_down_sync(0xFFFFFFFF, entryG, iter);
                if (colid + iter < 5 && tid < 30)
                    entryG += tmp;
            }
            if (colid == 0 && rowid < 6)
                atomic_add(execTag, &vtemp(bOffset + rowid % 3, inds[rowid / 3]), entryG);
        });
    // hess3
    {
        auto numRows = hess3.count() * 9;
        auto numWarps = (numRows + 3) / 4; // 8 threads per row
        pol(Collapse{numWarps * 32}, [execTag, hess3 = proxy<space>(hess3), vtemp = proxy<space>({}, vtemp),
                                      dxOffset = vtemp.getPropertyOffset(dxTag),
                                      bOffset = vtemp.getPropertyOffset(bTag), numRows] ZS_LAMBDA(int tid) mutable {
            int growid = tid / 8;
            int rowid = growid % 9;
            int i = growid / 9;
            int colid = tid % 8;

            auto inds = hess3.inds[i];
            auto H = hess3.hess[i];
            T entryG = 0;
            if (growid < numRows) {
                entryG = H.val(rowid * 9 + colid) * vtemp(dxOffset + colid % 3, inds[colid / 3]);
                if (colid == 0) {
                    auto cid = colid + 8;
                    entryG += H.val(rowid * 9 + cid) * vtemp(dxOffset + cid % 3, inds[cid / 3]);
                }
            }
            for (int iter = 1; iter <= 4; iter <<= 1) {
                T tmp = __shfl_down_sync(0xFFFFFFFF, entryG, iter);
                if (colid + iter < 8 && growid < numRows)
                    entryG += tmp;
            }
            if (colid == 0 && growid < numRows)
                atomic_add(execTag, &vtemp(bOffset + rowid % 3, inds[rowid / 3]), entryG);
        });
    }
    // hess4
    {
        // 0, 1, ..., 7, 0, 1, 2, 3
        pol(Collapse{hess4.count(), 32 * 3},
            [execTag, hess4 = proxy<space>(hess4), vtemp = proxy<space>({}, vtemp),
             dxOffset = vtemp.getPropertyOffset(dxTag),
             bOffset = vtemp.getPropertyOffset(bTag)] ZS_LAMBDA(int i, int tid) mutable {
                int rowid = tid / 8;
                int colid = tid % 8;

                auto inds = hess4.inds[i];
                auto H = hess4.hess[i];
                T entryH = 0, entryDx = 0, entryG = 0;
                {
                    entryH = H.val(rowid * 12 + colid);
                    entryDx = vtemp(dxOffset + colid % 3, inds[colid / 3]);
                    entryG = entryH * entryDx;
                    if (colid < 4) {
                        auto cid = colid + 8;
                        entryG += H.val(rowid * 12 + cid) * vtemp(dxOffset + cid % 3, inds[cid / 3]);
                    }
                }
                for (int iter = 1; iter <= 4; iter <<= 1) {
                    T tmp = __shfl_down_sync(0xFFFFFFFF, entryG, iter);
                    if (colid + iter < 8)
                        entryG += tmp;
                }
                if (colid == 0)
                    atomic_add(execTag, &vtemp(bOffset + rowid % 3, inds[rowid / 3]), entryG);
            });
    }
    // timer.tock("multiply takes");
}

template <typename Model>
typename UnifiedIPCSystem::T elasticityEnergy(zs::CudaExecutionPolicy &pol, typename UnifiedIPCSystem::dtiles_t &vtemp,
                                              typename UnifiedIPCSystem::PrimitiveHandle &primHandle,
                                              const Model &model, typename UnifiedIPCSystem::T dt,
                                              zs::Vector<typename UnifiedIPCSystem::T> &es) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    using mat3 = typename UnifiedIPCSystem::mat3;
    using vec3 = typename UnifiedIPCSystem::vec3;
    using T = typename UnifiedIPCSystem::T;

    auto &eles = primHandle.getEles();
    es.resize(count_warps(eles.size()));
    es.reset(0);
    const zs::SmallString tag = "xn";
    if (primHandle.category == ZenoParticles::curve) {
        if (primHandle.isBoundary() && !primHandle.isAuxiliary())
            return 0;
        // elasticity
        pol(range(eles.size()),
            [eles = proxy<space>({}, eles), vtemp = proxy<space>({}, vtemp), es = proxy<space>(es), tag, model = model,
             vOffset = primHandle.vOffset, n = eles.size()] __device__(int ei) mutable {
                auto inds = eles.pack(dim_c<2>, "inds", ei, int_c) + vOffset;

                int BCorder[2];
                for (int i = 0; i != 2; ++i)
                    BCorder[i] = vtemp("BCorder", inds[i]);
                T E;
                if (BCorder[0] == 3 && BCorder[1] == 3)
                    E = 0;
                else {
                    auto vole = eles("vol", ei);
                    auto k = eles("k", ei);
                    // auto k = model.mu;
                    auto rl = eles("rl", ei);
                    vec3 xs[2] = {vtemp.pack(dim_c<3>, tag, inds[0]), vtemp.pack(dim_c<3>, tag, inds[1])};
                    auto xij = xs[1] - xs[0];
                    auto lij = xij.norm();

                    E = (T)0.5 * k * zs::sqr(lij - rl) * vole;
                }
                reduce_to(ei, n, E, es[ei / 32]);
            });
        return reduce(pol, es) * dt * dt;
    } else if (primHandle.category == ZenoParticles::surface) {
        if (primHandle.isBoundary())
            return 0;
        // elasticity
        pol(range(eles.size()),
            [eles = proxy<space>({}, eles), vtemp = proxy<space>({}, vtemp), es = proxy<space>(es), tag, model = model,
             vOffset = primHandle.vOffset, n = eles.size()] __device__(int ei) mutable {
                auto IB = eles.template pack<2, 2>("IB", ei);
                auto inds = eles.pack(dim_c<3>, "inds", ei, int_c) + vOffset;

                int BCorder[3];
                for (int i = 0; i != 3; ++i)
                    BCorder[i] = vtemp("BCorder", inds[i]);
                T E;
                if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3)
                    E = 0;
                else {
                    auto vole = eles("vol", ei);
                    vec3 xs[3] = {vtemp.pack(dim_c<3>, tag, inds[0]), vtemp.pack(dim_c<3>, tag, inds[1]),
                                  vtemp.pack(dim_c<3>, tag, inds[2])};
                    auto x1x0 = xs[1] - xs[0];
                    auto x2x0 = xs[2] - xs[0];

                    zs::vec<T, 3, 2> Ds{x1x0[0], x2x0[0], x1x0[1], x2x0[1], x1x0[2], x2x0[2]};
                    auto F = Ds * IB;
                    auto f0 = col(F, 0);
                    auto f1 = col(F, 1);
                    auto f0Norm = zs::sqrt(f0.l2NormSqr());
                    auto f1Norm = zs::sqrt(f1.l2NormSqr());
                    auto Estretch = model.mu * vole * (zs::sqr(f0Norm - 1) + zs::sqr(f1Norm - 1));
                    auto Eshear = (model.mu * 0.3) * vole * zs::sqr(f0.dot(f1));
                    E = Estretch + Eshear;
                }
                reduce_to(ei, n, E, es[ei / 32]);
            });
        return (reduce(pol, es) * dt * dt);
    } else if (primHandle.category == ZenoParticles::tet) {
        pol(zs::range(eles.size()),
            [vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles), es = proxy<space>(es), model, tag,
             vOffset = primHandle.vOffset, n = eles.size()] __device__(int ei) mutable {
                auto IB = eles.pack(dim_c<3, 3>, "IB", ei);
                auto inds = eles.pack(dim_c<4>, "inds", ei, int_c) + vOffset;
                auto vole = eles("vol", ei);
                vec3 xs[4] = {vtemp.pack<3>(tag, inds[0]), vtemp.pack<3>(tag, inds[1]), vtemp.pack<3>(tag, inds[2]),
                              vtemp.pack<3>(tag, inds[3])};

                int BCorder[4];
                for (int i = 0; i != 4; ++i)
                    BCorder[i] = vtemp("BCorder", inds[i]);
                T E;
                if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3 && BCorder[3] == 3)
                    E = 0;
                else {
                    mat3 F{};
                    auto x1x0 = xs[1] - xs[0];
                    auto x2x0 = xs[2] - xs[0];
                    auto x3x0 = xs[3] - xs[0];
                    auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1], x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                    F = Ds * IB;
                    E = model.psi(F) * vole;
                }
                reduce_to(ei, n, E, es[ei / 32]);
            });
        return (reduce(pol, es) * dt * dt);
    }
    return 0;
}

typename UnifiedIPCSystem::T UnifiedIPCSystem::energy(zs::CudaExecutionPolicy &pol, const zs::SmallString tag) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    Vector<T> &es = temp;

    std::vector<T> Es(0);

    // inertial
    es.resize(count_warps(coOffset));
    es.reset(0);
    pol(range(coOffset), [vtemp = proxy<space>({}, vtemp), es = proxy<space>(es), tag, dt = this->dt,
                          n = coOffset] __device__(int vi) mutable {
        auto m = vtemp("ws", vi);
        auto x = vtemp.pack(dim_c<3>, tag, vi);
        auto xt = vtemp.pack(dim_c<3>, "xhat", vi);
        int BCorder = vtemp("BCorder", vi);
        T E = 0;
        if (BCorder == 0) {
            // inertia
            E = (T)0.5 * m * (x - vtemp.pack(dim_c<3>, "xtilde", vi)).l2NormSqr();
        }
        reduce_to(vi, n, E, es[vi / 32]);
    });
    Es.push_back(reduce(pol, es));

    if (vtemp.hasProperty("extf")) {
        es.resize(count_warps(coOffset));
        es.reset(0);
        pol(range(coOffset), [vtemp = proxy<space>({}, vtemp), es = proxy<space>(es), tag, dt = this->dt,
                              n = coOffset] ZS_LAMBDA(int vi) mutable {
            auto x = vtemp.pack<3>(tag, vi);
            auto xt = vtemp.pack<3>("xhat", vi);
            int BCorder = vtemp("BCorder", vi);
            T E = 0;
            {
                // external force
                // if (vtemp("BCsoft", vi) == 0 && vtemp("BCorder", vi) != 3)
                if (BCorder == 0) {
                    auto extf = vtemp.pack(dim_c<3>, "extf", vi);
                    E += -extf.dot(x - xt) * dt * dt;
                }
            }
            reduce_to(vi, n, E, es[vi / 32]);
        });
        Es.push_back(reduce(pol, es));
    }

    for (auto &primHandle : prims) {
        /// @note elasticity
        match([&](auto &elasticModel) {
            Es.push_back(elasticityEnergy(pol, vtemp, primHandle, elasticModel, dt, es));
        })(primHandle.getModels().getElasticModel());

        if (primHandle.hasBendingConstraints()) {
            /// @note bending energy (if exist)
            auto &bedges = *primHandle.bendingEdgesPtr;
            es.resize(count_warps(bedges.size()));
            es.reset(0);
            pol(range(bedges.size()),
                [vtemp = proxy<space>({}, vtemp), es = proxy<space>(es), bedges = proxy<space>({}, bedges), dt = dt,
                 vOffset = primHandle.vOffset, n = bedges.size()] __device__(int i) mutable {
                    auto stcl = bedges.pack(dim_c<4>, "inds", i, int_c) + vOffset;
                    auto x0 = vtemp.pack(dim_c<3>, "xn", stcl[0]);
                    auto x1 = vtemp.pack(dim_c<3>, "xn", stcl[1]);
                    auto x2 = vtemp.pack(dim_c<3>, "xn", stcl[2]);
                    auto x3 = vtemp.pack(dim_c<3>, "xn", stcl[3]);
                    auto e = bedges("e", i);
                    auto h = bedges("h", i);
                    auto k = bedges("k", i);
                    auto ra = bedges("ra", i);
                    auto theta = dihedral_angle(x0, x1, x2, x3);
                    T E = k * zs::sqr(theta - ra) * e / h * dt * dt;
                    reduce_to(i, n, E, es[i / 32]);
                });
            Es.push_back(reduce(pol, es));
        }
    }
    for (auto &primHandle : auxPrims) {
        using ModelT = RM_CVREF_T(primHandle.getModels().getElasticModel());
        const ModelT &model = primHandle.modelsPtr ? primHandle.getModels().getElasticModel() : ModelT{};
        match([&](auto &elasticModel) {
            Es.push_back(elasticityEnergy(pol, vtemp, primHandle, elasticModel, dt, es));
        })(model);
    }
    // contacts
    {
        if (enableContact) {
            auto activeGap2 = dHat * dHat + 2 * xi * dHat;
            auto numPP = PP.getCount();
            es.resize(count_warps(numPP));
            es.reset(0);
            pol(range(numPP), [vtemp = proxy<space>({}, vtemp), PP = PP.port(), es = proxy<space>(es), xi2 = xi * xi,
                               dHat = dHat, activeGap2, n = numPP] __device__(int ppi) mutable {
                auto pp = PP[ppi];
                auto x0 = vtemp.pack<3>("xn", pp[0]);
                auto x1 = vtemp.pack<3>("xn", pp[1]);
                auto dist2 = dist2_pp(x0, x1);
                if (dist2 < xi2)
                    printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                // es[ppi] = zs::barrier(dist2 - xi2, activeGap2, (T)1);

                auto I5 = dist2 / activeGap2;
                auto lenE = (dist2 - activeGap2);
                auto E = -lenE * lenE * zs::log(I5);
                reduce_to(ppi, n, E, es[ppi / 32]);
            });
            Es.push_back(reduce(pol, es) * kappa);

            auto numPE = PE.getCount();
            es.resize(count_warps(numPE));
            es.reset(0);
            pol(range(numPE), [vtemp = proxy<space>({}, vtemp), PE = PE.port(), es = proxy<space>(es), xi2 = xi * xi,
                               dHat = dHat, activeGap2, n = numPE] __device__(int pei) mutable {
                auto pe = PE[pei];
                auto p = vtemp.pack<3>("xn", pe[0]);
                auto e0 = vtemp.pack<3>("xn", pe[1]);
                auto e1 = vtemp.pack<3>("xn", pe[2]);

                auto dist2 = dist2_pe(p, e0, e1);
                if (dist2 < xi2)
                    printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                // es[pei] = zs::barrier(dist2 - xi2, activeGap2, (T)1);

                auto I5 = dist2 / activeGap2;
                auto lenE = (dist2 - activeGap2);
                auto E = -lenE * lenE * zs::log(I5);
                reduce_to(pei, n, E, es[pei / 32]);
            });
            Es.push_back(reduce(pol, es) * kappa);

            auto numPT = PT.getCount();
            es.resize(count_warps(numPT));
            es.reset(0);
            pol(range(numPT), [vtemp = proxy<space>({}, vtemp), PT = PT.port(), es = proxy<space>(es), xi2 = xi * xi,
                               dHat = dHat, activeGap2, n = numPT] __device__(int pti) mutable {
                auto pt = PT[pti];
                auto p = vtemp.pack<3>("xn", pt[0]);
                auto t0 = vtemp.pack<3>("xn", pt[1]);
                auto t1 = vtemp.pack<3>("xn", pt[2]);
                auto t2 = vtemp.pack<3>("xn", pt[3]);

                auto dist2 = dist2_pt(p, t0, t1, t2);
                if (dist2 < xi2)
                    printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                // es[pti] = zs::barrier(dist2 - xi2, activeGap2, (T)1);

                auto I5 = dist2 / activeGap2;
                auto lenE = (dist2 - activeGap2);
                auto E = -lenE * lenE * zs::log(I5);
                reduce_to(pti, n, E, es[pti / 32]);
            });
            Es.push_back(reduce(pol, es) * kappa);

            auto numEE = EE.getCount();
            es.resize(count_warps(numEE));
            es.reset(0);
            pol(range(numEE), [vtemp = proxy<space>({}, vtemp), EE = EE.port(), es = proxy<space>(es), xi2 = xi * xi,
                               dHat = dHat, activeGap2, n = numEE] __device__(int eei) mutable {
                auto ee = EE[eei];
                auto ea0 = vtemp.pack<3>("xn", ee[0]);
                auto ea1 = vtemp.pack<3>("xn", ee[1]);
                auto eb0 = vtemp.pack<3>("xn", ee[2]);
                auto eb1 = vtemp.pack<3>("xn", ee[3]);

                auto dist2 = dist2_ee(ea0, ea1, eb0, eb1);
                if (dist2 < xi2)
                    printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                // es[eei] = zs::barrier(dist2 - xi2, activeGap2, (T)1);

                auto I5 = dist2 / activeGap2;
                auto lenE = (dist2 - activeGap2);
                auto E = -lenE * lenE * zs::log(I5);
                reduce_to(eei, n, E, es[eei / 32]);
            });
            Es.push_back(reduce(pol, es) * kappa);

            if (enableMollification) {
                auto numEEM = EEM.getCount();
                es.resize(count_warps(numEEM));
                es.reset(0);
                pol(range(numEEM), [vtemp = proxy<space>({}, vtemp), EEM = EEM.port(), es = proxy<space>(es),
                                    xi2 = xi * xi, dHat = dHat, activeGap2, n = numEEM] __device__(int eemi) mutable {
                    auto eem = EEM[eemi];
                    auto ea0 = vtemp.pack<3>("xn", eem[0]);
                    auto ea1 = vtemp.pack<3>("xn", eem[1]);
                    auto eb0 = vtemp.pack<3>("xn", eem[2]);
                    auto eb1 = vtemp.pack<3>("xn", eem[3]);

                    auto v0 = ea1 - ea0;
                    auto v1 = eb1 - eb0;
                    auto c = v0.cross(v1).norm();
                    auto I1 = c * c;
                    T E = 0;
                    if (I1 != 0) {
                        auto dist2 = dist2_ee(ea0, ea1, eb0, eb1);
                        if (dist2 < xi2)
                            printf("dist already smaller than xi!\n");
                        auto I2 = dist2 / activeGap2;

                        auto rv0 = vtemp.pack<3>("x0", eem[0]);
                        auto rv1 = vtemp.pack<3>("x0", eem[1]);
                        auto rv2 = vtemp.pack<3>("x0", eem[2]);
                        auto rv3 = vtemp.pack<3>("x0", eem[3]);
                        T epsX = mollifier_threshold_ee(rv0, rv1, rv2, rv3);
                        E = (2 - I1 / epsX) * (I1 / epsX) * -zs::sqr(activeGap2 - activeGap2 * I2) * zs::log(I2);
                    }
                    reduce_to(eemi, n, E, es[eemi / 32]);
                });
                Es.push_back(reduce(pol, es) * kappa);

                auto numPPM = PPM.getCount();
                es.resize(count_warps(numPPM));
                es.reset(0);
                pol(range(numPPM), [vtemp = proxy<space>({}, vtemp), PPM = PPM.port(), es = proxy<space>(es),
                                    xi2 = xi * xi, dHat = dHat, activeGap2, n = numPPM] __device__(int ppmi) mutable {
                    auto ppm = PPM[ppmi];

                    auto v0 = vtemp.pack<3>("xn", ppm[1]) - vtemp.pack<3>("xn", ppm[0]);
                    auto v1 = vtemp.pack<3>("xn", ppm[3]) - vtemp.pack<3>("xn", ppm[2]);
                    auto c = v0.cross(v1).norm();
                    auto I1 = c * c;
                    T E = 0;
                    if (I1 != 0) {
                        auto dist2 = dist2_pp(vtemp.pack<3>("xn", ppm[0]), vtemp.pack<3>("xn", ppm[2]));
                        if (dist2 < xi2)
                            printf("dist already smaller than xi!\n");
                        auto I2 = dist2 / activeGap2;

                        auto rv0 = vtemp.pack<3>("x0", ppm[0]);
                        auto rv1 = vtemp.pack<3>("x0", ppm[1]);
                        auto rv2 = vtemp.pack<3>("x0", ppm[2]);
                        auto rv3 = vtemp.pack<3>("x0", ppm[3]);
                        T epsX = mollifier_threshold_ee(rv0, rv1, rv2, rv3);
                        E = (2 - I1 / epsX) * (I1 / epsX) * -zs::sqr(activeGap2 - activeGap2 * I2) * zs::log(I2);
                    }
                    reduce_to(ppmi, n, E, es[ppmi / 32]);
                });
                Es.push_back(reduce(pol, es) * kappa);

                auto numPEM = PEM.getCount();
                es.resize(count_warps(numPEM));
                es.reset(0);
                pol(range(numPEM), [vtemp = proxy<space>({}, vtemp), PEM = PEM.port(), es = proxy<space>(es),
                                    xi2 = xi * xi, dHat = dHat, activeGap2, n = numPEM] __device__(int pemi) mutable {
                    auto pem = PEM[pemi];

                    auto p = vtemp.pack<3>("xn", pem[0]);
                    auto e0 = vtemp.pack<3>("xn", pem[2]);
                    auto e1 = vtemp.pack<3>("xn", pem[3]);
                    auto v0 = vtemp.pack<3>("xn", pem[1]) - p;
                    auto v1 = e1 - e0;
                    auto c = v0.cross(v1).norm();
                    auto I1 = c * c;
                    T E = 0;
                    if (I1 != 0) {
                        auto dist2 = dist2_pe(p, e0, e1);
                        if (dist2 < xi2)
                            printf("dist already smaller than xi!\n");
                        auto I2 = dist2 / activeGap2;

                        auto rv0 = vtemp.pack<3>("x0", pem[0]);
                        auto rv1 = vtemp.pack<3>("x0", pem[1]);
                        auto rv2 = vtemp.pack<3>("x0", pem[2]);
                        auto rv3 = vtemp.pack<3>("x0", pem[3]);
                        T epsX = mollifier_threshold_ee(rv0, rv1, rv2, rv3);
                        E = (2 - I1 / epsX) * (I1 / epsX) * -zs::sqr(activeGap2 - activeGap2 * I2) * zs::log(I2);
                    }
                    reduce_to(pemi, n, E, es[pemi / 32]);
                });
                Es.push_back(reduce(pol, es) * kappa);
            } // mollification

            if (s_enableFriction) {
                if (fricMu != 0) {
                    if (s_enableSelfFriction) {
                        auto numFPP = FPP.getCount();
                        es.resize(count_warps(numFPP));
                        es.reset(0);
                        pol(range(numFPP),
                            [vtemp = proxy<space>({}, vtemp), fricPP = proxy<space>({}, fricPP), FPP = FPP.port(),
                             es = proxy<space>(es), epsvh = epsv * dt, n = numFPP] __device__(int fppi) mutable {
                                auto fpp = FPP[fppi];
                                auto p0 = vtemp.pack<3>("xn", fpp[0]) - vtemp.pack<3>("xhat", fpp[0]);
                                auto p1 = vtemp.pack<3>("xn", fpp[1]) - vtemp.pack<3>("xhat", fpp[1]);
                                auto basis = fricPP.template pack<3, 2>("basis", fppi);
                                auto fn = fricPP("fn", fppi);
                                auto relDX3D = point_point_rel_dx(p0, p1);
                                auto relDX = basis.transpose() * relDX3D;
                                auto relDXNorm2 = relDX.l2NormSqr();
                                auto E = f0_SF(relDXNorm2, epsvh) * fn;
                                reduce_to(fppi, n, E, es[fppi / 32]);
                            });
                        Es.push_back(reduce(pol, es) * fricMu);

                        auto numFPE = FPE.getCount();
                        es.resize(count_warps(numFPE));
                        es.reset(0);
                        pol(range(numFPE),
                            [vtemp = proxy<space>({}, vtemp), fricPE = proxy<space>({}, fricPE), FPE = FPE.port(),
                             es = proxy<space>(es), epsvh = epsv * dt, n = numFPE] __device__(int fpei) mutable {
                                auto fpe = FPE[fpei];
                                auto p = vtemp.pack<3>("xn", fpe[0]) - vtemp.pack<3>("xhat", fpe[0]);
                                auto e0 = vtemp.pack<3>("xn", fpe[1]) - vtemp.pack<3>("xhat", fpe[1]);
                                auto e1 = vtemp.pack<3>("xn", fpe[2]) - vtemp.pack<3>("xhat", fpe[2]);
                                auto basis = fricPE.template pack<3, 2>("basis", fpei);
                                auto fn = fricPE("fn", fpei);
                                auto yita = fricPE("yita", fpei);
                                auto relDX3D = point_edge_rel_dx(p, e0, e1, yita);
                                auto relDX = basis.transpose() * relDX3D;
                                auto relDXNorm2 = relDX.l2NormSqr();
                                auto E = f0_SF(relDXNorm2, epsvh) * fn;
                                reduce_to(fpei, n, E, es[fpei / 32]);
                            });
                        Es.push_back(reduce(pol, es) * fricMu);

                        auto numFPT = FPT.getCount();
                        es.resize(count_warps(numFPT));
                        es.reset(0);
                        pol(range(numFPT),
                            [vtemp = proxy<space>({}, vtemp), fricPT = proxy<space>({}, fricPT), FPT = FPT.port(),
                             es = proxy<space>(es), epsvh = epsv * dt, n = numFPT] __device__(int fpti) mutable {
                                auto fpt = FPT[fpti];
                                auto p = vtemp.pack<3>("xn", fpt[0]) - vtemp.pack<3>("xhat", fpt[0]);
                                auto v0 = vtemp.pack<3>("xn", fpt[1]) - vtemp.pack<3>("xhat", fpt[1]);
                                auto v1 = vtemp.pack<3>("xn", fpt[2]) - vtemp.pack<3>("xhat", fpt[2]);
                                auto v2 = vtemp.pack<3>("xn", fpt[3]) - vtemp.pack<3>("xhat", fpt[3]);
                                auto basis = fricPT.template pack<3, 2>("basis", fpti);
                                auto fn = fricPT("fn", fpti);
                                auto betas = fricPT.pack(dim_c<2>, "beta", fpti);
                                auto relDX3D = point_triangle_rel_dx(p, v0, v1, v2, betas[0], betas[1]);
                                auto relDX = basis.transpose() * relDX3D;
                                auto relDXNorm2 = relDX.l2NormSqr();
                                auto E = f0_SF(relDXNorm2, epsvh) * fn;
                                reduce_to(fpti, n, E, es[fpti / 32]);
                            });
                        Es.push_back(reduce(pol, es) * fricMu);

                        auto numFEE = FEE.getCount();
                        es.resize(count_warps(numFEE));
                        es.reset(0);
                        pol(range(numFEE),
                            [vtemp = proxy<space>({}, vtemp), fricEE = proxy<space>({}, fricEE), FEE = FEE.port(),
                             es = proxy<space>(es), epsvh = epsv * dt, n = numFEE] __device__(int feei) mutable {
                                auto fee = FEE[feei];
                                auto e0 = vtemp.pack<3>("xn", fee[0]) - vtemp.pack<3>("xhat", fee[0]);
                                auto e1 = vtemp.pack<3>("xn", fee[1]) - vtemp.pack<3>("xhat", fee[1]);
                                auto e2 = vtemp.pack<3>("xn", fee[2]) - vtemp.pack<3>("xhat", fee[2]);
                                auto e3 = vtemp.pack<3>("xn", fee[3]) - vtemp.pack<3>("xhat", fee[3]);
                                auto basis = fricEE.template pack<3, 2>("basis", feei);
                                auto fn = fricEE("fn", feei);
                                auto gammas = fricEE.pack(dim_c<2>, "gamma", feei);
                                auto relDX3D = edge_edge_rel_dx(e0, e1, e2, e3, gammas[0], gammas[1]);
                                auto relDX = basis.transpose() * relDX3D;
                                auto relDXNorm2 = relDX.l2NormSqr();
                                auto E = f0_SF(relDXNorm2, epsvh) * fn;
                                reduce_to(feei, n, E, es[feei / 32]);
                            });
                        Es.push_back(reduce(pol, es) * fricMu);
                    }
                }
            } // fric
        }
        if (enableGround) {
            for (auto &primHandle : prims) {
                if (primHandle.isBoundary()) // skip soft boundary
                    continue;
                const auto &svs = primHandle.getSurfVerts();
                // boundary
                es.resize(count_warps(svs.size()));
                es.reset(0);
                pol(range(svs.size()), [vtemp = proxy<space>({}, vtemp), svs = proxy<space>({}, svs),
                                        es = proxy<space>(es), gn = s_groundNormal, dHat2 = dHat * dHat, n = svs.size(),
                                        svOffset = primHandle.svOffset] ZS_LAMBDA(int svi) mutable {
                    const auto vi = svs("inds", svi, int_c) + svOffset;
                    auto x = vtemp.pack<3>("xn", vi);
                    auto dist = gn.dot(x);
                    auto dist2 = dist * dist;
                    T E;
                    if (dist2 < dHat2)
                        E = -zs::sqr(dist2 - dHat2) * zs::log(dist2 / dHat2);
                    else
                        E = 0;
                    reduce_to(svi, n, E, es[svi / 32]);
                });
                Es.push_back(reduce(pol, es) * kappa);

                if (s_enableFriction)
                    if (fricMu != 0) {
                        es.resize(count_warps(svs.size()));
                        es.reset(0);
                        pol(range(svs.size()),
                            [vtemp = proxy<space>({}, vtemp), svtemp = proxy<space>({}, primHandle.svtemp),
                             svs = proxy<space>({}, svs), es = proxy<space>(es), gn = s_groundNormal, dHat = dHat,
                             epsvh = epsv * dt, fricMu = fricMu, n = svs.size(),
                             svOffset = primHandle.svOffset] ZS_LAMBDA(int svi) mutable {
                                const auto vi = svs("inds", svi, int_c) + svOffset;
                                auto fn = svtemp("fn", svi);
                                T E = 0;
                                if (fn != 0) {
                                    auto x = vtemp.pack<3>("xn", vi);
                                    auto dx = x - vtemp.pack<3>("xhat", vi);
                                    auto relDX = dx - gn.dot(dx) * gn;
                                    auto relDXNorm2 = relDX.l2NormSqr();
                                    auto relDXNorm = zs::sqrt(relDXNorm2);
                                    if (relDXNorm > epsvh) {
                                        E = fn * (relDXNorm - epsvh / 2);
                                    } else {
                                        E = fn * relDXNorm2 / epsvh / 2;
                                    }
                                }
                                reduce_to(svi, n, E, es[svi / 32]);
                            });
                        Es.push_back(reduce(pol, es) * fricMu);
                    }
            }
        }
    }
    // constraints
    if (!BCsatisfied) {
        computeConstraints(pol);
        es.resize(count_warps(numDofs));
        es.reset(0);
        pol(range(numDofs), [vtemp = proxy<space>({}, vtemp), es = proxy<space>(es), n = numDofs,
                             boundaryKappa = boundaryKappa] __device__(int vi) mutable {
            // already updated during "xn" update
            auto cons = vtemp.pack(dim_c<3>, "cons", vi);
            auto w = vtemp("ws", vi);
            // auto lambda = vtemp.pack<3>("lambda", vi);
            int BCfixed = vtemp("BCfixed", vi);
            T E = 0;
            if (!BCfixed)
                // E = (T)(-lambda.dot(cons) * w + 0.5 * w * boundaryKappa * cons.l2NormSqr());
                E = (T)(0.5 * w * boundaryKappa * cons.l2NormSqr());
            reduce_to(vi, n, E, es[vi / 32]);
        });
        Es.push_back(reduce(pol, es));
    }

    std::sort(Es.begin(), Es.end());
    T E = 0;
    for (auto e : Es)
        E += e;
    return E;
}

void UnifiedIPCSystem::systemSolve(zs::CudaExecutionPolicy &cudaPol) {
    // input "grad", multiply, constraints
    // output "dir"
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    /// @note assume right-hand side is already projected
    /// copy diagonal block preconditioners
    cudaPol.sync(false);
    /// solve for A dir = grad;
    // initial guess for hard boundary constraints
    cudaPol(zs::range(numDofs),
            [vtemp = proxy<space>({}, vtemp), coOffset = coOffset, dt = dt, dirOffset = vtemp.getPropertyOffset("dir"),
             xtildeOffset = vtemp.getPropertyOffset("xtilde"),
             xnOffset = vtemp.getPropertyOffset("xn")] ZS_LAMBDA(int i) mutable {
                vtemp.tuple<3>(dirOffset, i) = vec3::zeros();
            });
    // temp = A * dir
    systemMultiply(cudaPol, "dir", "temp");
    project(cudaPol, "temp"); // project production
    // r = grad - temp
    cudaPol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), rOffset = vtemp.getPropertyOffset("r"),
                                 gradOffset = vtemp.getPropertyOffset("grad"),
                                 tempOffset = vtemp.getPropertyOffset("temp")] ZS_LAMBDA(int i) mutable {
        vtemp.tuple<3>(rOffset, i) = vtemp.pack<3>(gradOffset, i) - vtemp.pack<3>(tempOffset, i);
    });
    // project(cudaPol, "r"); // project right hand side
    precondition(cudaPol, "r", "q");
    cudaPol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), pOffset = vtemp.getPropertyOffset("p"),
                                 qOffset = vtemp.getPropertyOffset("q")] ZS_LAMBDA(int i) mutable {
        vtemp.tuple<3>(pOffset, i) = vtemp.pack<3>(qOffset, i);
    });
    double zTrk = dot(cudaPol, "r", "q");
    double residualPreconditionedNorm2 = zTrk;
    double localTol2 = cgRel * cgRel * residualPreconditionedNorm2;
    int iter = 0;

    CppTimer timer;
    timer.tick();
    for (; iter != CGCap; ++iter) {
        if (iter % 50 == 0)
            fmt::print("cg iter: {}, norm2: {} (zTrk: {})\n", iter, residualPreconditionedNorm2, zTrk);

        if (residualPreconditionedNorm2 <= localTol2)
            break;
        systemMultiply(cudaPol, "p", "temp");
        project(cudaPol, "temp"); // project production

        double alpha = zTrk / dot(cudaPol, "temp", "p");
        cudaPol(range(numDofs), [vtemp = proxy<space>({}, vtemp), dirOffset = vtemp.getPropertyOffset("dir"),
                                 pOffset = vtemp.getPropertyOffset("p"), rOffset = vtemp.getPropertyOffset("r"),
                                 tempOffset = vtemp.getPropertyOffset("temp"), alpha] ZS_LAMBDA(int vi) mutable {
            vtemp.tuple(dim_c<3>, dirOffset, vi) =
                vtemp.pack(dim_c<3>, dirOffset, vi) + alpha * vtemp.pack<3>(pOffset, vi);
            vtemp.tuple(dim_c<3>, rOffset, vi) =
                vtemp.pack(dim_c<3>, rOffset, vi) - alpha * vtemp.pack<3>(tempOffset, vi);
        });

        precondition(cudaPol, "r", "q");
        double zTrkLast = zTrk;
        zTrk = dot(cudaPol, "q", "r");
        if (zs::isnan(zTrk, zs::exec_seq)) {
            iter = CGCap;
            residualPreconditionedNorm2 = (localTol2 / (cgRel * cgRel)) + std::max((localTol2 / (cgRel * cgRel)), (T)1);
            continue;
        }
        double beta = zTrk / zTrkLast;
        cudaPol(range(numDofs), [vtemp = proxy<space>(vtemp), beta, pOffset = vtemp.getPropertyOffset("p"),
                                 qOffset = vtemp.getPropertyOffset("q")] ZS_LAMBDA(int vi) mutable {
            vtemp.tuple<3>(pOffset, vi) = vtemp.pack<3>(qOffset, vi) + beta * vtemp.pack<3>(pOffset, vi);
        });

        residualPreconditionedNorm2 = zTrk;
    } // end cg step
    /// copy back results
    if (iter == CGCap && residualPreconditionedNorm2 > (localTol2 / (cgRel * cgRel))) {
        // r = grad - temp
        cudaPol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), tempOffset = vtemp.getPropertyOffset("temp"),
                                     gradOffset = vtemp.getPropertyOffset("grad")] ZS_LAMBDA(int i) mutable {
            vtemp.tuple<3>(tempOffset, i) = vtemp.pack<3>(gradOffset, i);
        });
        precondition(cudaPol, "grad", "dir");
        zeno::log_warn("falling back to gradient descent.");
    }
    cudaPol.sync(true);
    timer.tock(fmt::format("{} cgiters", iter));
}

void UnifiedIPCSystem::groundIntersectionFreeStepsize(zs::CudaExecutionPolicy &pol, T &stepSize) {
    using namespace zs;
    // constexpr T slackness = 0.8;
    constexpr auto space = execspace_e::cuda;

    // zs::Vector<T> finalAlpha{vtemp.get_allocator(), 1};
    auto &finalAlpha = temp;
    finalAlpha.setVal(stepSize);
    pol(Collapse{coOffset},
        [vtemp = proxy<space>({}, vtemp),
         // boundary
         gn = s_groundNormal, finalAlpha = proxy<space>(finalAlpha), stepSize] ZS_LAMBDA(int vi) mutable {
            // this vert affected by sticky boundary conditions
            if (vtemp("BCorder", vi) == 3)
                return;
            auto dir = vtemp.pack<3>("dir", vi);
            auto coef = gn.dot(dir);
            if (coef < 0) { // impacting direction
                auto x = vtemp.pack<3>("xn", vi);
                auto dist = gn.dot(x);
                auto maxAlpha = (dist * 0.8) / (-coef);
                if (maxAlpha < stepSize)
                    atomic_min(exec_cuda, &finalAlpha[0], maxAlpha);
            }
        });
    stepSize = finalAlpha.getVal();
    fmt::print(fg(fmt::color::dark_cyan), "ground alpha: {}\n", stepSize);
}
void UnifiedIPCSystem::intersectionFreeStepsize(zs::CudaExecutionPolicy &pol, T xi, T &stepSize) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    // Vector<T> alpha{vtemp.get_allocator(), 1};
    auto &alpha = temp;
    alpha.setVal(stepSize);
    auto npt = csPT.getCount();
#if PROFILE_IPC
    CppTimer timer;
    timer.tick();
#endif
    pol.profile(PROFILE_IPC);
    pol(range(npt), [csPT = csPT.port(), vtemp = proxy<space>({}, vtemp), alpha = proxy<space>(alpha), stepSize, xi,
                     coOffset = (int)coOffset] __device__(int pti) {
        auto ids = csPT[pti];
        auto p = vtemp.pack(dim_c<3>, "xn", ids[0]);
        auto t0 = vtemp.pack(dim_c<3>, "xn", ids[1]);
        auto t1 = vtemp.pack(dim_c<3>, "xn", ids[2]);
        auto t2 = vtemp.pack(dim_c<3>, "xn", ids[3]);
        auto dp = vtemp.pack(dim_c<3>, "dir", ids[0]);
        auto dt0 = vtemp.pack(dim_c<3>, "dir", ids[1]);
        auto dt1 = vtemp.pack(dim_c<3>, "dir", ids[2]);
        auto dt2 = vtemp.pack(dim_c<3>, "dir", ids[3]);
        T tmp = alpha[0];
#if 1
        if (accd::ptccd(p, t0, t1, t2, dp, dt0, dt1, dt2, (T)0.2, xi, tmp))
#elif 1
            if (ticcd::ptccd(p, t0, t1, t2, dp, dt0, dt1, dt2, (T)0.2, xi, tmp))
#else
            if (pt_ccd(p, t0, t1, t2, dp, dt0, dt1, dt2, xi, tmp))
#endif
            atomic_min(exec_cuda, &alpha[0], tmp);
    });
    auto nee = csEE.getCount();
    pol(range(nee), [csEE = csEE.port(), vtemp = proxy<space>({}, vtemp), alpha = proxy<space>(alpha), stepSize, xi,
                     coOffset = (int)coOffset] __device__(int eei) {
        auto ids = csEE[eei];
        auto ea0 = vtemp.pack(dim_c<3>, "xn", ids[0]);
        auto ea1 = vtemp.pack(dim_c<3>, "xn", ids[1]);
        auto eb0 = vtemp.pack(dim_c<3>, "xn", ids[2]);
        auto eb1 = vtemp.pack(dim_c<3>, "xn", ids[3]);
        auto dea0 = vtemp.pack(dim_c<3>, "dir", ids[0]);
        auto dea1 = vtemp.pack(dim_c<3>, "dir", ids[1]);
        auto deb0 = vtemp.pack(dim_c<3>, "dir", ids[2]);
        auto deb1 = vtemp.pack(dim_c<3>, "dir", ids[3]);
        auto tmp = alpha[0];
#if 1
        if (accd::eeccd(ea0, ea1, eb0, eb1, dea0, dea1, deb0, deb1, (T)0.2, xi, tmp))
#elif 1
            if (ticcd::eeccd(ea0, ea1, eb0, eb1, dea0, dea1, deb0, deb1, (T)0.2, xi, tmp))
#else
            if (ee_ccd(ea0, ea1, eb0, eb1, dea0, dea1, deb0, deb1, xi, tmp))
#endif
            atomic_min(exec_cuda, &alpha[0], tmp);
    });
    pol.profile(false);
#if PROFILE_IPC
    timer.tock("ccd time");
#endif
    stepSize = alpha.getVal();
}
void UnifiedIPCSystem::lineSearch(zs::CudaExecutionPolicy &cudaPol, T &alpha) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    // initial energy
    T E0 = energy(cudaPol, "xn"); // must be "xn", cuz elasticity is hardcoded

    T E{E0};
    T c1m = 0;
    int lsIter = 0;
    // c1m = -armijoParam * dot(cudaPol, "dir", "grad");
    fmt::print(fg(fmt::color::white), "c1m : {}\n", c1m);
    do {
        cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp), alpha] __device__(int i) mutable {
            vtemp.tuple<3>("xn", i) = vtemp.pack<3>("xn0", i) + alpha * vtemp.pack<3>("dir", i);
        });

        if (enableContact)
            findCollisionConstraints(cudaPol, dHat, xi);

        E = energy(cudaPol, "xn"); // must be "xn", cuz elasticity is hardcoded

        fmt::print("E: {} at alpha {}. E0 {}\n", E, alpha, E0);
        if (E <= E0 + alpha * c1m)
            break;

        if (alpha < 1e-3) { // adhoc
            fmt::print(fg(fmt::color::light_yellow), "linesearch early exit with alpha {}\n", alpha);
            break;
        }

        alpha /= 2;
        if (++lsIter > 30) {
            auto cr = constraintResidual(cudaPol);
            fmt::print("too small stepsize at iteration [{}]! alpha: {}, cons "
                       "res: {}\n",
                       lsIter, alpha, cr);
        }
    } while (true);
}

bool UnifiedIPCSystem::newtonKrylov(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    if (!linsys.initialized) {
        initializeSystemHessian(pol);
        linsys.initialized = true;
    }

    /// optimizer
    int newtonIter = 0;
    T res = limits<T>::max();
    for (; newtonIter != PNCap; ++newtonIter) {
        // check constraints
        if (!BCsatisfied) {
            computeConstraints(pol);
            auto cr = constraintResidual(pol, true);
            if (cr < s_constraint_residual) {
                // zeno::log_info("satisfied cons res [{}] at newton iter [{}]\n", cr, newtonIter);
                projectDBC = true;
                BCsatisfied = true;
            }
            fmt::print(fg(fmt::color::alice_blue), "newton iter {} cons residual: {}\n", newtonIter, cr);
        }
        // PRECOMPUTE
        if (enableContact) {
            findCollisionConstraints(pol, dHat, xi);
        }
        if (s_enableFriction)
            if (fricMu != 0) {
                precomputeFrictions(pol, dHat, xi);
            }
        // GRAD, HESS, P
        pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int i) mutable {
            vtemp.tuple(dim_c<3, 3>, "P", i) = mat3::zeros();
            vtemp.tuple(dim_c<3>, "grad", i) = vec3::zeros();
        });

        /// prepare linsys.spmat
        updateInherentHessian(pol, "grad");
        /// prepare linsys.hessx
        updateDynamicHessian(pol, "grad");
        /// prepare diagonal block preconditioner
        prepareDiagonalPreconditioner(pol);

        /// MAS
        linsys.buildPreconditioner(pol, *this); // 1

        project(pol, "grad");

        // PREPARE P (INVERSION)
        pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int i) mutable {
            auto mat = vtemp.pack(dim_c<3, 3>, "P", i);
            if (zs::abs(zs::determinant(mat)) > limits<T>::epsilon() * 10)
                vtemp.tuple(dim_c<3, 3>, "P", i) = inverse(mat);
            else
                vtemp.tuple(dim_c<3, 3>, "P", i) = mat3::identity();
        });

        systemSolve(pol);

        // CHECK PN CONDITION
        res = infNorm(pol, "dir");
        T cons_res = constraintResidual(pol);
        /// @note do not exit in the beginning
        // if (res < targetGRes * dt && cons_res == 0 && newtonIter != 0)
        if (res < targetGRes * dt && newtonIter != 0) {
            break;
        }
        fmt::print(fg(fmt::color::aquamarine), "substep {} newton iter {}: direction residual {}\n", state.getSubstep(),
                   newtonIter, res);
        // LINESEARCH
        pol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int i) mutable {
            vtemp.tuple(dim_c<3>, "xn0", i) = vtemp.pack(dim_c<3>, "xn", i);
        });
        T alpha = 1.;
        if (enableContact) {
            /// @note dcd filter to reduce potential ccd stepsize
            intersectionFreeStepsize(pol, xi, alpha);
        }
        if (enableGround) {
            groundIntersectionFreeStepsize(pol, alpha);
        }
        if (enableContact) {
            // A.intersectionFreeStepsize(cudaPol, xi, alpha);
            // fmt::print("\tstepsize after intersection-free: {}\n", alpha);
            findCCDConstraints(pol, alpha, xi);
            intersectionFreeStepsize(pol, xi, alpha);
        }
        lineSearch(pol, alpha);
        pol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp), alpha] ZS_LAMBDA(int i) mutable {
            vtemp.tuple(dim_c<3>, "xn", i) = vtemp.pack(dim_c<3>, "xn0", i) + alpha * vtemp.pack(dim_c<3>, "dir", i);
        });
        // UPDATE RULE
        cons_res = constraintResidual(pol);
#if 0
        if (res * dt < updateZoneTol && cons_res > consTol) {
            if (boundaryKappa < kappaMax) {
                boundaryKappa *= 2;
                fmt::print(fg(fmt::color::ivory),
                           "increasing boundarykappa to {} due to constraint "
                           "difficulty.\n",
                           boundaryKappa);
                // getchar();
            } else {
#if 0
                pol(Collapse{numDofs},
                    [vtemp = proxy<space>({}, vtemp), boundaryKappa = boundaryKappa] ZS_LAMBDA(int vi) mutable {
                        if (int BCorder = vtemp("BCorder", vi); BCorder > 0) {
                            vtemp.tuple<3>("lambda", vi) = vtemp.pack<3>("lambda", vi) -
                                                           boundaryKappa * vtemp("ws", vi) * vtemp.pack<3>("cons", vi);
                        }
                    });
                fmt::print(fg(fmt::color::ivory), "updating constraint lambda due to constraint difficulty.\n");
                // getchar();
#endif
            }
        }
#endif
    }
    zeno::log_warn("\t# substep {} newton optimizer ends in {} iters with residual {}\n", state.getSubstep(),
                   newtonIter, res);
    return newtonIter != PNCap;
}

struct StepUnifiedIPCSystem : INode {
    void apply() override {
        using namespace zs;
        auto A = get_input<UnifiedIPCSystem>("ZSUnifiedIPCSystem");

        auto cudaPol = zs::cuda_exec();

        int nSubsteps = get_input2<int>("num_substeps");
        auto dt = get_input2<float>("dt");

        A->reinitialize(cudaPol, dt);
        A->suggestKappa(cudaPol);

        for (int subi = 0; subi != nSubsteps; ++subi) {
            A->advanceSubstep(cudaPol, (typename UnifiedIPCSystem::T)1 / nSubsteps);

            int numFricSolve = A->s_enableFriction && A->fricMu != 0 ? A->fricIterCap : 1;
        for_fric:
            A->needFricPrecompute = true;

            bool success = A->newtonKrylov(cudaPol);

            if (--numFricSolve > 0)
                goto for_fric;

            /// @note only update substep velocity when converged
            if (success)
                A->updateVelocities(cudaPol);
        }
        // update velocity and positions
        A->writebackPositionsAndVelocities(cudaPol);

        set_output("ZSUnifiedIPCSystem", A);
    }
};

ZENDEFNODE(StepUnifiedIPCSystem, {{
                                      "ZSUnifiedIPCSystem",
                                      {"int", "num_substeps", "1"},
                                      {"float", "dt", "0.01"},
                                  },
                                  {"ZSUnifiedIPCSystem"},
                                  {},
                                  {"FEM"}});

struct UnifiedIPCSystemClothBinding : INode { // usually called once before stepping
    using tiles_t = typename ZenoParticles::particles_t;
#if 1
    // unordered version
    using bvh_t = zs::LBvh<3, int, zs::f32>;
    using bv_t = typename bvh_t::Box;
#else
    using bvh_t = typename UnifiedIPCSystem::bvh_t;
    using bv_t = typename UnifiedIPCSystem::bv_t;
#endif
    template <typename VecT>
    static constexpr float distance(const bv_t &bv, const zs::VecInterface<VecT> &x) {
        using namespace zs;
        const auto &mi = bv._min;
        const auto &ma = bv._max;
        // const auto &[mi, ma] = bv;
        auto center = (mi + ma) / 2;
        auto point = (x - center).abs() - (ma - mi) / 2;
        float max = limits<float>::lowest();
        for (int d = 0; d != 3; ++d) {
            if (point[d] > max)
                max = point[d];
            if (point[d] < 0)
                point[d] = 0;
        }
        return (max < 0.f ? max : 0.f) + point.length();
    }
    template <typename VTilesT, typename LsView, typename Bvh>
    std::shared_ptr<tiles_t> bindStrings(zs::CudaExecutionPolicy &cudaPol, VTilesT &vtemp, std::size_t numVerts,
                                         LsView lsv, const Bvh &bvh, float k, float distCap, float rl) {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        // assume all verts
        Vector<int> nStrings{vtemp.get_allocator(), 1};
        nStrings.setVal(0);
        tiles_t strings{vtemp.get_allocator(), {{"inds", 2}, {"vol", 1}, {"k", 1}, {"rl", 1}}, numVerts};
        cudaPol(range(numVerts), [vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, strings), lsv, distCap,
                                  bvh = proxy<space>(bvh), cnt = proxy<space>(nStrings), coOffset = numVerts, k,
                                  rl] ZS_LAMBDA(int i) mutable {
            auto x = vtemp.pack(dim_c<3>, "xn", i);
            if (lsv.getSignedDistance(x) < 0) {
                float dist = distCap;
                int j = -1;
                int numNodes = bvh.numNodes();
#if 0
                auto nt = bvh.numLeaves() - 1;
                int node = bvh._root;
                while (node != -1) {
                    for (; node < nt; node = bvh._trunkTopo("lc", node))
                        if (auto d = distance(bvh.getNodeBV(node), x); d > dist)
                            break;
                    // leaf node check
                    if (node >= nt) {
                        auto bouId = bvh._leafTopo("inds", node - nt) + coOffset;
                        auto d = (vtemp.pack(dim_c<3>, "xn", bouId) - x).length();
                        if (d < dist) {
                            dist = d;
                            j = bouId;
                        }
                        node = bvh._leafTopo("esc", node - nt);
                    } else // separate at internal nodes
                        node = bvh._trunkTopo("esc", node);
                }
#else
                int node = 0;
                while (node != -1 && node != numNodes) {
                    int level = bvh._levels[node];
                    for (; level; --level, ++node)
                        if (auto d = distance(bvh.getNodeBV(node), x); d > dist)
                            break;
                    // leaf node check
                    if (level == 0) {
                        auto bouId = bvh._auxIndices[node] + coOffset;
                        auto d = (vtemp.pack(dim_c<3>, "xn", bouId) - x).length();
                        if (d < dist) {
                            dist = d;
                            j = bouId;
                        }
                        node++;
                    } else // separate at internal nodes
                        node = bvh._auxIndices[node];
                }
#endif
                if (j != -1) {
                    auto no = atomic_add(exec_cuda, &cnt[0], 1);
                    eles.tuple(dim_c<2>, "inds", no, int_c) = zs::vec<int, 2>{i, j};
                    eles("vol", no) = 1;
                    eles("k", no) = k;
                    eles("rl", no) = zs::min(dist / 4, rl);
                }
            }
        });
        auto cnt = nStrings.getVal();
        strings.resize(cnt);
        return std::make_shared<tiles_t>(std::move(strings));
    }
    void apply() override {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        auto A = get_input<UnifiedIPCSystem>("ZSUnifiedIPCSystem");
        if (!A->hasBoundary()) {
            set_output("ZSUnifiedIPCSystem", A);
            return;
        }
        const auto &bouVerts = *A->coVerts;
        const auto numBouVerts = bouVerts.size();
        if (numBouVerts == 0) {
            set_output("ZSUnifiedIPCSystem", A);
            return;
        }
        auto &vtemp = A->vtemp;
        const auto numVerts = A->coOffset;

        auto zsls = get_input<ZenoLevelSet>("ZSLevelSet");
        bool ifHardCons = get_input2<bool>("hard_constraint");

        auto cudaPol = zs::cuda_exec().sync(true);
        bvh_t bouBvh;
        Vector<bv_t> bouVertBvs{vtemp.get_allocator(), numBouVerts};
        cudaPol(enumerate(bouVertBvs),
                [vtemp = proxy<space>({}, vtemp), coOffset = numVerts] ZS_LAMBDA(int i, bv_t &bv) {
                    auto p = vtemp.pack(dim_c<3>, "xn", i + coOffset);
                    bv = bv_t{p - limits<float>::epsilon() * 8, p + limits<float>::epsilon() * 8};
                });
        bouBvh.build(cudaPol, bouVertBvs);

        // stiffness
        float k = get_input2<float>("strength"); // pulling stiffness
        if (k == 0)                              // auto setup
            k = A->largestMu() * 100;
        // dist cap
        float dist_cap = get_input2<float>("dist_cap"); // only proximity pairs within this range considered
        if (dist_cap == 0)
            dist_cap = limits<float>::max();
        float rl = get_input2<float>("rest_length"); // rest length cap
        match([&](const auto &ls) {
            using basic_ls_t = typename ZenoLevelSet::basic_ls_t;
            using const_sdf_vel_ls_t = typename ZenoLevelSet::const_sdf_vel_ls_t;
            using const_transition_ls_t = typename ZenoLevelSet::const_transition_ls_t;
            if constexpr (is_same_v<RM_CVREF_T(ls), basic_ls_t>) {
                match([&](const auto &lsPtr) {
                    auto lsv = get_level_set_view<execspace_e::cuda>(lsPtr);
                    A->pushBoundarySprings(bindStrings(cudaPol, vtemp, numVerts, lsv, bouBvh, k, dist_cap, rl),
                                           ifHardCons ? ZenoParticles::category_e::tracker
                                                      : ZenoParticles::category_e::curve);
                })(ls._ls);
            } else if constexpr (is_same_v<RM_CVREF_T(ls), const_sdf_vel_ls_t>) {
                match([&](auto lsv) {
                    A->pushBoundarySprings(
                        bindStrings(cudaPol, vtemp, numVerts, SdfVelFieldView{lsv}, bouBvh, k, dist_cap, rl),
                        ifHardCons ? ZenoParticles::category_e::tracker : ZenoParticles::category_e::curve);
                })(ls.template getView<execspace_e::cuda>());
            } else if constexpr (is_same_v<RM_CVREF_T(ls), const_transition_ls_t>) {
                match([&](auto fieldPair) {
                    auto &fvSrc = zs::get<0>(fieldPair);
                    auto &fvDst = zs::get<1>(fieldPair);
                    A->pushBoundarySprings(
                        bindStrings(cudaPol, vtemp, numVerts,
                                    TransitionLevelSetView{SdfVelFieldView{fvSrc}, SdfVelFieldView{fvDst}, ls._stepDt,
                                                           ls._alpha},
                                    bouBvh, k, dist_cap, rl),
                        ifHardCons ? ZenoParticles::category_e::tracker : ZenoParticles::category_e::curve);
                })(ls.template getView<zs::execspace_e::cuda>());
            }
        })(zsls->getLevelSet());

        set_output("ZSUnifiedIPCSystem", A);
    }
};

ZENDEFNODE(UnifiedIPCSystemClothBinding, {{
                                              "ZSUnifiedIPCSystem",
                                              "ZSLevelSet",
                                              {"bool", "hard_constraint", "1"},
                                              {"float", "dist_cap", "0"},
                                              {"float", "rest_length", "0.1"},
                                              {"float", "strength", "0"},
                                          },
                                          {"ZSUnifiedIPCSystem"},
                                          {},
                                          {"FEM"}});

struct UnifiedIPCSystemForceField : INode {
    template <typename VelSplsViewT>
    void computeForce(zs::CudaExecutionPolicy &cudaPol, float windDragCoeff, float windDensity, int vOffset,
                      VelSplsViewT velLs, typename UnifiedIPCSystem::dtiles_t &vtemp,
                      const typename UnifiedIPCSystem::tiles_t &eles) {
        using namespace zs;
        cudaPol(range(eles.size()), [windDragCoeff, windDensity, velLs, vtemp = proxy<execspace_e::cuda>({}, vtemp),
                                     eles = proxy<execspace_e::cuda>({}, eles), vOffset] ZS_LAMBDA(size_t ei) mutable {
            auto inds = eles.pack<3>("inds", ei, int_c) + vOffset;
            auto p0 = vtemp.pack(dim_c<3>, "xn", inds[0]);
            auto p1 = vtemp.pack(dim_c<3>, "xn", inds[1]);
            auto p2 = vtemp.pack(dim_c<3>, "xn", inds[2]);
            auto cp = (p1 - p0).cross(p2 - p0);
            auto area = cp.length();
            auto n = cp / area;
            area *= 0.5;

            auto pos = (p0 + p1 + p2) / 3; // get center to sample velocity
            auto windVel = velLs.getMaterialVelocity(pos);

            auto vel = (vtemp.pack(dim_c<3>, "vn", inds[0]) + vtemp.pack(dim_c<3>, "vn", inds[1]) +
                        vtemp.pack(dim_c<3>, "vn", inds[2])) /
                       3;
            auto vrel = windVel - vel;
            auto vnSignedLength = n.dot(vrel);
            auto vn = n * vnSignedLength;
            auto vt = vrel - vn; // tangent
            auto windForce = windDensity * area * zs::abs(vnSignedLength) * vn + windDragCoeff * area * vt;
            auto f = windForce;
            for (int i = 0; i != 3; ++i)
                for (int d = 0; d != 3; ++d) {
                    atomic_add(exec_cuda, &vtemp("extf", d, inds[i]), f[d] / 3);
                }
        });
    }
    void apply() override {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;

        auto A = get_input<UnifiedIPCSystem>("ZSUnifiedIPCSystem");
        auto &vtemp = A->vtemp;
        const auto numVerts = A->coOffset;
        auto zsls = get_input<ZenoLevelSet>("ZSLevelSet");

        auto cudaPol = zs::cuda_exec();
        vtemp.append_channels(cudaPol, {{"extf", 3}});
        cudaPol(range(numVerts), [vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int i) mutable {
            vtemp.template tuple<3>("extf", i) = zs::vec<double, 3>::zeros();
        });

        auto windDrag = get_input2<float>("wind_drag");
        auto windDensity = get_input2<float>("wind_density");

        for (auto &primHandle : A->prims) {
            if (primHandle.category != ZenoParticles::category_e::surface)
                continue;
            const auto &eles = primHandle.getEles();
            match([&](const auto &ls) {
                using basic_ls_t = typename ZenoLevelSet::basic_ls_t;
                using const_sdf_vel_ls_t = typename ZenoLevelSet::const_sdf_vel_ls_t;
                using const_transition_ls_t = typename ZenoLevelSet::const_transition_ls_t;
                if constexpr (is_same_v<RM_CVREF_T(ls), basic_ls_t>) {
                    match([&](const auto &lsPtr) {
                        auto lsv = get_level_set_view<execspace_e::cuda>(lsPtr);
                        computeForce(cudaPol, windDrag, windDensity, primHandle.vOffset, lsv, vtemp, eles);
                    })(ls._ls);
                } else if constexpr (is_same_v<RM_CVREF_T(ls), const_sdf_vel_ls_t>) {
                    match([&](auto lsv) {
                        computeForce(cudaPol, windDrag, windDensity, primHandle.vOffset, SdfVelFieldView{lsv}, vtemp,
                                     eles);
                    })(ls.template getView<execspace_e::cuda>());
                } else if constexpr (is_same_v<RM_CVREF_T(ls), const_transition_ls_t>) {
                    match([&](auto fieldPair) {
                        auto &fvSrc = zs::get<0>(fieldPair);
                        auto &fvDst = zs::get<1>(fieldPair);
                        computeForce(cudaPol, windDrag, windDensity, primHandle.vOffset,
                                     TransitionLevelSetView{SdfVelFieldView{fvSrc}, SdfVelFieldView{fvDst}, ls._stepDt,
                                                            ls._alpha},
                                     vtemp, eles);
                    })(ls.template getView<zs::execspace_e::cuda>());
                }
            })(zsls->getLevelSet());
        }

        set_output("ZSUnifiedIPCSystem", A);
    }
};

ZENDEFNODE(UnifiedIPCSystemForceField,
           {
               {"ZSUnifiedIPCSystem", "ZSLevelSet", {"float", "wind_drag", "0"}, {"float", "wind_density", "1"}},
               {"ZSUnifiedIPCSystem"},
               {},
               {"FEM"},
           });

} // namespace zeno