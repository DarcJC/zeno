#include "hip/hip_runtime.h"
#include "UnifiedSolver.cuh"
#include "Utils.hpp"
#include "zensim/geometry/Distance.hpp"
#include "zensim/geometry/Friction.hpp"
#include "zensim/geometry/SpatialQuery.hpp"
#include "zensim/math/DihedralAngle.hpp"
#include "zensim/types/SmallVector.hpp"

namespace zeno {

/// inertia
void UnifiedIPCSystem::computeInertialAndGravityPotentialGradient(zs::CudaExecutionPolicy &cudaPol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    // inertial
    cudaPol(zs::range(coOffset), [tempI = proxy<space>({}, tempI),
                                  vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int i) mutable {
        auto m = vtemp("ws", i);
        vtemp.tuple(dim_c<3>, "grad", i) =
            vtemp.pack(dim_c<3>, "grad", i) - m * (vtemp.pack(dim_c<3>, "xn", i) - vtemp.pack(dim_c<3>, "xtilde", i));

        int BCorder[1] = {(int)vtemp("BCorder", i)};
        auto M = mat3::identity() * m;
        for (int d = 0; d != BCorder[0]; ++d)
            M.val(d * 4) = 0;
        tempI.tuple(dim_c<9>, "Hi", i) = M;
        // prepare preconditioner
        for (int d = 0; d != 3; ++d)
            vtemp("P", d * 3 + d, i) += M(d, d);
    });
    if (vtemp.hasProperty("extf")) {
        cudaPol(zs::range(coOffset), [vtemp = proxy<space>({}, vtemp), dt = dt] ZS_LAMBDA(int vi) mutable {
            int BCorder = vtemp("BCorder", vi);
            if (BCorder == 0) // BCsoft == 0 &&
                vtemp.tuple(dim_c<3>, "grad", vi) =
                    vtemp.pack(dim_c<3>, "grad", vi) + vtemp.pack(dim_c<3>, "extf", vi) * dt * dt;
        });
    }
}
void UnifiedIPCSystem::computeInertialPotentialGradient(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString &gTag) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    // inertial
    cudaPol(zs::range(coOffset), [vtemp = proxy<space>({}, vtemp), gTag, dt = dt] ZS_LAMBDA(int i) mutable {
        auto m = vtemp("ws", i);
        vtemp.tuple(dim_c<3>, gTag, i) =
            vtemp.pack(dim_c<3>, gTag, i) - m * (vtemp.pack(dim_c<3>, "xn", i) - vtemp.pack(dim_c<3>, "xtilde", i));
    });
}

/// @note writes to sparse matrix with fixed topo
template <typename Model, typename SpmatH>
void computeElasticGradientAndHessianImpl(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString &gTag,
                                          typename UnifiedIPCSystem::dtiles_t &vtemp,
                                          typename UnifiedIPCSystem::PrimitiveHandle &primHandle, const Model &model,
                                          typename UnifiedIPCSystem::T dt, SpmatH &spmat) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    using mat3 = typename UnifiedIPCSystem::mat3;
    using vec3 = typename UnifiedIPCSystem::vec3;
    using T = typename UnifiedIPCSystem::T;
    if (primHandle.category == ZenoParticles::curve) {
        if (primHandle.isBoundary() && !primHandle.isAuxiliary())
            return;
        /// ref: Fast Simulation of Mass-Spring Systems
        /// credits: Tiantian Liu
        cudaPol(zs::range(primHandle.getEles().size()),
                [vtemp = proxy<space>({}, vtemp), spmat = view<space>(spmat),
                 eles = proxy<space>({}, primHandle.getEles()), model, gTag, dt = dt,
                 vOffset = primHandle.vOffset] __device__(int ei) mutable {
                    auto inds = eles.pack(dim_c<2>, "inds", ei, int_c) + vOffset;
                    int BCorder[2];
                    for (int i = 0; i != 2; ++i) {
                        BCorder[i] = vtemp("BCorder", inds[i]);
                    }

                    if (BCorder[0] == 3 && BCorder[1] == 3)
                        return;

                    auto vole = eles("vol", ei);
                    auto k = eles("k", ei);
                    auto rl = eles("rl", ei);

                    vec3 xs[2] = {vtemp.pack(dim_c<3>, "xn", inds[0]), vtemp.pack(dim_c<3>, "xn", inds[1])};
                    auto xij = xs[1] - xs[0];
                    auto lij = xij.norm();
                    auto dij = xij / lij;
                    auto gij = k * (lij - rl) * dij;

                    /// gradient
                    auto vfdt2 = gij * (dt * dt) * vole;
                    for (int d = 0; d != 3; ++d) {
                        atomic_add(exec_cuda, &vtemp(gTag, d, inds[0]), (T)vfdt2(d));
                        atomic_add(exec_cuda, &vtemp(gTag, d, inds[1]), (T)-vfdt2(d));
                    }

                    auto H = zs::vec<T, 6, 6>::zeros();
                    auto K = k * (mat3::identity() - rl / lij * (mat3::identity() - dyadic_prod(dij, dij)));
                    // make_pd(K);  // symmetric semi-definite positive, not
                    // necessary

                    for (int i = 0; i != 3; ++i)
                        for (int j = 0; j != 3; ++j) {
                            H(i, j) = K(i, j);
                            H(i, 3 + j) = -K(i, j);
                            H(3 + i, j) = -K(i, j);
                            H(3 + i, 3 + j) = K(i, j);
                        }
                    H *= dt * dt * vole;

                    for (int vi = 0; vi != 2; ++vi) {
                        auto i = inds[vi];
                        for (int vj = 0; vj != 2; ++vj) {
                            auto j = inds[vj];
                            if (i > j)
                                continue;
                            auto loc = spmat.locate(i, j, true_c);
                            auto &mat = spmat._vals[loc];
                            for (int r = 0; r != 3; ++r)
                                for (int c = 0; c != 3; ++c) {
                                    atomic_add(exec_cuda, &mat(r, c), H(vi * 3 + r, vj * 3 + c));
                                }
                        }
                    }
                });
    } else if (primHandle.category == ZenoParticles::surface) {
        if (primHandle.isBoundary())
            return;
        cudaPol(zs::range(primHandle.getEles().size()), [vtemp = proxy<space>({}, vtemp), spmat = view<space>(spmat),
                                                         eles = proxy<space>({}, primHandle.getEles()), model, gTag,
                                                         dt = dt,
                                                         vOffset = primHandle.vOffset] __device__(int ei) mutable {
            auto IB = eles.pack(dim_c<2, 2>, "IB", ei);
            auto inds = eles.pack(dim_c<3>, "inds", ei, int_c) + vOffset;
            auto vole = eles("vol", ei);
            vec3 xs[3] = {vtemp.pack(dim_c<3>, "xn", inds[0]), vtemp.pack(dim_c<3>, "xn", inds[1]),
                          vtemp.pack(dim_c<3>, "xn", inds[2])};
            auto x1x0 = xs[1] - xs[0];
            auto x2x0 = xs[2] - xs[0];

            int BCorder[3];
            for (int i = 0; i != 3; ++i) {
                BCorder[i] = vtemp("BCorder", inds[i]);
            }
            zs::vec<T, 9, 9> H;
            if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3) {
                return;
            }

            zs::vec<T, 3, 2> Ds{x1x0[0], x2x0[0], x1x0[1], x2x0[1], x1x0[2], x2x0[2]};
            auto F = Ds * IB;

            auto dFdX = dFdXMatrix(IB, wrapv<3>{});
            auto dFdXT = dFdX.transpose();
            auto f0 = col(F, 0);
            auto f1 = col(F, 1);

            /// gradient
            auto f0Norm = zs::sqrt(f0.l2NormSqr());
            auto f1Norm = zs::sqrt(f1.l2NormSqr());
            auto f0Tf1 = f0.dot(f1);
            zs::vec<T, 3, 2> Pstretch, Pshear;
            for (int d = 0; d != 3; ++d) {
                Pstretch(d, 0) = 2 * (1 - 1 / f0Norm) * F(d, 0);
                Pstretch(d, 1) = 2 * (1 - 1 / f1Norm) * F(d, 1);
                Pshear(d, 0) = 2 * f0Tf1 * f1(d);
                Pshear(d, 1) = 2 * f0Tf1 * f0(d);
            }
            auto vecP = flatten(model.mu * Pstretch + (model.mu * 0.3) * Pshear);
            auto vfdt2 = -vole * (dFdXT * vecP) * (dt * dt);

            for (int i = 0; i != 3; ++i) {
                auto vi = inds[i];
                for (int d = 0; d != 3; ++d)
                    atomic_add(exec_cuda, &vtemp(gTag, d, vi), (T)vfdt2(i * 3 + d));
            }
            /// hessian
            /// ref: A Finite Element Formulation of Baraff-Witkin Cloth
            // suggested by huang kemeng
            auto stretchHessian = [&F, &f0, &f1, &model]() {
                auto H = zs::vec<T, 6, 6>::zeros();
                // const zs::vec<T, 2> u{1, 0};
                // const zs::vec<T, 2> v{0, 1};
                const T I5u = f0.l2NormSqr(); // Fu
                const T I5v = f1.l2NormSqr(); // Fv
                const T invSqrtI5u = (T)1 / zs::sqrt(I5u);
                const T invSqrtI5v = (T)1 / zs::sqrt(I5v);

                H(0, 0) = H(1, 1) = H(2, 2) = zs::max(1 - invSqrtI5u, (T)0);
                H(3, 3) = H(4, 4) = H(5, 5) = zs::max(1 - invSqrtI5v, (T)0);

                const auto fu = f0.normalized();
                const T uCoeff = (1 - invSqrtI5u >= 0) ? invSqrtI5u : (T)1;
                for (int i = 0; i != 3; ++i)
                    for (int j = 0; j != 3; ++j)
                        H(i, j) += uCoeff * fu(i) * fu(j);

                const auto fv = f1.normalized();
                const T vCoeff = (1 - invSqrtI5v >= 0) ? invSqrtI5v : (T)1;
                for (int i = 0; i != 3; ++i)
                    for (int j = 0; j != 3; ++j)
                        H(3 + i, 3 + j) += vCoeff * fv(i) * fv(j);

                H *= model.mu;
                return H;
            };
            auto shearHessian = [&F, &f0, &f1, &model]() {
                using mat6 = zs::vec<T, 6, 6>;
                auto H = mat6::zeros();
                // const zs::vec<T, 2> u{1, 0};
                // const zs::vec<T, 2> v{0, 1};
                const T I6 = f0.dot(f1);
                const T signI6 = I6 >= 0 ? 1 : -1;

                H(3, 0) = H(4, 1) = H(5, 2) = H(0, 3) = H(1, 4) = H(2, 5) = (T)1;

                // F * | 0  1 |
                //     | 1  0 |
                // =
                // | F01 F00 |
                // | F11 F10 |
                // | F21 F20 |
                // const auto g_ = F * (dyadic_prod(u, v) + dyadic_prod(v, u));
                zs::vec<T, 6> g{F(0, 1), F(1, 1), F(2, 1), F(0, 0), F(1, 0), F(2, 0)};
#if 0
                        for (int j = 0, offset = 0; j != 2; ++j) {
                            for (int i = 0; i != 3; ++i)
                                g(offset++) = g_(i, j);
                        }
#endif

                const T I2 = F.l2NormSqr();
                const T lambda0 = (T)0.5 * (I2 + zs::sqrt(I2 * I2 + (T)12 * I6 * I6));

                const zs::vec<T, 6> q0 = (I6 * H * g + lambda0 * g).normalized();

                auto t = 0.5 * (mat6::identity() + signI6 * H);

                const zs::vec<T, 6> Tq = t * q0;
                const auto normTq = Tq.l2NormSqr();

                mat6 dPdF = zs::abs(I6) * (t - (dyadic_prod(Tq, Tq) / normTq)) + lambda0 * (dyadic_prod(q0, q0));
                dPdF *= (model.mu * 0.3);
                return dPdF;
            };
            auto He = stretchHessian() + shearHessian();
            H = dFdXT * He * dFdX;
            H *= dt * dt * vole;

            for (int vi = 0; vi != 3; ++vi) {
                auto i = inds[vi];
                for (int vj = 0; vj != 3; ++vj) {
                    auto j = inds[vj];
                    if (i > j)
                        continue;
                    auto loc = spmat.locate(i, j, true_c);
                    auto &mat = spmat._vals[loc];
                    for (int r = 0; r != 3; ++r)
                        for (int c = 0; c != 3; ++c) {
                            atomic_add(exec_cuda, &mat(r, c), H(vi * 3 + r, vj * 3 + c));
                        }
                }
            }
        });
    } else if (primHandle.category == ZenoParticles::tet)
        cudaPol(zs::range(primHandle.getEles().size()),
                [vtemp = proxy<space>({}, vtemp), spmat = view<space>(spmat),
                 eles = proxy<space>({}, primHandle.getEles()), model, gTag, dt = dt,
                 vOffset = primHandle.vOffset] __device__(int ei) mutable {
                    auto IB = eles.pack(dim_c<3, 3>, "IB", ei);
                    auto inds = eles.pack(dim_c<4>, "inds", ei, int_c) + vOffset;
                    auto vole = eles("vol", ei);
                    vec3 xs[4] = {vtemp.pack<3>("xn", inds[0]), vtemp.pack<3>("xn", inds[1]),
                                  vtemp.pack<3>("xn", inds[2]), vtemp.pack<3>("xn", inds[3])};

                    int BCorder[4];
                    for (int i = 0; i != 4; ++i) {
                        BCorder[i] = vtemp("BCorder", inds[i]);
                    }
                    zs::vec<T, 12, 12> H;
                    if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3 && BCorder[3] == 3) {
                        return;
                    }
                    mat3 F{};
                    {
                        auto x1x0 = xs[1] - xs[0];
                        auto x2x0 = xs[2] - xs[0];
                        auto x3x0 = xs[3] - xs[0];
                        auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1], x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                        F = Ds * IB;
                    }
                    auto dFdX = dFdXMatrix(IB);
                    auto dFdXT = dFdX.transpose();
                    /// gradient
                    auto P = model.first_piola(F);
                    auto vecP = flatten(P);
                    auto vfdt2 = -vole * (dFdXT * vecP) * dt * dt;

                    for (int i = 0; i != 4; ++i) {
                        auto vi = inds[i];
                        for (int d = 0; d != 3; ++d)
                            atomic_add(exec_cuda, &vtemp(gTag, d, vi), (T)vfdt2(i * 3 + d));
                    }

                    /// hessian
                    auto Hq = model.first_piola_derivative(F, true_c);
                    H = dFdXT * Hq * dFdX * vole * dt * dt;

                    for (int vi = 0; vi != 4; ++vi) {
                        auto i = inds[vi];
                        for (int vj = 0; vj != 4; ++vj) {
                            auto j = inds[vj];
                            if (i > j)
                                continue;
                            auto loc = spmat.locate(i, j, true_c);
                            auto &mat = spmat._vals[loc];
                            for (int r = 0; r != 3; ++r)
                                for (int c = 0; c != 3; ++c) {
                                    atomic_add(exec_cuda, &mat(r, c), H(vi * 3 + r, vj * 3 + c));
                                }
                        }
                    }
                });
}
/// @brief inertial, kinetic, external force, elasticity, bending, boundary motion, ground collision
void UnifiedIPCSystem::updateInherentHessian(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString &gTag) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    auto &spmat = linsys.spmat;
    /// clear entry values
    spmat._vals.reset(0);
    /// @note inertial, gravity
    cudaPol(zs::range(coOffset), [spmat = view<space>(spmat), tempI = proxy<space>({}, tempI),
                                  vtemp = proxy<space>({}, vtemp),
                                  boundaryKappa = boundaryKappa] ZS_LAMBDA(int i) mutable {
        using mat3 = RM_CVREF_T(spmat)::value_type;
        auto m = vtemp("ws", i);
        int BCorder = {(int)vtemp("BCorder", i)};

        /// inertial gradient
        if (BCorder == 0)
            vtemp.tuple(dim_c<3>, "grad", i) = vtemp.pack(dim_c<3>, "grad", i) -
                                               m * (vtemp.pack(dim_c<3>, "xn", i) - vtemp.pack(dim_c<3>, "xtilde", i));

        /// hesssian
        auto Hi = mat3::identity() * m;
        for (int d = 0; d != BCorder; ++d)
            Hi.val(d * 4) = 0;
        auto loc = spmat._ptrs[i];
        auto &mat = spmat._vals[loc];
        for (int r = 0; r != 3; ++r)
            for (int c = 0; c != 3; ++c)
                mat(r, c) = Hi(r, c);
    });
    /// @note force field gradient
    if (vtemp.hasProperty("extf")) {
        cudaPol(zs::range(coOffset), [vtemp = proxy<space>({}, vtemp), dt = dt] ZS_LAMBDA(int vi) mutable {
            int BCorder = vtemp("BCorder", vi);
            if (BCorder == 0) // BCsoft == 0 &&
                vtemp.tuple(dim_c<3>, "grad", vi) =
                    vtemp.pack(dim_c<3>, "grad", vi) + vtemp.pack(dim_c<3>, "extf", vi) * dt * dt;
        });
    }
    /// @note boundary motion
    if (!BCsatisfied) {
        cudaPol(zs::range(numDofs), [spmat = view<space>(spmat), vtemp = proxy<space>({}, vtemp),
                                     boundaryKappa = boundaryKappa] ZS_LAMBDA(int vi) mutable {
            int BCfixed = vtemp("BCfixed", vi);
            if (!BCfixed) {
                auto w = vtemp("ws", vi);
                auto cons = vtemp.pack(dim_c<3>, "cons", vi);
                auto loc = spmat._ptrs[vi];
                auto &mat = spmat._vals[loc];
                int BCorder = vtemp("BCorder", vi);
                for (int d = 0; d != BCorder; ++d) {
                    /// gradient
                    vtemp("grad", d, vi) -= boundaryKappa * w * cons(d);
                    /// hessian
                    mat.val(d * 4) += boundaryKappa * w;
                }
            }
        });
    }
#if 0
    /// deprecated
    cudaPol(zs::range(numDofs * 3), [spmat = view<space>(spmat), vtemp = proxy<space>({}, vtemp),
                                     boundaryKappa = boundaryKappa] ZS_LAMBDA(int i) mutable {
        auto dofi = i / 3;
        int d = i % 3;
        int BCfixed = vtemp("BCfixed", dofi);
        if (!BCfixed) {
            auto m = vtemp("ws", dofi);
            int BCorder = vtemp("BCorder", dofi);
            auto loc = spmat.locate(dofi, dofi, true_c);
            auto &mat = spmat._vals[loc];

            auto val = d < BCorder ? boundaryKappa * m : m;
            mat(d, d) = val;
        }
    });
#endif

    /// @note ground collision
    if (enableGround) {
        for (auto &primHandle : prims) {
            if (primHandle.isBoundary()) // skip soft boundary
                continue;
            const auto &svs = primHandle.getSurfVerts();
            cudaPol(range(svs.size()),
                    [vtemp = proxy<space>({}, vtemp), svtemp = proxy<space>({}, primHandle.svtemp),
                     spmat = view<space>(linsys.spmat), svs = proxy<space>({}, svs), gn = s_groundNormal,
                     dHat2 = dHat * dHat, kappa = kappa, svOffset = primHandle.svOffset] ZS_LAMBDA(int svi) mutable {
                        const auto vi = svs("inds", svi, int_c) + svOffset;
                        auto x = vtemp.pack<3>("xn", vi);
                        auto dist = gn.dot(x);
                        auto dist2 = dist * dist;
                        auto t = dist2 - dHat2;
                        auto g_b = t * zs::log(dist2 / dHat2) * -2 - (t * t) / dist2;
                        auto H_b = (zs::log(dist2 / dHat2) * -2.0 - t * 4.0 / dist2) + 1.0 / (dist2 * dist2) * (t * t);
                        if (dist2 < dHat2) {
                            auto grad = -gn * (kappa * g_b * 2 * dist);
                            for (int d = 0; d != 3; ++d)
                                atomic_add(exec_cuda, &vtemp("grad", d, vi), grad(d));
                        }

                        auto param = 4 * H_b * dist2 + 2 * g_b;
                        auto hess = mat3::zeros();
                        if (dist2 < dHat2 && param > 0) {
                            auto nn = dyadic_prod(gn, gn);
                            hess = (kappa * param) * nn;
                        }

                        // make_pd(hess);
                        auto loc = spmat._ptrs[vi];
                        auto &mat = spmat._vals[loc];
                        for (int r = 0; r != 3; ++r) {
                            for (int c = 0; c != 3; ++c) {
                                mat(r, c) += hess(r, c);
                            }
                        }
                    });

            if (s_enableFriction)
                if (fricMu != 0) {
                    cudaPol(range(svs.size()),
                            [vtemp = proxy<space>({}, vtemp), svtemp = proxy<space>({}, primHandle.svtemp),
                             spmat = view<space>(linsys.spmat), svs = proxy<space>({}, svs), epsvh = epsv * dt,
                             gn = s_groundNormal, fricMu = fricMu,
                             svOffset = primHandle.svOffset] ZS_LAMBDA(int svi) mutable {
                                const auto vi = svs("inds", svi, int_c) + svOffset;
                                auto dx = vtemp.pack<3>("xn", vi) - vtemp.pack<3>("xhat", vi);
                                auto fn = svtemp("fn", svi);
                                if (fn == 0) {
                                    return;
                                }
                                auto coeff = fn * fricMu;
                                auto relDX = dx - gn.dot(dx) * gn;
                                auto relDXNorm2 = relDX.l2NormSqr();
                                auto relDXNorm = zs::sqrt(relDXNorm2);
                                vec3 grad{};
                                if (relDXNorm2 > epsvh * epsvh)
                                    grad = -relDX * (coeff / relDXNorm);
                                else
                                    grad = -relDX * (coeff / epsvh);
                                for (int d = 0; d != 3; ++d)
                                    atomic_add(exec_cuda, &vtemp("grad", d, vi), grad(d));

                                auto hess = mat3::zeros();
                                if (relDXNorm2 > epsvh * epsvh) {
                                    zs::vec<T, 2, 2> mat{
                                        relDX[0] * relDX[0] * -coeff / relDXNorm2 / relDXNorm + coeff / relDXNorm,
                                        relDX[0] * relDX[2] * -coeff / relDXNorm2 / relDXNorm,
                                        relDX[0] * relDX[2] * -coeff / relDXNorm2 / relDXNorm,
                                        relDX[2] * relDX[2] * -coeff / relDXNorm2 / relDXNorm + coeff / relDXNorm};
                                    make_pd(mat);
                                    hess(0, 0) = mat(0, 0);
                                    hess(0, 2) = mat(0, 1);
                                    hess(2, 0) = mat(1, 0);
                                    hess(2, 2) = mat(1, 1);
                                } else {
                                    hess(0, 0) = coeff / epsvh;
                                    hess(2, 2) = coeff / epsvh;
                                }
                                auto loc = spmat._ptrs[vi];
                                auto &mat = spmat._vals[loc];
                                for (int r = 0; r != 3; ++r) {
                                    for (int c = 0; c != 3; ++c) {
                                        mat(r, c) += hess(r, c);
                                    }
                                }
                            });
                }
        }
    }

    /// @note bending
    for (auto &primHandle : prims) {
        if (primHandle.hasBendingConstraints()) {
            auto &bedges = *primHandle.bendingEdgesPtr;
            cudaPol(range(bedges.size()), [bedges = view<space>({}, bedges), spmat = view<space>(linsys.spmat),
                                           vtemp = view<space>({}, vtemp), dt2 = dt * dt,
                                           vOffset = primHandle.vOffset] __device__(int i) mutable {
                auto stcl = bedges.pack(dim_c<4>, "inds", i, int_c) + vOffset;

                int BCorder[4];
#pragma unroll
                for (int i = 0; i != 4; ++i)
                    BCorder[i] = vtemp("BCorder", stcl[i]);
                if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3 && BCorder[3] == 3)
                    return;
                auto e = bedges("e", i);
                auto h = bedges("h", i);
                auto k = bedges("k", i);
                auto ra = bedges("ra", i);
                auto x0 = vtemp.pack(dim_c<3>, "xn", stcl[0]);
                auto x1 = vtemp.pack(dim_c<3>, "xn", stcl[1]);
                auto x2 = vtemp.pack(dim_c<3>, "xn", stcl[2]);
                auto x3 = vtemp.pack(dim_c<3>, "xn", stcl[3]);
                auto theta = dihedral_angle(x0, x1, x2, x3);

                auto localGrad = dihedral_angle_gradient(x0, x1, x2, x3);
                auto grad = -localGrad * dt2 * k * 2 * (theta - ra) * e / h;
                for (int j = 0; j != 4; ++j)
                    for (int d = 0; d != 3; ++d)
                        atomic_add(exec_cuda, &vtemp("grad", d, stcl[j]), grad(j * 3 + d));

                // rotate and project
                auto H = (dihedral_angle_hessian(x0, x1, x2, x3) * (theta - ra) + dyadic_prod(localGrad, localGrad)) *
                         k * 2 * e / h;
                make_pd(H);
                H *= dt2;

                // 12 * 12 = 16 * 9
                for (int vi = 0; vi < 4; ++vi) {
                    auto i = stcl[vi];
                    for (int vj = 0; vj < 4; ++vj) {
                        auto j = stcl[vj];
                        if (i > j)
                            continue;
                        auto loc = spmat.locate(i, j, true_c);
                        auto &mat = spmat._vals[loc];
                        for (int r = 0; r != 3; ++r) {
                            for (int c = 0; c != 3; ++c) {
                                atomic_add(exec_cuda, &mat(r, c), H(vi * 3 + r, vj * 3 + c));
                            }
                        }
                    }
                }
            });
        }
    }
    /// @note elasticity
    for (auto &primHandle : prims) {
        match([&](auto &elasticModel) {
            computeElasticGradientAndHessianImpl(cudaPol, gTag, vtemp, primHandle, elasticModel, dt, spmat);
        })(primHandle.getModels().getElasticModel());
    }
    for (auto &primHandle : auxPrims) {
        using ModelT = RM_CVREF_T(primHandle.getModels().getElasticModel());
        const ModelT &model = primHandle.modelsPtr ? primHandle.getModels().getElasticModel() : ModelT{};
        match([&](auto &elasticModel) {
            computeElasticGradientAndHessianImpl(cudaPol, gTag, vtemp, primHandle, elasticModel, dt, spmat);
        })(model);
    }
}

void UnifiedIPCSystem::updateDynamicHessian(zs::CudaExecutionPolicy &pol, const zs::SmallString &gTag) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    auto &hess2 = linsys.hess2;
    auto &hess3 = linsys.hess3;
    auto &hess4 = linsys.hess4;
    hess2.reset(false, 0); // overwrite style
    hess3.reset(false, 0);
    hess4.reset(false, 0);
    if (enableContact) {
        updateBarrierGradientAndHessian(pol, gTag);

        if (s_enableFriction) {
            if (s_enableSelfFriction) {
                if (fricMu != 0) {
                    updateFrictionBarrierGradientAndHessian(pol, gTag);
                } //
            }     // enable self friction, fricmu
        }         // enable friction
    }             // enable contact
}

void UnifiedIPCSystem::prepareDiagonalPreconditioner(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr execspace_e space = execspace_e::cuda;
    constexpr auto execTag = wrapv<space>{};
    using T = typename RM_CVREF_T(linsys)::T;
    using vec3 = zs::vec<T, 3>;
    {
        const auto &spmat = linsys.spmat;
        /// should group by 8, not 3
        pol(range(spmat.outerSize() * 3), [vtemp = view<space>({}, vtemp), // dxOffset = vtemp.getPropertyOffset(dxTag),
                                           spmat = proxy<space>(spmat)] ZS_LAMBDA(int tid) mutable {
            auto row = tid / 3;
            auto d = tid % 3;
            auto mat = spmat._vals[spmat._ptrs[row]];
            vtemp("P", d, row) += mat(0, d);
            vtemp("P", 3 + d, row) += mat(1, d);
            vtemp("P", 6 + d, row) += mat(2, d);
        });
    }

    // hess2
    const auto &hess2 = linsys.hess2;
    const auto &hess3 = linsys.hess3;
    const auto &hess4 = linsys.hess4;
    pol(Collapse{hess2.count() * 3},
        [execTag, hess2 = proxy<space>(hess2), vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int ei) mutable {
            auto d = ei % 3;
            ei /= 3;
            auto inds = hess2.inds[ei];
            auto mat = hess2.hess[ei];
            for (int k = 0; k != 2; ++k) {
                auto row = inds[k];
                auto offset = k * 3;
                atomic_add(execTag, &vtemp("P", d, row), mat(offset + 0, offset + d));
                atomic_add(execTag, &vtemp("P", 3 + d, row), mat(offset + 1, offset + d));
                atomic_add(execTag, &vtemp("P", 6 + d, row), mat(offset + 2, offset + d));
            }
        });
    // hess3
    {
        pol(Collapse{hess3.count() * 3},
            [execTag, hess3 = proxy<space>(hess3), vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int ei) mutable {
                auto d = ei % 3;
                ei /= 3;
                auto inds = hess3.inds[ei];
                auto mat = hess3.hess[ei];
                for (int k = 0; k != 3; ++k) {
                    auto row = inds[k];
                    auto offset = k * 3;
                    atomic_add(execTag, &vtemp("P", d, row), mat(offset + 0, offset + d));
                    atomic_add(execTag, &vtemp("P", 3 + d, row), mat(offset + 1, offset + d));
                    atomic_add(execTag, &vtemp("P", 6 + d, row), mat(offset + 2, offset + d));
                }
            });
    }
    // hess4
    {
        pol(Collapse{hess4.count() * 3},
            [execTag, hess4 = proxy<space>(hess4), vtemp = proxy<space>({}, vtemp)] ZS_LAMBDA(int ei) mutable {
                auto d = ei % 3;
                ei /= 3;
                auto inds = hess4.inds[ei];
                auto mat = hess4.hess[ei];
                for (int k = 0; k != 4; ++k) {
                    auto row = inds[k];
                    auto offset = k * 3;
                    atomic_add(execTag, &vtemp("P", d, row), mat(offset + 0, offset + d));
                    atomic_add(execTag, &vtemp("P", 3 + d, row), mat(offset + 1, offset + d));
                    atomic_add(execTag, &vtemp("P", 6 + d, row), mat(offset + 2, offset + d));
                }
            });
    }
    // timer.tock("multiply takes");
}

/// elasticity
template <typename Model>
void computeElasticGradientAndHessianImpl(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString &gTag,
                                          typename UnifiedIPCSystem::dtiles_t &vtemp,
                                          typename UnifiedIPCSystem::PrimitiveHandle &primHandle, const Model &model,
                                          typename UnifiedIPCSystem::T dt, bool projectDBC, bool includeHessian) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    using mat3 = typename UnifiedIPCSystem::mat3;
    using vec3 = typename UnifiedIPCSystem::vec3;
    using T = typename UnifiedIPCSystem::T;
    if (primHandle.category == ZenoParticles::curve) {
        if (primHandle.isBoundary() && !primHandle.isAuxiliary())
            return;
        /// ref: Fast Simulation of Mass-Spring Systems
        /// credits: Tiantian Liu
        cudaPol(zs::range(primHandle.getEles().size()),
                [vtemp = proxy<space>({}, vtemp), etemp = proxy<space>({}, primHandle.etemp),
                 eles = proxy<space>({}, primHandle.getEles()), model, gTag, dt = dt, projectDBC = projectDBC,
                 vOffset = primHandle.vOffset, includeHessian,
                 n = primHandle.getEles().size()] __device__(int ei) mutable {
                    auto inds = eles.pack(dim_c<2>, "inds", ei, int_c) + vOffset;
                    int BCorder[2];
                    for (int i = 0; i != 2; ++i) {
                        BCorder[i] = vtemp("BCorder", inds[i]);
                    }

                    if (BCorder[0] == 3 && BCorder[1] == 3) {
                        etemp.tuple<6 * 6>("He", ei) = zs::vec<T, 6, 6>::zeros();
                        return;
                    }

                    auto vole = eles("vol", ei);
                    auto k = eles("k", ei);
                    auto rl = eles("rl", ei);

                    vec3 xs[2] = {vtemp.pack(dim_c<3>, "xn", inds[0]), vtemp.pack(dim_c<3>, "xn", inds[1])};
                    auto xij = xs[1] - xs[0];
                    auto lij = xij.norm();
                    auto dij = xij / lij;
                    auto gij = k * (lij - rl) * dij;

                    // gradient
                    auto vfdt2 = gij * (dt * dt) * vole;
                    for (int d = 0; d != 3; ++d) {
                        atomic_add(exec_cuda, &vtemp(gTag, d, inds[0]), (T)vfdt2(d));
                        atomic_add(exec_cuda, &vtemp(gTag, d, inds[1]), (T)-vfdt2(d));
                    }

                    if (!includeHessian)
                        return;
                    auto H = zs::vec<T, 6, 6>::zeros();
                    auto K = k * (mat3::identity() - rl / lij * (mat3::identity() - dyadic_prod(dij, dij)));
                    // make_pd(K);  // symmetric semi-definite positive, not
                    // necessary

                    for (int i = 0; i != 3; ++i)
                        for (int j = 0; j != 3; ++j) {
                            H(i, j) = K(i, j);
                            H(i, 3 + j) = -K(i, j);
                            H(3 + i, j) = -K(i, j);
                            H(3 + i, 3 + j) = K(i, j);
                        }
                    H *= dt * dt * vole;

                    // rotate and project
                    etemp.tuple(dim_c<6, 6>, "He", ei) = H;
                    for (int vi = 0; vi != 2; ++vi) {
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j) {
                                atomic_add(exec_cuda, &vtemp("P", i * 3 + j, inds[vi]), H(vi * 3 + i, vi * 3 + j));
                            }
                    }
                });
    } else if (primHandle.category == ZenoParticles::surface) {
        if (primHandle.isBoundary())
            return;
        cudaPol(zs::range(primHandle.getEles().size()),
                [vtemp = proxy<space>({}, vtemp), etemp = proxy<space>({}, primHandle.etemp),
                 eles = proxy<space>({}, primHandle.getEles()), model, gTag, dt = dt, projectDBC = projectDBC,
                 vOffset = primHandle.vOffset, includeHessian] __device__(int ei) mutable {
                    auto IB = eles.template pack<2, 2>("IB", ei);
                    auto inds = eles.pack(dim_c<3>, "inds", ei, int_c) + vOffset;
                    auto vole = eles("vol", ei);
                    vec3 xs[3] = {vtemp.pack(dim_c<3>, "xn", inds[0]), vtemp.pack(dim_c<3>, "xn", inds[1]),
                                  vtemp.pack(dim_c<3>, "xn", inds[2])};
                    auto x1x0 = xs[1] - xs[0];
                    auto x2x0 = xs[2] - xs[0];

                    int BCorder[3];
                    for (int i = 0; i != 3; ++i) {
                        BCorder[i] = vtemp("BCorder", inds[i]);
                    }
                    zs::vec<T, 9, 9> H;
                    if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3) {
                        etemp.tuple(dim_c<9, 9>, "He", ei) = H.zeros();
                        return;
                    }

                    zs::vec<T, 3, 2> Ds{x1x0[0], x2x0[0], x1x0[1], x2x0[1], x1x0[2], x2x0[2]};
                    auto F = Ds * IB;

                    auto dFdX = dFdXMatrix(IB, wrapv<3>{});
                    auto dFdXT = dFdX.transpose();
                    auto f0 = col(F, 0);
                    auto f1 = col(F, 1);
                    auto f0Norm = zs::sqrt(f0.l2NormSqr());
                    auto f1Norm = zs::sqrt(f1.l2NormSqr());
                    auto f0Tf1 = f0.dot(f1);
                    zs::vec<T, 3, 2> Pstretch, Pshear;
                    for (int d = 0; d != 3; ++d) {
                        Pstretch(d, 0) = 2 * (1 - 1 / f0Norm) * F(d, 0);
                        Pstretch(d, 1) = 2 * (1 - 1 / f1Norm) * F(d, 1);
                        Pshear(d, 0) = 2 * f0Tf1 * f1(d);
                        Pshear(d, 1) = 2 * f0Tf1 * f0(d);
                    }
                    auto vecP = flatten(model.mu * Pstretch + (model.mu * 0.3) * Pshear);
                    auto vfdt2 = -vole * (dFdXT * vecP) * (dt * dt);

                    for (int i = 0; i != 3; ++i) {
                        auto vi = inds[i];
                        for (int d = 0; d != 3; ++d)
                            atomic_add(exec_cuda, &vtemp(gTag, d, vi), (T)vfdt2(i * 3 + d));
                    }

                    if (!includeHessian)
                        return;
                    /// ref: A Finite Element Formulation of Baraff-Witkin Cloth
                    // suggested by huang kemeng
                    auto stretchHessian = [&F, &model]() {
                        auto H = zs::vec<T, 6, 6>::zeros();
                        const zs::vec<T, 2> u{1, 0};
                        const zs::vec<T, 2> v{0, 1};
                        const T I5u = (F * u).l2NormSqr();
                        const T I5v = (F * v).l2NormSqr();
                        const T invSqrtI5u = (T)1 / zs::sqrt(I5u);
                        const T invSqrtI5v = (T)1 / zs::sqrt(I5v);

                        H(0, 0) = H(1, 1) = H(2, 2) = zs::max(1 - invSqrtI5u, (T)0);
                        H(3, 3) = H(4, 4) = H(5, 5) = zs::max(1 - invSqrtI5v, (T)0);

                        const auto fu = col(F, 0).normalized();
                        const T uCoeff = (1 - invSqrtI5u >= 0) ? invSqrtI5u : (T)1;
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j)
                                H(i, j) += uCoeff * fu(i) * fu(j);

                        const auto fv = col(F, 1).normalized();
                        const T vCoeff = (1 - invSqrtI5v >= 0) ? invSqrtI5v : (T)1;
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j)
                                H(3 + i, 3 + j) += vCoeff * fv(i) * fv(j);

                        H *= model.mu;
                        return H;
                    };
                    auto shearHessian = [&F, &model]() {
                        using mat6 = zs::vec<T, 6, 6>;
                        auto H = mat6::zeros();
                        const zs::vec<T, 2> u{1, 0};
                        const zs::vec<T, 2> v{0, 1};
                        const T I6 = (F * u).dot(F * v);
                        const T signI6 = I6 >= 0 ? 1 : -1;

                        H(3, 0) = H(4, 1) = H(5, 2) = H(0, 3) = H(1, 4) = H(2, 5) = (T)1;

                        const auto g_ = F * (dyadic_prod(u, v) + dyadic_prod(v, u));
                        zs::vec<T, 6> g{};
                        for (int j = 0, offset = 0; j != 2; ++j) {
                            for (int i = 0; i != 3; ++i)
                                g(offset++) = g_(i, j);
                        }

                        const T I2 = F.l2NormSqr();
                        const T lambda0 = (T)0.5 * (I2 + zs::sqrt(I2 * I2 + (T)12 * I6 * I6));

                        const zs::vec<T, 6> q0 = (I6 * H * g + lambda0 * g).normalized();

                        auto t = mat6::identity();
                        t = 0.5 * (t + signI6 * H);

                        const zs::vec<T, 6> Tq = t * q0;
                        const auto normTq = Tq.l2NormSqr();

                        mat6 dPdF =
                            zs::abs(I6) * (t - (dyadic_prod(Tq, Tq) / normTq)) + lambda0 * (dyadic_prod(q0, q0));
                        dPdF *= (model.mu * 0.3);
                        return dPdF;
                    };
                    auto He = stretchHessian() + shearHessian();
                    H = dFdX.transpose() * He * dFdX;
                    H *= dt * dt * vole;

                    // rotate and project
                    etemp.tuple(dim_c<9, 9>, "He", ei) = H;
                    for (int vi = 0; vi != 3; ++vi) {
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j) {
                                atomic_add(exec_cuda, &vtemp("P", i * 3 + j, inds[vi]), H(vi * 3 + i, vi * 3 + j));
                            }
                    }
                });
    } else if (primHandle.category == ZenoParticles::tet)
        cudaPol(zs::range(primHandle.getEles().size()),
                [vtemp = proxy<space>({}, vtemp), etemp = proxy<space>({}, primHandle.etemp),
                 eles = proxy<space>({}, primHandle.getEles()), model, gTag, dt = dt, projectDBC = projectDBC,
                 vOffset = primHandle.vOffset, includeHessian] __device__(int ei) mutable {
                    auto IB = eles.pack(dim_c<3, 3>, "IB", ei);
                    auto inds = eles.pack(dim_c<4>, "inds", ei, int_c) + vOffset;
                    auto vole = eles("vol", ei);
                    vec3 xs[4] = {vtemp.pack<3>("xn", inds[0]), vtemp.pack<3>("xn", inds[1]),
                                  vtemp.pack<3>("xn", inds[2]), vtemp.pack<3>("xn", inds[3])};

                    int BCorder[4];
                    for (int i = 0; i != 4; ++i) {
                        BCorder[i] = vtemp("BCorder", inds[i]);
                    }
                    zs::vec<T, 12, 12> H;
                    if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3 && BCorder[3] == 3) {
                        etemp.tuple<12 * 12>("He", ei) = H.zeros();
                        return;
                    }
                    mat3 F{};
                    {
                        auto x1x0 = xs[1] - xs[0];
                        auto x2x0 = xs[2] - xs[0];
                        auto x3x0 = xs[3] - xs[0];
                        auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1], x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                        F = Ds * IB;
                    }
                    auto P = model.first_piola(F);
                    auto vecP = flatten(P);
                    auto dFdX = dFdXMatrix(IB);
                    auto dFdXT = dFdX.transpose();
                    auto vfdt2 = -vole * (dFdXT * vecP) * dt * dt;

                    for (int i = 0; i != 4; ++i) {
                        auto vi = inds[i];
                        for (int d = 0; d != 3; ++d)
                            atomic_add(exec_cuda, &vtemp(gTag, d, vi), (T)vfdt2(i * 3 + d));
                    }

                    if (!includeHessian)
                        return;
                    auto Hq = model.first_piola_derivative(F, true_c);
                    H = dFdXT * Hq * dFdX * vole * dt * dt;

                    // rotate and project
                    etemp.tuple<12 * 12>("He", ei) = H;
                    for (int vi = 0; vi != 4; ++vi) {
                        for (int i = 0; i != 3; ++i)
                            for (int j = 0; j != 3; ++j) {
                                atomic_add(exec_cuda, &vtemp("P", i * 3 + j, inds[vi]), H(vi * 3 + i, vi * 3 + j));
                            }
                    }
                });
}

void UnifiedIPCSystem::computeElasticGradientAndHessian(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString &gTag,
                                                        bool includeHessian) {
    using namespace zs;
    for (auto &primHandle : prims) {
        match([&](auto &elasticModel) {
            computeElasticGradientAndHessianImpl(cudaPol, gTag, vtemp, primHandle, elasticModel, dt, projectDBC,
                                                 includeHessian);
        })(primHandle.getModels().getElasticModel());
    }
    for (auto &primHandle : auxPrims) {
        using ModelT = RM_CVREF_T(primHandle.getModels().getElasticModel());
        const ModelT &model = primHandle.modelsPtr ? primHandle.getModels().getElasticModel() : ModelT{};
        match([&](auto &elasticModel) {
            computeElasticGradientAndHessianImpl(cudaPol, gTag, vtemp, primHandle, elasticModel, dt, projectDBC,
                                                 includeHessian);
        })(model);
    }
}

void UnifiedIPCSystem::computeBendingGradientAndHessian(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString &gTag,
                                                        bool includeHessian) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    for (auto &primHandle : prims) {
        if (!primHandle.hasBendingConstraints())
            continue;
        auto &btemp = primHandle.btemp;
        auto &bedges = *primHandle.bendingEdgesPtr;
        cudaPol(range(btemp.size()), [bedges = proxy<space>({}, bedges), btemp = proxy<space>(btemp),
                                      vtemp = proxy<space>({}, vtemp), dt2 = dt * dt, projectDBC = projectDBC,
                                      vOffset = primHandle.vOffset, includeHessian] __device__(int i) mutable {
            auto stcl = bedges.pack(dim_c<4>, "inds", i, int_c) + vOffset;

            int BCorder[4];
            for (int i = 0; i != 4; ++i) {
                BCorder[i] = vtemp("BCorder", stcl[i]);
            }
            if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3 && BCorder[3] == 3) {
                btemp.tuple(dim_c<12 * 12>, 0, i) = zs::vec<T, 12, 12>::zeros();
                return;
            }

            auto e = bedges("e", i);
            auto h = bedges("h", i);
            auto k = bedges("k", i);
            auto ra = bedges("ra", i);
            auto x0 = vtemp.pack(dim_c<3>, "xn", stcl[0]);
            auto x1 = vtemp.pack(dim_c<3>, "xn", stcl[1]);
            auto x2 = vtemp.pack(dim_c<3>, "xn", stcl[2]);
            auto x3 = vtemp.pack(dim_c<3>, "xn", stcl[3]);
            auto theta = dihedral_angle(x0, x1, x2, x3);

            auto localGrad = dihedral_angle_gradient(x0, x1, x2, x3);
            auto grad = -localGrad * dt2 * k * 2 * (theta - ra) * e / h;
            for (int j = 0; j != 4; ++j)
                for (int d = 0; d != 3; ++d)
                    atomic_add(exec_cuda, &vtemp("grad", d, stcl[j]), grad(j * 3 + d));

            if (!includeHessian)
                return;

            // rotate and project
            auto H = (dihedral_angle_hessian(x0, x1, x2, x3) * (theta - ra) + dyadic_prod(localGrad, localGrad)) * k *
                     2 * e / h;
            make_pd(H);
            H *= dt2;

            btemp.tuple(dim_c<12 * 12>, 0, i) = H;
            for (int vi = 0; vi != 4; ++vi) {
                for (int i = 0; i != 3; ++i)
                    for (int j = 0; j != 3; ++j) {
                        atomic_add(exec_cuda, &vtemp("P", i * 3 + j, stcl[vi]), H(vi * 3 + i, vi * 3 + j));
                    }
            }
        });
    }
}

void UnifiedIPCSystem::computeBoundaryBarrierGradientAndHessian(zs::CudaExecutionPolicy &pol, bool includeHessian) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    for (auto &primHandle : prims) {
        if (primHandle.isBoundary()) // skip soft boundary
            continue;
        const auto &svs = primHandle.getSurfVerts();
        pol(range(svs.size()),
            [vtemp = proxy<space>({}, vtemp), svtemp = proxy<space>({}, primHandle.svtemp), svs = proxy<space>({}, svs),
             gn = s_groundNormal, dHat2 = dHat * dHat, kappa = kappa, projectDBC = projectDBC, includeHessian,
             svOffset = primHandle.svOffset] ZS_LAMBDA(int svi) mutable {
                const auto vi = svs("inds", svi, int_c) + svOffset;
                auto x = vtemp.pack<3>("xn", vi);
                auto dist = gn.dot(x);
                auto dist2 = dist * dist;
                auto t = dist2 - dHat2;
                auto g_b = t * zs::log(dist2 / dHat2) * -2 - (t * t) / dist2;
                auto H_b = (zs::log(dist2 / dHat2) * -2.0 - t * 4.0 / dist2) + 1.0 / (dist2 * dist2) * (t * t);
                if (dist2 < dHat2) {
                    auto grad = -gn * (kappa * g_b * 2 * dist);
                    for (int d = 0; d != 3; ++d)
                        atomic_add(exec_cuda, &vtemp("grad", d, vi), grad(d));
                }

                if (!includeHessian)
                    return;
                auto param = 4 * H_b * dist2 + 2 * g_b;
                auto hess = mat3::zeros();
                if (dist2 < dHat2 && param > 0) {
                    auto nn = dyadic_prod(gn, gn);
                    hess = (kappa * param) * nn;
                }

                // make_pd(hess);
                int BCorder[1] = {(int)vtemp("BCorder", vi)};
                svtemp.tuple<9>("H", svi) = hess;
                for (int i = 0; i != 3; ++i)
                    for (int j = 0; j != 3; ++j) {
                        atomic_add(exec_cuda, &vtemp("P", i * 3 + j, vi), hess(i, j));
                    }
            });

        if (s_enableFriction)
            if (fricMu != 0) {
                pol(range(svs.size()), [vtemp = proxy<space>({}, vtemp), svtemp = proxy<space>({}, primHandle.svtemp),
                                        svs = proxy<space>({}, svs), epsvh = epsv * dt, gn = s_groundNormal,
                                        fricMu = fricMu, projectDBC = projectDBC, includeHessian,
                                        svOffset = primHandle.svOffset] ZS_LAMBDA(int svi) mutable {
                    const auto vi = svs("inds", svi, int_c) + svOffset;
                    auto dx = vtemp.pack<3>("xn", vi) - vtemp.pack<3>("xhat", vi);
                    auto fn = svtemp("fn", svi);
                    if (fn == 0) {
                        return;
                    }
                    auto coeff = fn * fricMu;
                    auto relDX = dx - gn.dot(dx) * gn;
                    auto relDXNorm2 = relDX.l2NormSqr();
                    auto relDXNorm = zs::sqrt(relDXNorm2);

                    vec3 grad{};
                    if (relDXNorm2 > epsvh * epsvh)
                        grad = -relDX * (coeff / relDXNorm);
                    else
                        grad = -relDX * (coeff / epsvh);
                    for (int d = 0; d != 3; ++d)
                        atomic_add(exec_cuda, &vtemp("grad", d, vi), grad(d));

                    if (!includeHessian)
                        return;

                    auto hess = mat3::zeros();
                    if (relDXNorm2 > epsvh * epsvh) {
                        zs::vec<T, 2, 2> mat{relDX[0] * relDX[0] * -coeff / relDXNorm2 / relDXNorm + coeff / relDXNorm,
                                             relDX[0] * relDX[2] * -coeff / relDXNorm2 / relDXNorm,
                                             relDX[0] * relDX[2] * -coeff / relDXNorm2 / relDXNorm,
                                             relDX[2] * relDX[2] * -coeff / relDXNorm2 / relDXNorm + coeff / relDXNorm};
                        make_pd(mat);
                        hess(0, 0) = mat(0, 0);
                        hess(0, 2) = mat(0, 1);
                        hess(2, 0) = mat(1, 0);
                        hess(2, 2) = mat(1, 1);
                    } else {
                        hess(0, 0) = coeff / epsvh;
                        hess(2, 2) = coeff / epsvh;
                    }

                    int BCorder[1] = {(int)vtemp("BCorder", vi)};
                    svtemp.tuple(dim_c<9>, "H", svi) = svtemp.pack(dim_c<3, 3>, "H", svi) + hess;
                    for (int i = 0; i != 3; ++i)
                        for (int j = 0; j != 3; ++j) {
                            atomic_add(exec_cuda, &vtemp("P", i * 3 + j, vi), hess(i, j));
                        }
                });
            }
    }
    return;
}

UnifiedIPCSystem::T UnifiedIPCSystem::infNorm(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString tag) {
    using namespace zs;
    using T = typename UnifiedIPCSystem::T;
    constexpr auto space = execspace_e::cuda;
    auto &vertData = vtemp;
    auto &res = temp;
    res.resize(count_warps(vertData.size()));
    res.reset(0);
    cudaPol(range((vertData.size() + 31) / 32 * 32),
            [data = proxy<space>({}, vertData), res = proxy<space>(res), n = vertData.size(),
             offset = vertData.getPropertyOffset(tag)] __device__(int pi) mutable {
                T val = 0;
                if (pi < n) {
                    auto v = data.pack<3>(offset, pi);
                    val = v.abs().max();
                }

#if __CUDA_ARCH__ >= 800
                auto tile = zs::cg::tiled_partition<32>(zs::cg::this_thread_block());
                auto ret = zs::cg::reduce(tile, val, zs::cg::greater<T>());
                if (tile.thread_rank() == 0)
                    res[pi / 32] = ret;
#else
        auto [mask, numValid] = warp_mask(pi, n);
        auto locid = threadIdx.x & 31;
        for (int stride = 1; stride < 32; stride <<= 1) {
            auto tmp = __shfl_down_sync(mask, val, stride);
            if (locid + stride < numValid)
                val = zs::max(val, tmp);
        }
        if (locid == 0)
            res[pi / 32] = val;
#endif
            });
    return reduce(cudaPol, res, thrust::maximum<T>{});
}
UnifiedIPCSystem::T UnifiedIPCSystem::dot(zs::CudaExecutionPolicy &cudaPol, const zs::SmallString tag0,
                                          const zs::SmallString tag1) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    auto &vertData = vtemp;
    auto &res = temp;
    res.resize(count_warps(vertData.size()));
    cudaPol(range((vertData.size() + 31) / 32 * 32),
            [data = proxy<space>({}, vertData), res = proxy<space>(res), n = vertData.size(),
             offset0 = vertData.getPropertyOffset(tag0),
             offset1 = vertData.getPropertyOffset(tag1)] __device__(int pi) mutable {
                T val = 0;
                if (pi < n) {
                    auto v0 = data.pack(dim_c<3>, offset0, pi);
                    auto v1 = data.pack(dim_c<3>, offset1, pi);
                    val = v0.dot(v1);
                }
        // reduce_to(pi, n, v, res[pi / 32]);

#if __CUDA_ARCH__ >= 800
                auto tile = zs::cg::tiled_partition<32>(zs::cg::this_thread_block());
                auto ret = zs::cg::reduce(tile, val, zs::cg::plus<T>());
                if (tile.thread_rank() == 0)
                    res[pi / 32] = ret;
#else
        auto [mask, numValid] = warp_mask(pi, n);
        auto locid = threadIdx.x & 31;
        for (int stride = 1; stride < 32; stride <<= 1) {
            auto tmp = __shfl_down_sync(mask, val, stride);
            if (locid + stride < numValid)
                val = val + tmp;
        }
        if (locid == 0)
            res[pi / 32] = val;
#endif
            });
    return reduce(cudaPol, res, thrust::plus<double>{});
}

} // namespace zeno