#include "hip/hip_runtime.h"
#include "RapidCloth.cuh"
#include "Structures.hpp"
#include "TopoUtils.hpp"
#include "zensim/geometry/Distance.hpp"
#include <zeno/types/ListObject.h>

#define RETRIEVE_OBJECT_PTRS(T, STR)                                                  \
    ([this](const std::string_view str) {                                             \
        std::vector<T *> objPtrs{};                                                   \
        if (has_input<T>(str.data()))                                                 \
            objPtrs.push_back(get_input<T>(str.data()).get());                        \
        else if (has_input<zeno::ListObject>(str.data())) {                           \
            auto &objSharedPtrLists = *get_input<zeno::ListObject>(str.data());       \
            for (auto &&objSharedPtr : objSharedPtrLists.get())                       \
                if (auto ptr = dynamic_cast<T *>(objSharedPtr.get()); ptr != nullptr) \
                    objPtrs.push_back(ptr);                                           \
        }                                                                             \
        return objPtrs;                                                               \
    })(STR);

namespace zeno {

RapidClothSystem::PrimitiveHandle::PrimitiveHandle(std::shared_ptr<tiles_t> elesPtr_, ZenoParticles::category_e category)
    : zsprimPtr{}, modelsPtr{}, vertsPtr{}, elesPtr{elesPtr_},
      etemp{elesPtr_->get_allocator(), {{"He", 6 * 6}}, elesPtr_->size()}, surfTrisPtr{}, surfEdgesPtr{},
      surfVertsPtr{}, svtemp{}, vOffset{0}, sfOffset{0}, seOffset{0}, svOffset{0}, category{category} {
    ;
}
RapidClothSystem::PrimitiveHandle::PrimitiveHandle(ZenoParticles &zsprim, Ti &vOffset, Ti &sfOffset, Ti &seOffset,
                                                  Ti &svOffset, zs::wrapv<2>)
    : zsprimPtr{&zsprim, [](void *) {}}, modelsPtr{&zsprim.getModel(), [](void *) {}}, vertsPtr{&zsprim.getParticles(),
                                                                                                [](void *) {}},
      elesPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, etemp{zsprim.getQuadraturePoints().get_allocator(),
                                                                   {{"He", 6 * 6}},
                                                                   zsprim.numElements()},
      surfTrisPtr{&zsprim.getQuadraturePoints(), [](void *) {}},  // this is fake!
      surfEdgesPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, // all elements are surface edges
      surfVertsPtr{&zsprim[ZenoParticles::s_surfVertTag], [](void *) {}}, vOffset{vOffset},
      svtemp{zsprim.getQuadraturePoints().get_allocator(),
             {{"H", 3 * 3}, {"fn", 1}},
             zsprim[ZenoParticles::s_surfVertTag].size()},
      sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset}, category{zsprim.category} {
    if (category != ZenoParticles::curve)
        throw std::runtime_error("dimension of 2 but is not curve");
    vOffset += getVerts().size();
    // sfOffset += 0; // no surface triangles
    seOffset += getSurfEdges().size();
    svOffset += getSurfVerts().size();
}
RapidClothSystem::PrimitiveHandle::PrimitiveHandle(ZenoParticles &zsprim, Ti &vOffset, Ti &sfOffset, Ti &seOffset,
                                                  Ti &svOffset, zs::wrapv<3>)
    : zsprimPtr{&zsprim, [](void *) {}}, modelsPtr{&zsprim.getModel(), [](void *) {}}, vertsPtr{&zsprim.getParticles(),
                                                                                                [](void *) {}},
      elesPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, etemp{zsprim.getQuadraturePoints().get_allocator(),
                                                                   {{"He", 9 * 9}},
                                                                   zsprim.numElements()},
      surfTrisPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, surfEdgesPtr{&zsprim[ZenoParticles::s_surfEdgeTag],
                                                                              [](void *) {}},
      surfVertsPtr{&zsprim[ZenoParticles::s_surfVertTag], [](void *) {}}, vOffset{vOffset},
      svtemp{zsprim.getQuadraturePoints().get_allocator(),
             {{"H", 3 * 3}, {"fn", 1}},
             zsprim[ZenoParticles::s_surfVertTag].size()},
      sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset}, category{zsprim.category} {
    if (category != ZenoParticles::surface)
        throw std::runtime_error("dimension of 3 but is not surface");
    vOffset += getVerts().size();
    sfOffset += getSurfTris().size();
    seOffset += getSurfEdges().size();
    svOffset += getSurfVerts().size();
}
RapidClothSystem::PrimitiveHandle::PrimitiveHandle(ZenoParticles &zsprim, Ti &vOffset, Ti &sfOffset, Ti &seOffset,
                                                  Ti &svOffset, zs::wrapv<4>)
    : zsprimPtr{&zsprim, [](void *) {}}, modelsPtr{&zsprim.getModel(), [](void *) {}}, vertsPtr{&zsprim.getParticles(),
                                                                                                [](void *) {}},
      elesPtr{&zsprim.getQuadraturePoints(), [](void *) {}}, etemp{zsprim.getQuadraturePoints().get_allocator(),
                                                                   {{"He", 12 * 12}},
                                                                   zsprim.numElements()},
      surfTrisPtr{&zsprim[ZenoParticles::s_surfTriTag], [](void *) {}},
      surfEdgesPtr{&zsprim[ZenoParticles::s_surfEdgeTag], [](void *) {}},
      surfVertsPtr{&zsprim[ZenoParticles::s_surfVertTag], [](void *) {}}, vOffset{vOffset},
      svtemp{zsprim.getQuadraturePoints().get_allocator(),
             {{"H", 3 * 3}, {"fn", 1}},
             zsprim[ZenoParticles::s_surfVertTag].size()},
      sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset}, category{zsprim.category} {
    if (category != ZenoParticles::tet)
        throw std::runtime_error("dimension of 4 but is not tetrahedra");
    vOffset += getVerts().size();
    sfOffset += getSurfTris().size();
    seOffset += getSurfEdges().size();
    svOffset += getSurfVerts().size();
}
typename RapidClothSystem::T RapidClothSystem::PrimitiveHandle::maximumSurfEdgeLength(zs::CudaExecutionPolicy &pol,
                                                                                    zs::Vector<T> &temp) const {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    auto &verts = getVerts();
    auto &edges = getSurfEdges();
    temp.resize(edges.size());
    auto &edgeLengths = temp;
    pol(Collapse{edges.size()}, [edges = view<space>({}, edges), verts = view<space>({}, verts),
                                 edgeLengths = view<space>(edgeLengths)] ZS_LAMBDA(int ei) mutable {
        auto inds = edges.pack(dim_c<2>, "inds", ei, int_c);
        edgeLengths[ei] = (verts.pack<3>("x0", inds[0]) - verts.pack<3>("x0", inds[1])).norm();
    });
    auto tmp = reduce(pol, edgeLengths, thrust::maximum<T>());
    return tmp;
}
typename RapidClothSystem::T RapidClothSystem::PrimitiveHandle::averageNodalMass(zs::CudaExecutionPolicy &pol) const {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    if (zsprimPtr->hasMeta(s_meanMassTag))
        return zsprimPtr->readMeta(s_meanMassTag, zs::wrapt<T>{});
    auto &verts = getVerts();
    Vector<T> masses{verts.get_allocator(), verts.size()};
    pol(Collapse{verts.size()}, [verts = view<space>({}, verts), masses = view<space>(masses)] ZS_LAMBDA(
                                    int vi) mutable { masses[vi] = verts("m", vi); });
    auto tmp = reduce(pol, masses) / masses.size();
    zsprimPtr->setMeta(s_meanMassTag, tmp);
    return tmp;
}
typename RapidClothSystem::T RapidClothSystem::PrimitiveHandle::totalVolume(zs::CudaExecutionPolicy &pol) const {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    if (zsprimPtr->hasMeta(s_totalVolumeTag))
        return zsprimPtr->readMeta(s_totalVolumeTag, zs::wrapt<T>{});
    auto &eles = getEles();
    Vector<T> vols{eles.get_allocator(), eles.size()};
    pol(Collapse{eles.size()}, [eles = view<space>({}, eles), vols = view<space>(vols)] ZS_LAMBDA(int ei) mutable {
        vols[ei] = eles("vol", ei);
    });
    auto tmp = reduce(pol, vols);
    zsprimPtr->setMeta(s_totalVolumeTag, tmp);
    return tmp;
}

/// RapidClothSystem
typename RapidClothSystem::T RapidClothSystem::maximumSurfEdgeLength(zs::CudaExecutionPolicy &pol, bool includeBoundary) {
    using T = typename RapidClothSystem::T;
    T maxEdgeLength = 0;
    for (auto &&primHandle : prims) {
        if (primHandle.isBoundary())
            continue;
        if (auto tmp = primHandle.maximumSurfEdgeLength(pol, temp); tmp > maxEdgeLength)
            maxEdgeLength = tmp;
    }
    if (coVerts && includeBoundary) {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        // auto &verts = vtemp;
        auto &edges = *coEdges;
        temp.resize(edges.size());
        auto &edgeLengths = temp;
        pol(Collapse{edges.size()},
            [edges = view<space>({}, edges), verts = view<space>({}, vtemp), edgeLengths = view<space>(edgeLengths),
             coOffset = coOffset] ZS_LAMBDA(int ei) mutable {
                auto inds = edges.pack(dim_c<2>, "inds", ei, int_c) + coOffset;
                edgeLengths[ei] = (verts.pack<3>("x[k]", inds[0]) - verts.pack<3>("x[k]", inds[1])).norm();
            });
        if (auto tmp = reduce(pol, edgeLengths, thrust::maximum<T>()); tmp > maxEdgeLength)
            maxEdgeLength = tmp;
    }
    return maxEdgeLength;
}
typename RapidClothSystem::T RapidClothSystem::averageNodalMass(zs::CudaExecutionPolicy &pol) {
    using T = typename RapidClothSystem::T;
    T sumNodalMass = 0;
    int sumNodes = 0;
    for (auto &&primHandle : prims) {
        if (primHandle.isBoundary())
            continue;
        auto numNodes = primHandle.getVerts().size();
        sumNodes += numNodes;
        sumNodalMass += primHandle.averageNodalMass(pol) * numNodes;
    }
    if (sumNodes)
        return sumNodalMass / sumNodes;
    else
        return 0;
}
typename RapidClothSystem::T RapidClothSystem::totalVolume(zs::CudaExecutionPolicy &pol) {
    using T = typename RapidClothSystem::T;
    T sumVolume = 0;
    for (auto &&primHandle : prims) {
        if (primHandle.isBoundary())
            continue;
        sumVolume += primHandle.totalVolume(pol);
    }
    return sumVolume;
}

// TODO first
void RapidClothSystem::initialize(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    /// @brief cloth system surface topo construction
    stInds = tiles_t{vtemp.get_allocator(), {{"inds", 3}}, (std::size_t)sfOffset};
    seInds = tiles_t{vtemp.get_allocator(), {{"inds", 2}}, (std::size_t)seOffset};
    svInds = tiles_t{vtemp.get_allocator(), {{"inds", 1}}, (std::size_t)svOffset};
    ne = seInds.size(); 
    tempE.resize(ne); 
    nE.setVal(ne); 

    for (auto &primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        const auto &verts = primHandle.getVerts();
        // record surface (tri) indices
        if (primHandle.category != ZenoParticles::category_e::curve) {
            auto &tris = primHandle.getSurfTris();
            pol(Collapse(tris.size()),
                [stInds = view<space>({}, stInds), tris = view<space>({}, tris), voffset = primHandle.vOffset,
                 sfoffset = primHandle.sfOffset] __device__(int i) mutable {
                    stInds.tuple(dim_c<3>, "inds", sfoffset + i, int_c) =
                        (tris.pack(dim_c<3>, "inds", i, int_c) + (int)voffset);
                });
        }
        const auto &edges = primHandle.getSurfEdges();
        pol(Collapse(edges.size()), [seInds = view<space>({}, seInds), edges = view<space>({}, edges),
                                     voffset = primHandle.vOffset, seoffset = primHandle.seOffset,
                                     tempE = proxy<space>({}, tempE), 
                                     verts = view<space>({}, verts)] __device__(int i) mutable {
            auto inds = edges.pack(dim_c<2>, "inds", i, int_c);
            auto edge = inds + (int)voffset;
            seInds.tuple(dim_c<2>, "inds", seoffset + i, int_c) = edge;
            tempE.tuple(dim_c<2>, "inds", seoffset + i, int_c) = edge; 
        });
        const auto &points = primHandle.getSurfVerts();
        pol(Collapse(points.size()),
            [svInds = view<space>({}, svInds), points = view<space>({}, points), voffset = primHandle.vOffset,
             svoffset = primHandle.svOffset] __device__(int i) mutable {
                svInds("inds", svoffset + i, int_c) = points("inds", i, int_c) + (int)voffset;
            });
    }
    spInds = svInds; 
    spInds.resize((std::size_t)(svInds.size() + coPoints->size())); 
    fmt::print("spInds.size: {}\n", spInds.size()); 
    pol(range(coPoints->size()), 
        [spInds = proxy<space>({}, spInds), 
         svoffset = svInds.size(), 
         coOffset = coOffset] __device__ (int i) mutable {
            spInds("inds", i + svoffset, int_c) = coOffset + i; 
        }); 

    auto deduce_node_cnt = [](std::size_t numLeaves) {
        if (numLeaves <= 2)
            return numLeaves;
        return numLeaves * 2 - 1;
    };
    selfStFront = bvfront_t{(int)deduce_node_cnt(stInds.size()), (int)bvhFrontCps, zs::memsrc_e::um, vtemp.devid()};
    selfSeeFront = bvfront_t{(int)deduce_node_cnt(seInds.size()), (int)bvhFrontCps, zs::memsrc_e::um, vtemp.devid()};
    if constexpr (enablePE_c)
        selfSevFront = bvfront_t{(int)deduce_node_cnt(seInds.size()), (int)bvhFrontCps, zs::memsrc_e::um, vtemp.devid()};
    if constexpr (enablePP_c)
        selfSvFront = bvfront_t{(int)deduce_node_cnt(svInds.size()), (int)bvhFrontCps, zs::memsrc_e::um, vtemp.devid()};
    if (hasBoundary()) {
        boundaryStFront = bvfront_t{(int)deduce_node_cnt(coEles->size()), (int)bvhFrontCps, zs::memsrc_e::um, vtemp.devid()};
        boundarySeeFront = bvfront_t{(int)deduce_node_cnt(coEdges->size()), (int)bvhFrontCps, zs::memsrc_e::um, vtemp.devid()};
        if constexpr (enablePE_c)
            boundarySevFront = bvfront_t{(int)deduce_node_cnt(coEdges->size()), (int)bvhFrontCps, zs::memsrc_e::um, vtemp.devid()};
        // boundarySvFront = bvfront_t{(int)deduce_node_cnt(coPoints->size()), (int)bvhFrontCps, zs::memsrc_e::um, vtemp.devid()};
    }

    /// WARN: ignore BC verts initialization here
    D = 0;  
    reinitialize(pol, dt); 
}

void RapidClothSystem::reinitialize(zs::CudaExecutionPolicy &pol, T framedt) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    dt = framedt;
    this->framedt = framedt;
    curRatio = 0;
    substep = -1;

    /// cloth dynamics status
    for (auto &primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        auto &verts = primHandle.getVerts();
        pol(Collapse(verts.size()), [vtemp = view<space>({}, vtemp), verts = view<space>({}, verts),
                                     voffset = primHandle.vOffset, dt = dt] __device__(int i) mutable {
            auto x = verts.pack<3>("x", i);
            auto v = verts.pack<3>("v", i);
            auto vi = voffset + i; 

            vtemp("ws", vi) = verts("m", i);
            vtemp.tuple(dim_c<3>, "x[0]", vi) = x;
            vtemp.tuple(dim_c<3>, "x[k]", vi) = x;
            vtemp.tuple(dim_c<3>, "x(l)", vi) = x; 
            vtemp.tuple(dim_c<3>, "v[0]", vi) = v;
        });
    }
    if (hasBoundary())
        if (auto coSize = coVerts->size(); coSize) {
            pol(Collapse(coSize),
                [vtemp = view<space>({}, vtemp), coverts = view<space>({}, *coVerts), coOffset = coOffset, dt = dt,
                 augLagCoeff = augLagCoeff, avgNodeMass = avgNodeMass] __device__(int i) mutable {
                    auto x = coverts.pack<3>("x", i);
                    auto v = coverts.pack<3>("v", i);
                    int vi = coOffset + i; 

                    vtemp("ws", coOffset + i) = avgNodeMass * augLagCoeff;
                    vtemp.tuple(dim_c<3>, "x[0]", vi) = x;
                    vtemp.tuple(dim_c<3>, "x[k]", vi) = x;
                    vtemp.tuple(dim_c<3>, "x(l)", vi) = x; 
                    vtemp.tuple(dim_c<3>, "y[k+1]", vi) = x + v * dt; 
                    vtemp.tuple(dim_c<3>, "y(l)", vi) = x + v * dt; 
                    vtemp.tuple(dim_c<3>, "v[0]", vi) = v;
                });
        }

    frontManageRequired = true; 
#define init_front(sInds, front)                                                                           \
    {                                                                                                      \
        auto numNodes = front.numNodes();                                                                  \
        if (numNodes <= 2) {                                                                               \
            front.reserve(sInds.size() * numNodes);                                                        \
            front.setCounter(sInds.size() * numNodes);                                                     \
            pol(Collapse{sInds.size()}, [front = proxy<space>(front), numNodes] ZS_LAMBDA(int i) mutable { \
                for (int j = 0; j != numNodes; ++j)                                                        \
                    front.assign(i *numNodes + j, i, j);                                                   \
            });                                                                                            \
        } else {                                                                                           \
            front.reserve(sInds.size());                                                                   \
            front.setCounter(sInds.size());                                                                \
            pol(Collapse{sInds.size()},                                                                    \
                [front = proxy<space>(front)] ZS_LAMBDA(int i) mutable { front.assign(i, i, 0); });        \
        }                                                                                                  \
    }
    {
        if constexpr (enablePP_c)
        {
            bvs.resize(svInds.size()); 
            retrieve_bounding_volumes(pol, vtemp, "x[0]", svInds, zs::wrapv<1>{}, 0, bvs);
            svBvh.build(pol, bvs); 
            init_front(spInds, selfSvFront);             
        }

        bvs.resize(stInds.size());
        retrieve_bounding_volumes(pol, vtemp, "x[0]", stInds, zs::wrapv<3>{}, 0, bvs);
        stBvh.build(pol, bvs);
        init_front(spInds, selfStFront); 

        bvs.resize(seInds.size());
        retrieve_bounding_volumes(pol, vtemp, "x[0]", seInds, zs::wrapv<2>{}, 0, bvs);
        seBvh.build(pol, bvs);
        init_front(seInds, selfSeeFront);
        if constexpr (enablePE_c)
            init_front(spInds, selfSevFront); 
    }
    if (hasBoundary()) {
        bvs.resize(coEles->size());
        retrieve_bounding_volumes(pol, vtemp, "x[0]", *coEles, zs::wrapv<3>{}, coOffset, bvs);
        bouStBvh.build(pol, bvs);
        init_front(svInds, boundaryStFront);

        bvs.resize(coEdges->size());
        retrieve_bounding_volumes(pol, vtemp, "x[0]", *coEdges, zs::wrapv<2>{}, coOffset, bvs);
        bouSeBvh.build(pol, bvs);
        init_front(seInds, boundarySeeFront);
        if constexpr (enablePE_c)
            init_front(svInds, boundarySevFront);
    }
}

RapidClothSystem::RapidClothSystem(std::vector<ZenoParticles *> zsprims, tiles_t *coVerts, tiles_t *coPoints, tiles_t *coEdges,
                    tiles_t *coEles, T dt, std::size_t ncps, std::size_t bvhFrontCps, bool withContact, T augLagCoeff, T cgRel, 
                    T lcpTol, int PNCap, int CGCap, int lcpCap, T gravity, int L, T delta, T sigma, T gamma, T eps, int maxVertCons, 
                    T BCStiffness, T shrinkFactor)
    : coVerts{coVerts}, coPoints{coPoints}, coEdges{coEdges}, coEles{coEles}, estNumCps{ncps}, bvhFrontCps{bvhFrontCps}, 
        nPP{zsprims[0]->getParticles().get_allocator(), 1}, nPE{zsprims[0]->getParticles().get_allocator(), 1},
        nPT{zsprims[0]->getParticles().get_allocator(), 1}, nEE{zsprims[0]->getParticles().get_allocator(), 1},
        nE{zsprims[0]->getParticles().get_allocator(), 1}, temp{estNumCps, zs::memsrc_e::um, 0},
        oPP{zsprims[0]->getParticles().get_allocator(), 1}, oPE{zsprims[0]->getParticles().get_allocator(), 1},
        oPT{zsprims[0]->getParticles().get_allocator(), 1}, oEE{zsprims[0]->getParticles().get_allocator(), 1},
        oE{zsprims[0]->getParticles().get_allocator(), 1}, 
        dt{dt}, framedt{dt}, curRatio{0}, enableContact{withContact}, augLagCoeff{augLagCoeff},
        cgRel{cgRel}, lcpTol{lcpTol}, PNCap{PNCap}, CGCap{CGCap}, lcpCap{lcpCap}, gravAccel{0, gravity, 0}, L{L}, delta{delta}, 
        D_min{delta * 2}, D_max{delta * 4}, sigma{sigma}, gamma{gamma}, eps{eps}, maxVertCons{maxVertCons}, 
        consDegree{maxVertCons * 4}, BCStiffness{BCStiffness}, consShrinking{shrinkFactor} {
    auto cudaPol = zs::cuda_exec();
    coOffset = sfOffset = seOffset = svOffset = 0;
    for (auto primPtr : zsprims) {
        auto bv = primPtr->computeBoundingVolume(cudaPol, "x");
        primPtr->orderByMortonCode(cudaPol, bv);

        if (primPtr->category == ZenoParticles::category_e::curve) {
            prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset, zs::wrapv<2>{});
        } else if (primPtr->category == ZenoParticles::category_e::surface)
            prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset, zs::wrapv<3>{});
        else if (primPtr->category == ZenoParticles::category_e::tet)
            prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset, zs::wrapv<4>{});
    }
    numDofs = coOffset;
    if (hasBoundary())
        numDofs += coVerts->size();
    numBouDofs = numDofs - coOffset;

    fmt::print("num total obj <verts, bouVerts, surfV, surfE, surfT>: {}, {}, {}, {}, {}\n", coOffset, numBouDofs,
               svOffset, seOffset, sfOffset);

    tempCons = itiles_t{
        zsprims[0]->getParticles().get_allocator(), 
        {
            // graph coloring 
            {"fixed", 1},        
            {"max_color", 1}, 
            {"num_color", 1}, 
            {"tmp", 1}, 
            // use its bits, e.g. 110 means 
            // color-0: not available, color-1: okay, color-2: okay
            // {"colors", 1}, use tempColors which is a zs::Vector
            {"color", 1}, 
            // topology
            {"vi", 4}, 
            {"vN", 1}, 
            {"adj", maxVertCons * 4}, 
            // LCP
            {"lambda", 1},  // float, lambda in LCP  
            {"grad", 12},   // float, constraint gradient 
            {"dist", 1}, 
            {"val", 1},     // float, constraint value
            {"b", 1},       // float, b in A*lambda+b, for LCP  
            {"diag", 1}     // float, diag element in the current row 
        }, 
        (std::size_t)estNumCps
    }; 
    tempColors = {
        zsprims[0]->getParticles().get_allocator(), 
        (std::size_t)estNumCps
    }; 
    tempPP = tiles_t{
        zsprims[0]->getParticles().get_allocator(), 
        {
            {"inds", 2}, 
            {"dist", 1}
        }, 
        (std::size_t)estNumCps
    }; 
    tempPE = tiles_t{
        zsprims[0]->getParticles().get_allocator(), 
        {
            {"inds", 3}, 
            {"dist", 1}
        }, 
        (std::size_t)estNumCps
    }; 
    tempPT = tiles_t{
        zsprims[0]->getParticles().get_allocator(), 
        {
            {"inds", 4}, 
            {"dist", 1}
        }, 
        (std::size_t)estNumCps
    }; 
    tempEE = tiles_t{
        zsprims[0]->getParticles().get_allocator(), 
        {
            {"inds", 4}, 
            {"dist", 1}
        }, 
        (std::size_t)estNumCps
    }; 
    tempE = tiles_t{
        zsprims[0]->getParticles().get_allocator(), 
        {
            {"inds", 2},  
            {"dist", 1}
        }, 
        (std::size_t)estNumCps
    }; 
    vCons = itiles_t{zsprims[0]->getParticles().get_allocator(), 
                    {
                        {"n", 1}, 
                        {"nE", 1}, 
                        {"cons", maxVertCons}, 
                        {"ind", maxVertCons}    // its index in each constraint 
                    }, 
                    (std::size_t)coOffset       // stiff BC
    }; 
    opp = ope = opt = oee = oe = 0;             // offsets
    npp = npe = npt = nee = ne = 0;
    vtemp = tiles_t{zsprims[0]->getParticles().get_allocator(),
                    {
                        // boundary
                        {"ws", 1},
                        {"cons", 3},
                        {"isBC", 1},            // 0 or 1
                        {"BCtarget", 3},  
                        // cloth dynamics
                        {"x[0]", 3},
                        {"x[k]", 3},  
                        {"y[k+1]", 3}, 
                        {"v[0]", 3}, 
                        {"x(l)", 3}, 
                        {"r(l)", 1}, 
                        {"y(l)", 3}, 
                        {"disp", 1}, 
                        {"x_tilde", 3},
                        {"x_hat", 3}, 
                        // linear solver
                        {"dir", 3},
                        {"grad", 3},
                        {"P", 9},               // implement Newton solver first 
                        {"r", 3},
                        {"p", 3},
                        {"q", 3},
                        // forward step
                        {"Di", 1}, 
                        // intermediate
                        {"temp", 3},
                    },
                    (std::size_t)numDofs};
    bvs = zs::Vector<bv_t>{vtemp.get_allocator(), vtemp.size()}; // this size is the upper bound
    consColorBits = zs::Vector<int>{(std::size_t)sizeof(zs::i64) * 8, zs::memsrc_e::um, 0}; 
    lcpMat = spmat_t{zs::memsrc_e::device}; 
    // lcpMatIs = lcpMatJs = {vtemp.get_allocator(), maxVertCons * 3 * estNumCps * 4}; 
    // TODO: use a different parameter instead of this estNumCps to control lcpMat nnz size 
    lcpMatIs = lcpMatJs = {vtemp.get_allocator(), estNumCps}; 
    fmt::print("init lcpMatIs size: {}\n", lcpMatIs.size()); 
    lcpConverged = lcpMatSize = {vtemp.get_allocator(), 1}; 
    // average edge length (for CCD filtering)
    initialize(cudaPol); // update vtemp, bvh, boxsize, targetGRes
                         // adaptive dhat, targetGRes, kappa
}

void RapidClothSystem::advanceSubstep(zs::CudaExecutionPolicy &pol, T ratio) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;

    // setup substep dt
    ++substep;
    dt = framedt * ratio;
    curRatio += ratio;

    pol(Collapse(coOffset), [vtemp = view<space>({}, vtemp), coOffset = coOffset, dt = dt] __device__(int vi) mutable {
        auto xk = vtemp.pack(dim_c<3>, "x[k]", vi);
        vtemp.tuple(dim_c<3>, "x_hat", vi) = xk;
        auto newX = xk + vtemp.pack(dim_c<3>, "v[0]", vi) * dt;
        vtemp.tuple(dim_c<3>, "x_tilde", vi) = newX;
    });
    if (hasBoundary())
        if (auto coSize = coVerts->size(); coSize)
            pol(Collapse(coSize), [vtemp = view<space>({}, vtemp), coverts = view<space>({}, *coVerts),
                                   coOffset = coOffset, dt = dt] __device__(int i) mutable {
                auto xk = vtemp.pack(dim_c<3>, "x[k]", coOffset + i);
                vtemp.tuple(dim_c<3>, "x_hat", coOffset + i) = xk;
                auto newX = xk + coverts.pack(dim_c<3>, "v", i) * dt;
                vtemp.tuple(dim_c<3>, "x_tilde", coOffset + i) = newX;
            });
}

void RapidClothSystem::updateVelocities(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    pol(zs::range(coOffset), [vtemp = view<space>({}, vtemp), dt = dt] __device__(int vi) mutable {
        auto newX = vtemp.pack<3>("x[k]", vi);
        auto dv = (newX - vtemp.pack<3>("x_tilde", vi)) / dt;
        auto vn = vtemp.pack<3>("v[0]", vi);
        vn += dv;
        vtemp.tuple<3>("v[0]", vi) = vn;
    });
}

void RapidClothSystem::writebackPositionsAndVelocities(zs::CudaExecutionPolicy &pol) {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    for (auto &primHandle : prims) {
        if (primHandle.isAuxiliary())
            continue;
        if (primHandle.isBoundary())
            continue;
        auto &verts = primHandle.getVerts();
        // update velocity and positions
        pol(zs::range(verts.size()),
            [vtemp = view<space>({}, vtemp), verts = view<space>({}, verts), dt = dt, vOffset = primHandle.vOffset,
             asBoundary = primHandle.isBoundary()] __device__(int vi) mutable {
                verts.tuple<3>("x", vi) = vtemp.pack<3>("x[k]", vOffset + vi);
                if (!asBoundary)
                    verts.tuple<3>("v", vi) = vtemp.pack<3>("v[0]", vOffset + vi);
            });
    }
    if (hasBoundary())
        pol(Collapse(coVerts->size()),
            [vtemp = view<space>({}, vtemp), verts = view<space>({}, *const_cast<tiles_t *>(coVerts)),
             coOffset = coOffset] ZS_LAMBDA(int vi) mutable {
                verts.tuple(dim_c<3>, "x", vi) = vtemp.pack(dim_c<3>, "x[k]", coOffset + vi);
                // no need to update v here. positions are moved accordingly
                // also, boundary velocies are set elsewhere
            });
}

struct MakeRapidClothSystem : INode {
    using tiles_t = typename RapidClothSystem::tiles_t; 

    void apply() override {
        using namespace zs;
        auto zsprims = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");
        std::shared_ptr<ZenoParticles> zsboundary;
        if (has_input<ZenoParticles>("ZSBoundaryPrimitives"))
            zsboundary = get_input<ZenoParticles>("ZSBoundaryPrimitives");

        tiles_t *coVerts = zsboundary ? &zsboundary->getParticles() : nullptr;
        tiles_t *coPoints =
            zsboundary ? &(*zsboundary)[ZenoParticles::s_surfVertTag] : nullptr;
        tiles_t *coEdges =
            zsboundary ? &(*zsboundary)[ZenoParticles::s_surfEdgeTag] : nullptr;
        tiles_t *coEles = zsboundary ? &zsboundary->getQuadraturePoints() : nullptr;

        if (zsboundary) {
            auto pol = cuda_exec();
            compute_surface_neighbors(pol, *coEles, *coEdges, *coPoints);
            coEles->append_channels(pol, {{"nrm", 3}});
            coEdges->append_channels(pol, {{"nrm", 3}});
        }

        /// solver parameters
        auto input_est_num_cps = get_input2<int>("est_num_cps");
        auto input_bvh_front_cps = get_input2<int>("bvh_front_cps"); 
        auto input_withContact = get_input2<bool>("with_contact");
        auto input_contactEE = get_input2<bool>("contact_with_ee");
        auto input_contactSelf = get_input2<bool>("contact_with_self");
        auto input_aug_coeff = get_input2<float>("aug_coeff");
        auto input_cg_rel = get_input2<float>("cg_rel");
        auto input_pn_cap = get_input2<int>("pn_iter_cap");
        auto input_cg_cap = get_input2<int>("cg_iter_cap");
        auto input_gravity = get_input2<float>("gravity");
        auto input_BC_stiffness = get_input2<float>("BC_stiffness"); 
        auto input_dt = get_input2<float>("dt");
        auto input_L = get_input2<int>("collision_iters");
        auto input_delta = get_input2<float>("delta"); 
        auto input_sigma = get_input2<float>("edge_violation_ratio"); 
        auto input_gamma = get_input2<float>("stepping_limit"); 
        auto input_eps = get_input2<float>("term_thresh");
        auto input_max_vert_cons = get_input2<int>("max_vert_cons");  
        auto input_lcp_tol = get_input2<int>("lcp_tol"); 
        auto input_lcp_cap = get_input2<int>("lcp_cap"); 
        auto input_shrink_factor = get_input2<float>("shrink_factor"); 

        // T delta, T sigma, T gamma, T eps
        auto A = std::make_shared<RapidClothSystem>(zsprims, coVerts, coPoints, coEdges, coEles, input_dt,
                                                   (std::size_t)(input_est_num_cps ? input_est_num_cps : 100000),
                                                   (std::size_t)(input_bvh_front_cps ? input_bvh_front_cps : 10000000), 
                                                   input_withContact, input_aug_coeff, input_cg_rel, input_lcp_tol,  
                                                   input_pn_cap, input_cg_cap, input_lcp_cap, input_gravity, input_L, 
                                                   input_delta, input_sigma, input_gamma, input_eps, 
                                                   input_max_vert_cons, input_BC_stiffness, input_shrink_factor);
        A->enableContactSelf = input_contactSelf;

        set_output("ZSClothSystem", A);
    }
};

ZENDEFNODE(MakeRapidClothSystem, {{"ZSParticles",
                              "ZSBoundaryPrimitives",
                              {"int", "est_num_cps", "100000"},
                              {"int", "bvh_front_cps", "10000000"}, 
                              {"int", "max_vert_cons", "32"}, 
                              {"bool", "with_contact", "1"},
                              {"bool", "contact_with_ee", "1"},
                              {"bool", "contact_with_self", "1"},
                              {"float", "dt", "0.01"},
                              {"float", "aug_coeff", "1e2"},
                              {"float", "cg_rel", "0.001"},
                              {"float", "lcp_tol", "0.001"}, 
                              {"int", "pn_iter_cap", "3"},
                              {"int", "cg_iter_cap", "200"},
                              {"int", "lcp_cap", "256"}, 
                              {"float", "gravity", "-9.8"},
                              {"int", "collision_iters", "512"}, 
                              {"float", "delta", "1"}, 
                              {"float", "shrink_factor", "1.1"}, 
                              {"float", "edge_violation_ratio", "1.1"}, 
                              {"float", "stepping_limit", "0.9"},  
                              {"float", "term_thresh", "1e-4"}, 
                              {"float", "BC_stiffness", "1000"}},
                             {"ZSClothSystem"},
                             {},
                             {"FEM"}});

} // namespace zeno