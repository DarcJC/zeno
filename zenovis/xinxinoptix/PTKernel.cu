#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/random.h>
#include <cuda/helpers.h>
#include <sutil/vec_math.h>

#include "optixPathTracer.h"
#include "TraceStuff.h"
#include "DisneyBSDF.h"
#include "zxxglslvec.h"
#include "proceduralSky.h"

#include <hip/hip_fp16.h>

extern "C" {
__constant__ Params params;

}
//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------
static __inline__ __device__
vec3 RRTAndODTFit(vec3 v)
{
    vec3 a = v * (v + 0.0245786f) - 0.000090537f;
    vec3 b = v * (0.983729f * v + 0.4329510f) + 0.238081f;
    return a / b;
}
static __inline__ __device__
vec3 ACESFitted(vec3 color, float gamma)
{
//    const mat3x3 ACESInputMat = mat3x3
//        (
//            0.59719, 0.35458, 0.04823,
//            0.07600, 0.90834, 0.01566,
//            0.02840, 0.13383, 0.83777
//        );
//    mat3x3 ACESOutputMat = mat3x3
//    (
//        1.60475, -0.53108, -0.07367,
//        -0.10208,  1.10813, -0.00605,
//        -0.00327, -0.07276,  1.07602
//    );
    vec3 v1 = vec3(0.59719, 0.35458, 0.04823);
    vec3 v2 = vec3(0.07600, 0.90834, 0.01566);
    vec3 v3 = vec3(0.02840, 0.13383, 0.83777);
    color = vec3(dot(color, v1), dot(color, v2), dot(color, v3));
    // Apply RRT and ODT
    color = RRTAndODTFit(color);

    v1 = vec3(1.60475, -0.53108, -0.07367);
    v2 = vec3(-0.10208,  1.10813, -0.00605);
    v3 = vec3(-0.00327, -0.07276,  1.07602);
    color = vec3(dot(color, v1), dot(color, v2), dot(color, v3));

    // Clamp to [0, 1]
    color = clamp(color, 0.0f, 1.0f);

    color = pow(color, vec3(1.0f / gamma));

    return color;
}

extern "C" __global__ void __raygen__rg()
{

      const int    w   = params.windowSpace.x;
      const int    h   = params.windowSpace.y;
      //const float3 eye = params.eye;
      const uint3  idxx = optixGetLaunchIndex();
      uint3 idx;
      idx.x = idxx.x + params.tile_i * params.tile_w;
      idx.y = idxx.y + params.tile_j * params.tile_h;
      if(idx.x>w || idx.y>h)
        return;
      const unsigned int image_index  = idx.y * w + idx.x;
      const int    subframe_index = params.subframe_index;
      const CameraInfo cam = params.cam;

      int seedy = idx.y/4, seedx = idx.x/8;
      int sid = (idx.y%4) * 8 + idx.x%8;
      unsigned int seed = tea<4>( idx.y * w + idx.x, subframe_index);
    //   auto tmp = idx.y * w + idx.x + subframe_index * w * h;
    //   unsigned int seed = pcg_hash(tmp);

      unsigned int eventseed = seed; //tea<4>( idx.y * w + idx.x, subframe_index+1);
      float focalPlaneDistance = cam.focalPlaneDistance>0.01f? cam.focalPlaneDistance : 0.01f;
      float aperture = clamp(cam.aperture,0.0f,100.0f);
      aperture/=10;

      float3 result = make_float3( 0.0f );
      float3 result_d = make_float3( 0.0f );
      float3 result_s = make_float3( 0.0f );
      float3 result_t = make_float3( 0.0f );
      float3 result_b = make_float3( 0.0f );
      int i = params.samples_per_launch;

      float3 tmp_albedo{};
      float3 tmp_normal{};
      unsigned int sobolseed = subframe_index;
      do
      {
          // The center of each pixel is at fraction (0.5,0.5)
          float2 subpixel_jitter = sobolRnd(sobolseed);

          float2 d = 2.0f * make_float2(
                  ( static_cast<float>( idx.x + params.windowCrop_min.x ) + subpixel_jitter.x ) / static_cast<float>( w ),
                  ( static_cast<float>( idx.y + params.windowCrop_min.y ) + subpixel_jitter.y ) / static_cast<float>( h )
                  ) - 1.0f;

          float2 r01 = sobolRnd(sobolseed);

          float r0 = r01.x * 2.0f * M_PIf;
          float r1 = r01.y * aperture * aperture;
          r1 = sqrtf(r1);

          float3 eye_shake     = r1 * ( cosf(r0)* normalize(cam.right) + sinf(r0)* normalize(cam.up)); // Camera local space
          float3 ray_origin    = cam.eye + eye_shake;
          float3 ray_direction = focalPlaneDistance / length(cam.front) * (cam.right * d.x + cam.up * d.y + cam.front) - eye_shake; // Camera local space
                 ray_direction = normalize(ray_direction);

          RadiancePRD prd;
          prd.emission     = make_float3(0.f);
          prd.radiance     = make_float3(0.f);
          prd.attenuation  = make_float3(1.f);
          prd.attenuation2 = make_float3(1.f);
          prd.prob         = 1.0f;
          prd.prob2        = 1.0f;
          prd.countEmitted = true;
          prd.done         = false;
          prd.seed         = seed;
          prd.eventseed    = eventseed;
          prd.flags        = 0;
          prd.maxDistance  = 1e16f;
          prd.medium       = DisneyBSDF::PhaseFunctions::vacuum;

        prd.origin = ray_origin;
        prd.direction = ray_direction;
        prd.samplePdf = 1.0f;

        prd.depth = 0;
        prd.diffDepth = 0;
        prd.isSS = false;
        prd.curMatIdx = 0;
        prd.test_distance = false;
        prd.ss_alpha_queue[0] = vec3(-1.0f);
        prd.minSpecRough = 0.01;
        prd.samplePdf = 1.0f;
        prd.first_hit_type = 0;
        prd.hitEnv = false;
        auto _tmin_ = prd._tmin_;
        auto _mask_ = prd._mask_;
        
        //if constexpr(params.denoise) 
        if (params.denoise) 
        {
            prd.trace_denoise_albedo = true;
            prd.trace_denoise_normal = true;
        }

        // Primary Ray
        traceRadiance(params.handle, ray_origin, ray_direction, _tmin_, prd.maxDistance, &prd, _mask_);

        tmp_albedo = prd.tmp_albedo;
        tmp_normal = prd.tmp_normal;

        prd.trace_denoise_albedo = false;
        prd.trace_denoise_normal = false;

        for(;;)
        {
            prd.radiance_d = make_float3(0);
            prd.radiance_s = make_float3(0);
            prd.radiance_t = make_float3(0);

            _tmin_ = prd._tmin_;
            _mask_ = prd._mask_;

            prd._tmin_ = 0;
            prd._mask_ = EverythingMask; 

            ray_origin = prd.origin;
            ray_direction = prd.direction;

            if(prd.countEmitted==false || prd.depth>0) {
                auto temp_radiance = prd.radiance * prd.attenuation2;

                //float upperBound = prd.fromDiff?1.0f:1.0f;
                float3 clampped = clamp(vec3(temp_radiance), vec3(0), vec3(10));

                result += prd.depth>1?clampped:temp_radiance;
                if(prd.depth==1 && prd.hitEnv == false)
                {
                    result_d += prd.radiance_d * prd.attenuation2;
                    result_s += prd.radiance_s * prd.attenuation2;
                    result_t += prd.radiance_t * prd.attenuation2;
                }
                if(prd.depth>1 || (prd.depth==1 && prd.hitEnv == true)) {
                    result_d +=
                        prd.first_hit_type == 1 ? clampped : make_float3(0, 0, 0);
                    result_s +=
                        prd.first_hit_type == 2 ? clampped : make_float3(0, 0, 0);
                    result_t +=
                        prd.first_hit_type == 3 ? clampped : make_float3(0, 0, 0);
                }

            }

            prd.radiance = make_float3(0);
            prd.emission = make_float3(0);

            if(prd.countEmitted==true && prd.depth>0){
                prd.done = true;
            }

            if( prd.done || params.simpleRender==true){
                break;
            }

            if(prd.depth>16) {
                float RRprob = clamp(length(prd.attenuation),0.1f, 0.95f);
                if(rnd(prd.seed) > RRprob || prd.depth > 24) {
                    prd.done=true;
                } else {
                    prd.attenuation = prd.attenuation / RRprob;
                }
            }
            if(prd.countEmitted == true)
                prd.passed = true;

            traceRadiance(params.handle, ray_origin, ray_direction, _tmin_, prd.maxDistance, &prd, _mask_);
        }
        result_b += prd.first_hit_type == 0 ? make_float3(0, 0, 0)
                                            : make_float3(1, 1, 1);
        seed = prd.seed;
    }
    while( --i );

    auto samples_per_launch = static_cast<float>( params.samples_per_launch );

    float3         accum_color    = result   / samples_per_launch;
    float3         accum_color_d  = result_d / samples_per_launch;
    float3         accum_color_s  = result_s / samples_per_launch;
    float3         accum_color_t  = result_t / samples_per_launch;
    float3         accum_color_b  = result_b / samples_per_launch;
    
    if( subframe_index > 0 )
    {
        const float                 a = 1.0f / static_cast<float>( subframe_index+1 );
        const float3 accum_color_prev = make_float3( params.accum_buffer[ image_index ]);
        const float3 accum_color_prev_d = make_float3( params.accum_buffer_D[ image_index ]);
        const float3 accum_color_prev_s = make_float3( params.accum_buffer_S[ image_index ]);
        const float3 accum_color_prev_t = make_float3( params.accum_buffer_T[ image_index ]);
        const float3 accum_color_prev_b = make_float3( params.accum_buffer_B[ image_index ]);
        accum_color   = lerp( accum_color_prev, accum_color, a );
        accum_color_d = lerp( accum_color_prev_d, accum_color_d, a );
        accum_color_s = lerp( accum_color_prev_s, accum_color_s, a );
        accum_color_t = lerp( accum_color_prev_t, accum_color_t, a );
        accum_color_b = lerp( accum_color_prev_b, accum_color_b, a );

        if (params.denoise) {

            const float3 accum_albedo_prev = params.albedo_buffer[ image_index ];
            tmp_albedo = lerp(accum_albedo_prev, tmp_albedo, a);

            const float3 accum_normal_prev = params.normal_buffer[ image_index ];
            tmp_normal = lerp(accum_normal_prev, tmp_normal, a);
        }
    }

    params.accum_buffer[ image_index ] = make_float4( accum_color, 1.0f);
    params.accum_buffer_D[ image_index ] = make_float4( accum_color_d, 1.0f);
    params.accum_buffer_S[ image_index ] = make_float4( accum_color_s, 1.0f);
    params.accum_buffer_T[ image_index ] = make_float4( accum_color_t, 1.0f);
    params.accum_buffer_B[ image_index ] = make_float4( accum_color_b, 1.0f);
    //vec3 aecs_fitted = ACESFitted(vec3(accum_color), 2.2);
    float3 out_color = accum_color;
    float3 out_color_d = accum_color_d;
    float3 out_color_s = accum_color_s;
    float3 out_color_t = accum_color_t;
    float3 out_color_b = accum_color_b;
    params.frame_buffer[ image_index ] = make_color ( out_color );
    params.frame_buffer_C[ image_index ] = accum_color;
    params.frame_buffer_D[ image_index ] = accum_color_d;
    params.frame_buffer_S[ image_index ] = accum_color_s;
    params.frame_buffer_T[ image_index ] = accum_color_t;
    params.frame_buffer_B[ image_index ] = accum_color_b;

    if (params.denoise) {
        params.albedo_buffer[ image_index ] = tmp_albedo;
        params.normal_buffer[ image_index ] = tmp_normal;
    }
}

extern "C" __global__ void __miss__radiance()
{
    vec3 sunLightDir = vec3(
            params.sunLightDirX,
            params.sunLightDirY,
            params.sunLightDirZ
            );
    MissData* rt_data  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    RadiancePRD* prd = getPRD();
    prd->attenuation2 = prd->attenuation;
    prd->passed = false;
    prd->countEmitted = false;
    
    if(prd->medium != DisneyBSDF::PhaseFunctions::isotropic){
        float upperBound = 100.0f;
        float envPdf = 0.0f;
        vec3 skysample =
            envSky(
            normalize(prd->direction),
            sunLightDir,
            make_float3(0., 0., 1.),
            40, // be careful
            .45,
            15.,
            1.030725f * 0.3f,
            params.elapsedTime,
            envPdf,
            upperBound,
            0.0

        );

        float misWeight = BRDFBasics::PowerHeuristic(prd->samplePdf,envPdf);

        misWeight = misWeight>0.0f?misWeight:0.0f;
        misWeight = envPdf>0.0f?misWeight:1.0f;
        misWeight = prd->depth>=1?misWeight:1.0f;
        misWeight = prd->samplePdf>0.0f?misWeight:1.0f;
        
        prd->radiance = misWeight * skysample;

        if (params.show_background == false) {
            prd->radiance = prd->depth>=1?prd->radiance:make_float3(0,0,0);
        }

        prd->done      = true;
        prd->hitEnv    = true;
        return;
    }

    vec3 sigma_t, ss_alpha;
    //vec3 sigma_t, ss_alpha;
    prd->readMat(sigma_t, ss_alpha);


    vec3 transmittance;
    if (ss_alpha.x < 0.0f) { // is inside Glass
        transmittance = DisneyBSDF::Transmission(sigma_t, optixGetRayTmax());
    } else {
        transmittance = DisneyBSDF::Transmission2(sigma_t * ss_alpha, sigma_t, prd->channelPDF, optixGetRayTmax(), false);
    }

    prd->attenuation *= transmittance;//DisneyBSDF::Transmission(prd->extinction,optixGetRayTmax());
    prd->attenuation2 *= transmittance;//DisneyBSDF::Transmission(prd->extinction,optixGetRayTmax());
    prd->origin += prd->direction * optixGetRayTmax();
    prd->direction = DisneyBSDF::SampleScatterDirection(prd->seed);

    vec3 channelPDF = vec3(1.0f/3.0f);
    prd->channelPDF = channelPDF;
    if (ss_alpha.x < 0.0f) { // is inside Glass
        prd->maxDistance = DisneyBSDF::SampleDistance(prd->seed, prd->scatterDistance);
    } else
    {
        prd->maxDistance =
            DisneyBSDF::SampleDistance2(prd->seed, vec3(prd->attenuation) * ss_alpha, sigma_t, channelPDF);
        prd->channelPDF = channelPDF;
    }

    prd->depth++;

    if(length(prd->attenuation)<1e-7f){
        prd->done = true;
    }
}

extern "C" __global__ void __miss__occlusion()
{
    setPayloadOcclusion( false );
}

extern "C" __global__ void __closesthit__occlusion()
{
    setPayloadOcclusion( true );
}